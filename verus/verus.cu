#include "hip/hip_runtime.h"
#include <miner.h>

#include <cuda_helper.h>



#define saes_data(w) {\
    w(0x63), w(0x7c), w(0x77), w(0x7b), w(0xf2), w(0x6b), w(0x6f), w(0xc5),\
    w(0x30), w(0x01), w(0x67), w(0x2b), w(0xfe), w(0xd7), w(0xab), w(0x76),\
    w(0xca), w(0x82), w(0xc9), w(0x7d), w(0xfa), w(0x59), w(0x47), w(0xf0),\
    w(0xad), w(0xd4), w(0xa2), w(0xaf), w(0x9c), w(0xa4), w(0x72), w(0xc0),\
    w(0xb7), w(0xfd), w(0x93), w(0x26), w(0x36), w(0x3f), w(0xf7), w(0xcc),\
    w(0x34), w(0xa5), w(0xe5), w(0xf1), w(0x71), w(0xd8), w(0x31), w(0x15),\
    w(0x04), w(0xc7), w(0x23), w(0xc3), w(0x18), w(0x96), w(0x05), w(0x9a),\
    w(0x07), w(0x12), w(0x80), w(0xe2), w(0xeb), w(0x27), w(0xb2), w(0x75),\
    w(0x09), w(0x83), w(0x2c), w(0x1a), w(0x1b), w(0x6e), w(0x5a), w(0xa0),\
    w(0x52), w(0x3b), w(0xd6), w(0xb3), w(0x29), w(0xe3), w(0x2f), w(0x84),\
    w(0x53), w(0xd1), w(0x00), w(0xed), w(0x20), w(0xfc), w(0xb1), w(0x5b),\
    w(0x6a), w(0xcb), w(0xbe), w(0x39), w(0x4a), w(0x4c), w(0x58), w(0xcf),\
    w(0xd0), w(0xef), w(0xaa), w(0xfb), w(0x43), w(0x4d), w(0x33), w(0x85),\
    w(0x45), w(0xf9), w(0x02), w(0x7f), w(0x50), w(0x3c), w(0x9f), w(0xa8),\
    w(0x51), w(0xa3), w(0x40), w(0x8f), w(0x92), w(0x9d), w(0x38), w(0xf5),\
    w(0xbc), w(0xb6), w(0xda), w(0x21), w(0x10), w(0xff), w(0xf3), w(0xd2),\
    w(0xcd), w(0x0c), w(0x13), w(0xec), w(0x5f), w(0x97), w(0x44), w(0x17),\
    w(0xc4), w(0xa7), w(0x7e), w(0x3d), w(0x64), w(0x5d), w(0x19), w(0x73),\
    w(0x60), w(0x81), w(0x4f), w(0xdc), w(0x22), w(0x2a), w(0x90), w(0x88),\
    w(0x46), w(0xee), w(0xb8), w(0x14), w(0xde), w(0x5e), w(0x0b), w(0xdb),\
    w(0xe0), w(0x32), w(0x3a), w(0x0a), w(0x49), w(0x06), w(0x24), w(0x5c),\
    w(0xc2), w(0xd3), w(0xac), w(0x62), w(0x91), w(0x95), w(0xe4), w(0x79),\
    w(0xe7), w(0xc8), w(0x37), w(0x6d), w(0x8d), w(0xd5), w(0x4e), w(0xa9),\
    w(0x6c), w(0x56), w(0xf4), w(0xea), w(0x65), w(0x7a), w(0xae), w(0x08),\
    w(0xba), w(0x78), w(0x25), w(0x2e), w(0x1c), w(0xa6), w(0xb4), w(0xc6),\
    w(0xe8), w(0xdd), w(0x74), w(0x1f), w(0x4b), w(0xbd), w(0x8b), w(0x8a),\
    w(0x70), w(0x3e), w(0xb5), w(0x66), w(0x48), w(0x03), w(0xf6), w(0x0e),\
    w(0x61), w(0x35), w(0x57), w(0xb9), w(0x86), w(0xc1), w(0x1d), w(0x9e),\
    w(0xe1), w(0xf8), w(0x98), w(0x11), w(0x69), w(0xd9), w(0x8e), w(0x94),\
    w(0x9b), w(0x1e), w(0x87), w(0xe9), w(0xce), w(0x55), w(0x28), w(0xdf),\
    w(0x8c), w(0xa1), w(0x89), w(0x0d), w(0xbf), w(0xe6), w(0x42), w(0x68),\
    w(0x41), w(0x99), w(0x2d), w(0x0f), w(0xb0), w(0x54), w(0xbb), w(0x16) }

#define SAES_WPOLY           0x011b

#define saes_b2w(b0, b1, b2, b3) (((uint32_t)(b3) << 24) | \
    ((uint32_t)(b2) << 16) | ((uint32_t)(b1) << 8) | (b0))

#define saes_f2(x)   ((x<<1) ^ (((x>>7) & 1) * SAES_WPOLY))
#define saes_f3(x)   (saes_f2(x) ^ x)
#define saes_h0(x)   (x)

#define saes_u0(p)   saes_b2w(saes_f2(p),          p,          p, saes_f3(p))
#define saes_u1(p)   saes_b2w(saes_f3(p), saes_f2(p),          p,          p)
#define saes_u2(p)   saes_b2w(         p, saes_f3(p), saes_f2(p),          p)
#define saes_u3(p)   saes_b2w(         p,          p, saes_f3(p), saes_f2(p))

__device__ const uint32_t saes_table[4][256] = { saes_data(saes_u0), saes_data(saes_u1), saes_data(saes_u2), saes_data(saes_u3) };



typedef uint4 uint128m;
#define GPU_DEBUG
#define VERUS_KEY_SIZE 8832
#define VERUS_KEY_SIZE128 552
#define THREADS 64
#define INNERLOOP 16

#define AES2_EMU(s0, s1, rci) \
  aesenc(&s0, &rc[rci],sharedMemory1); \
  aesenc(&s1, &rc[rci + 1],sharedMemory1); \
  aesenc(&s0, &rc[rci + 2],sharedMemory1); \
  aesenc(&s1, &rc[rci + 3],sharedMemory1);

#define AES4(s0, s1, s2, s3, rci) \
  aesenc(&s0, &rc[rci],sharedMemory1); \
  aesenc(&s1, &rc[rci + 1],sharedMemory1); \
  aesenc(&s2, &rc[rci + 2],sharedMemory1); \
  aesenc(&s3, &rc[rci + 3],sharedMemory1); \
  aesenc(&s0, &rc[rci + 4], sharedMemory1); \
  aesenc(&s1, &rc[rci + 5], sharedMemory1); \
  aesenc(&s2, &rc[rci + 6], sharedMemory1); \
  aesenc(&s3, &rc[rci + 7], sharedMemory1);


#define AES4_LAST(s3, rci) \
  aesenc(&s3, &rc[rci + 2],sharedMemory1); \
  aesenc(&s3, &rc[rci + 6], sharedMemory1); \


#define TRUNCSTORE(out, s4) \
  *(uint32_t*)(out + 28) = s4.y;

#define MIX2_EMU(s0, s1) \
  tmp = _mm_unpacklo_epi32_emu(s0, s1); \
  s1 = _mm_unpackhi_epi32_emu(s0, s1); \
  s0 = tmp;

#define MIX4(s0, s1, s2, s3) \
  tmp  = _mm_unpacklo_epi32_emu(s0, s1); \
  s0 = _mm_unpackhi_epi32_emu(s0, s1); \
  s1 = _mm_unpacklo_epi32_emu(s2, s3); \
  s2 = _mm_unpackhi_epi32_emu(s2, s3); \
  s3 = _mm_unpacklo_epi32_emu(s0, s2); \
  s0 = _mm_unpackhi_epi32_emu(s0, s2); \
  s2 = _mm_unpackhi_epi32_emu(s1, tmp); \
  s1 = _mm_unpacklo_epi32_emu(s1, tmp);

#define MIX4_LASTBUT1(s0, s1, s2, s3) \
  tmp  = _mm_unpacklo_epi32_emu(s0, s1); \
  s1 = _mm_unpacklo_epi32_emu(s2, s3); \
  s2 = _mm_unpackhi_epi32_emu(s1, tmp); 


__host__ void verus_setBlock(uint8_t *blockf, uint32_t *pTargetIn, uint8_t *lkey, int thr_id);


__global__ void verus_gpu_hash(uint32_t threads, uint32_t startNonce, uint32_t *resNonce, uint128m * d_key_input, uint128m * d_mid, uint32_t *d_fix_r, uint32_t *d_fix_rex);
__global__ void verus_gpu_final(uint32_t threads, uint32_t startNonce, uint32_t *resNonce, uint128m * d_key_input, const  uint128m * d_mid);
__global__ void verus_extra_gpu_prepare(const uint32_t threads, uint128m * d_key_input);
__global__ void verus_extra_gpu_fix(const uint32_t threads, uint128m * d_key_input, uint32_t *d_fix_r, uint32_t *d_fix_rex);


static uint32_t *d_nonces[MAX_GPUS];
static uint32_t *d_fix_rand[MAX_GPUS];
static uint32_t *d_fix_randex[MAX_GPUS];
static uint4 *d_long_keys[MAX_GPUS];

static uint4 *d_mid[MAX_GPUS];

static hipStream_t streams[MAX_GPUS];
static uint8_t run[MAX_GPUS];

__device__ __constant__ uint128m vkey[VERUS_KEY_SIZE128];
__device__ __constant__ uint128m blockhash_half[4];
__device__ __constant__ uint32_t ptarget[8];

__host__
void verus_init(int thr_id, uint32_t throughput)
{
	//hipFuncSetCacheConfig(reinterpret_cast<const void*>(verus_gpu_hash), hipFuncCachePreferL1);
	CUDA_SAFE_CALL(hipMalloc(&d_nonces[thr_id], 1 * sizeof(uint32_t)));

	CUDA_SAFE_CALL(hipMalloc(&d_long_keys[thr_id], throughput * VERUS_KEY_SIZE));
	CUDA_SAFE_CALL(hipMalloc(&d_mid[thr_id], throughput * 16));
	CUDA_SAFE_CALL(hipMalloc(&d_fix_rand[thr_id], throughput * sizeof(uint32_t) * 32));
	CUDA_SAFE_CALL(hipMalloc(&d_fix_randex[thr_id], throughput * sizeof(uint32_t) * 32));

};

__host__
void verus_setBlock(uint8_t *blockf, uint32_t *pTargetIn, uint8_t *lkey, int thr_id, uint32_t throughput)
{
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(ptarget), (void**)pTargetIn, 8 * sizeof(uint32_t), 0, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(blockhash_half), (void**)blockf, 64 * sizeof(uint8_t), 0, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(vkey),(void**)lkey, VERUS_KEY_SIZE * sizeof(uint8_t), 0, hipMemcpyHostToDevice));
	dim3 grid2(throughput);
	verus_extra_gpu_prepare << <grid2, 128 >> > (0, d_long_keys[thr_id]); //setup global mem with lots of keys	

};
__host__
void verus_hash(int thr_id, uint32_t threads, uint32_t startNonce, uint32_t *resNonces)
{
	hipMemset(d_nonces[thr_id], 0xff, 1 * sizeof(uint32_t));
	const uint32_t threadsperblock = THREADS;
	const uint32_t threadsperblock2 = 256;

	dim3 grid((threads + threadsperblock - 1) / threadsperblock);
	dim3 grid3((threads + threadsperblock2 - 1) / threadsperblock2);
	dim3 grid2(threads);
	dim3 block(threadsperblock);


	if (run[thr_id] == 0) {
		hipStreamCreate(&streams[thr_id]);
		run[thr_id] = 1;
	}
//	verus_extra_gpu_prepare << <grid2, 128 >> > (0, d_long_keys[thr_id]); //setup global mem with lots of keys	
	verus_gpu_hash << <grid, block, 0, streams[thr_id] >> >(threads, startNonce, d_nonces[thr_id], d_long_keys[thr_id], d_mid[thr_id], d_fix_rand[thr_id], d_fix_randex[thr_id]);
	verus_gpu_final << <grid3, 256, 0, streams[thr_id] >> >(threads, startNonce, d_nonces[thr_id], d_long_keys[thr_id], d_mid[thr_id]);
	verus_extra_gpu_fix << <grid2, 32, 0, streams[thr_id] >> > (0, d_long_keys[thr_id], d_fix_rand[thr_id], d_fix_randex[thr_id]); //setup global mem with lots of keys	
	CUDA_SAFE_CALL(hipMemcpy(resNonces, d_nonces[thr_id], 1 * sizeof(uint32_t), hipMemcpyDeviceToHost));

};
__device__ __forceinline__
uint32_t xor3x(uint32_t a, uint32_t b, uint32_t c) {
	uint32_t result;
#if __CUDA_ARCH__ >= 500 && CUDA_VERSION >= 7050
	asm("lop3.b32 %0, %1, %2, %3, 0x96;" : "=r"(result) : "r"(a), "r"(b), "r"(c)); //0x96 = 0xF0 ^ 0xCC ^ 0xAA
#else
	result = a^b^c;
#endif
	return result;
}

__device__  __forceinline__  uint128m _mm_xor_si128_emu(uint128m a, uint128m b)
{
	uint128m result;
	asm("xor.b32 %0, %1, %2; // xor1" : "=r"(result.x) : "r"(a.x), "r"(b.x));
	asm("xor.b32 %0, %1, %2; // xor1" : "=r"(result.y) : "r"(a.y), "r"(b.y));
	asm("xor.b32 %0, %1, %2; // xor1" : "=r"(result.z) : "r"(a.z), "r"(b.z));
	asm("xor.b32 %0, %1, %2; // xor1" : "=r"(result.w) : "r"(a.w), "r"(b.w));
	return result;


}


__device__  __forceinline__  uint32_t XT4(uint32_t b)
{
	uint32_t tmp1,tmp2,tmp3;
	
	tmp1 = (b << 1) & 0xfefefefe;
	tmp2 = (b >> 7) & 0x1010101;
	asm("mul.lo.u32 %0, %1, 27; ": "=r"(tmp3) : "r"(tmp2));
	asm("xor.b32 %0, %1, %2; // xor1" : "=r"(tmp2) : "r"(tmp1), "r"(tmp3));
	
	return tmp2;
}

__device__  uint128m _mm_clmulepi64_si128_emu(uint128m ai, uint128m bi, int imm)
{
	uint64_t a = ((uint64_t*)&ai)[0]; // (0xffffffffull & ai.x) | ((0x00000000ffffffffull & ai.y) << 32);//+ (imm & 1));

	uint64_t b = ((uint64_t*)&bi)[1]; // (0xffffffffull & bi.z) | ((0x00000000ffffffffull & bi.w) << 32);
	
//	uint8_t  i; 
//	uint2 u[8];
	uint64_t r[2]; //uint128m r;
	//uint2 tmp;

if(__popcll(a) > __popcll(b)){

a=b;b= ((uint64_t*)&ai)[0];
}
r[0] =0; r[1] =0;

uint64_t w = a; int counter=0; int first;

while((first=__clzll(w)+1) !=65 ){
w <<=(first);
counter+=(first);

r[0] ^= b << (64 - counter);
r[1] ^= b >> ((counter));
};


 /*
//      XCHG(a,b);
	u[0].x = 0; //000 x b u[0].y = 0;
}
     
	u[1].x = bi.z; //001 x b u[1].y = bi.w; //001 x   
          
	u[2].x = u[1].x << 1; //010 x b
  u[2].y = __funnelshift_l(u[1].x, u[1].y, 1); //010 x b
     
	u[3].x = u[2].x ^ bi.z;  //011 x b
  u[3].y = u[2].y ^ bi.w;  //011 x b
   
	u[4].x = u[2].x << 1; //100 x b
  u[4].y = __funnelshift_l(u[2].x, u[2].y, 1); //010 x b
   
	u[5].x = u[4].x ^ bi.z;  //101 x b
  u[5].y = u[4].y ^ bi.w;  //101 x b
   
	u[6].x = u[3].x << 1; //110 x b
  u[6].y = __funnelshift_l(u[3].x, u[3].y, 1); //010 x b
   
	u[7].x = u[6].x ^ bi.z;  //111 x b
  u[7].y = u[6].y ^ bi.w;  
					  //Multiply
	r.x = u[a & 7].x; //first window only affects lower word
	r.y = u[a & 7].y;
	r.z = r.w = 0;
	//#pragma unroll
	for (i = 3; i < 31; i += 3) {
		tmp.x = u[a >> i & 7].x;
    tmp.y = u[a >> i & 7].y;
		r.x ^= (tmp.x << i) ;
		r.y ^= __funnelshift_l(tmp.x, tmp.y, i);
    r.z ^= (	tmp.y >> ( 32 - i));

	}
 
 	//#pragma unroll
	for (i = 33; i < 64; i += 3) {
		tmp.x = u[a >> i & 7].x;
    tmp.y = u[a >> i & 7].y;
		r.y ^= (tmp.x << (i - 32 ));
    r.z ^= __funnelshift_r(tmp.x, tmp.y, (64-i));
		r.w ^=  tmp.y >> (64 - i);
	}
 

	if ((bi.w ) & 0x80000000)
	{
		uint32_t t0 = LIMMY_R(ai.x, ai.y, 1);
		uint32_t t1 = ai.y >> 1;
		r.z ^= (t0 & 0xDB6DB6DB); //0, 21x 110
		r.w ^= (t1 & 0x36DB6DB6); //0x6DB6DB6DB6DB6DB6 -> 0x36DB6DB6DB6DB6DB after >>1
	}
	if ((bi.w ) &  0x40000000)
	{
		uint32_t t0 = LIMMY_R(ai.x, ai.y, 2);
		uint32_t t1 = ai.y >> 2;
		r.z ^= (t0 & 0x49249249); //0, 21x 100
		r.w ^= (t1 & 0x12492492); //0x4924924924924924 -> 0x1249249249249249 after >>2
	}
*/
	return ((uint128m*)&r)[0];
}

__device__  __forceinline__ uint128m _mm_clmulepi64_si128_emu2(uint128m ai)
{
	uint64_t a = ((uint64_t*)&ai)[1];

	//uint64_t b = 27 ;
	uint8_t  i; //window size s = 4,
				//uint64_t two_s = 16; //2^s
				//uint64_t smask = 15; //s 15 
	uint8_t u[8];
	uint128m r;
	uint64_t tmp;
	//Precomputation

	//#pragma unroll
	u[0] = 0;  //000 x b
	u[1] = 27;  //001 x b
	u[2] = 54; // u[1] << 1; //010 x b
	u[3] = 45;  //011 x b
	u[4] = 108; //100 x b
	u[5] = 119;  //101 x b
	u[6] = 90; //110 x b
	u[7] = 65;  //111 x b
					  //Multiply
	((uint64_t*)&r)[0] = u[a & 7]; //first window only affects lower word

	r.z = r.w = 0;
	//#pragma unroll
	for (i = 3; i < 64; i += 3) {
		tmp = u[a >> i & 7];
		r.x ^= (tmp << i) & 0xffffffff;
		r.y ^= ((tmp << i) & 0xffffffff00000000) >> 32;
		//	((uint64_t*)&r)[1] ^= tmp >> (64 - i);
		r.z ^= (tmp >> (64 - i)) & 0xffffffff;
		r.w ^= ((tmp >> (64 - i)) & 0xffffffff00000000) >> 32;
	}

	return r;
}

#define _mm_load_si128_emu(p) (*(uint128m*)(p));

#define _mm_cvtsi128_si64_emu(p) (((int64_t *)&p)[0]);

#define _mm_cvtsi128_si32_emu(p) (((int32_t *)&a)[0]);


__device__  void _mm_unpackboth_epi32_emu(uint128m &a, uint128m &b)
{
	uint64_t value;

	asm("mov.b64 %0, {%1, %2}; ": "=l"(value) : "r"(a.z), "r"(a.y));
	asm("mov.b64 {%0, %1}, %2; ": "=r"(a.y), "=r"(a.z) : "l"(value));

	asm("mov.b64 %0, {%1, %2}; ": "=l"(value) : "r"(b.x), "r"(a.y));
	asm("mov.b64 {%0, %1}, %2; ": "=r"(a.y), "=r"(b.x) : "l"(value));

	asm("mov.b64 %0, {%1, %2}; ": "=l"(value) : "r"(b.z), "r"(a.w));
	asm("mov.b64 {%0, %1}, %2; ": "=r"(a.w), "=r"(b.z) : "l"(value));
	
	asm("mov.b64 %0, {%1, %2}; ": "=l"(value) : "r"(b.y), "r"(a.w));
	asm("mov.b64 {%0, %1}, %2; ": "=r"(a.w), "=r"(b.y) : "l"(value));
}


__device__  __forceinline__ uint128m _mm_unpacklo_epi32_emu(uint128m a, uint128m b)
{

	//uint4 t;

//	t.x = a.x;
	a.z = a.y;
	a.y = b.x;
	a.w = b.y;
	return a;
}

__device__  __forceinline__ uint128m _mm_unpackhi_epi32_emu(uint128m a, uint128m b)
{

	//uint4 t;
	b.x = a.z;
	b.y = b.z;
	b.z = a.w;
	//t.w = b.w;

	return b;
}


__device__   __forceinline__ void aesenc(uint4 * __restrict__ ptr, const uint128m * __restrict__ key, uint32_t * __restrict__ t)
{
//#define XT(x) (((x) << 1) ^ (((x) >> 7) ? 0x1b : 0))

//#define XT4(x) ((((x) << 1) & 0xfefefefe) ^ ((((x) >> 31) & 1) ? 0x1b000000 : 0)^ ((((x) >> 23)&1) ? 0x001b0000 : 0)^ ((((x) >> 15)&1) ? 0x00001b00 : 0)^ ((((x) >> 7)&1) ? 0x0000001b : 0))
	uint32_t x0 = ptr[0].x;
	uint32_t x1 = ptr[0].y;
	uint32_t x2 = ptr[0].z;
	uint32_t x3 = ptr[0].w;

	uint32_t y0 = t[x0 & 0xff]; x0 >>= 8;
	uint32_t y1 = t[x1 & 0xff]; x1 >>= 8;
	uint32_t y2 = t[x2 & 0xff]; x2 >>= 8;
	uint32_t y3 = t[x3 & 0xff]; x3 >>= 8;
	t += 256;

	y0 ^= t[x1 & 0xff]; x1 >>= 8;
	y1 ^= t[x2 & 0xff]; x2 >>= 8;
	y2 ^= t[x3 & 0xff]; x3 >>= 8;
	y3 ^= t[x0 & 0xff]; x0 >>= 8;
	t += 256;

	y0 ^= t[x2 & 0xff]; x2 >>= 8;
	y1 ^= t[x3 & 0xff]; x3 >>= 8;
	y2 ^= t[x0 & 0xff]; x0 >>= 8;
	y3 ^= t[x1 & 0xff]; x1 >>= 8;
	t += 256;

	y0 ^= t[x3];
	y1 ^= t[x0];
	y2 ^= t[x1];
	y3 ^= t[x2];

	ptr[0].x = y0 ^ key[0].x;
	ptr[0].y = y1 ^ key[0].y;
	ptr[0].z = y2 ^ key[0].z;
	ptr[0].w = y3 ^ key[0].w;

}


__device__  __forceinline__ uint128m _mm_cvtsi32_si128_emu(uint32_t lo)
{
	uint128m result = { 0 };
	result.x= lo;

	return result;
}
__device__  __forceinline__ uint128m _mm_cvtsi64_si128_emu(uint64_t lo)
{
	uint128m result = {0};
	((uint64_t *)&result)[0] = lo;
	//((uint64_t *)&result)[1] = 0;
	return result;
}
__device__  __forceinline__ uint128m _mm_set_epi64x_emu(uint64_t hi, uint64_t lo)
{
	uint128m result;
	((uint64_t *)&result)[0] = lo;
	((uint64_t *)&result)[1] = hi;
	return result;
}
__device__ __forceinline__ uint128m _mm_shuffle_epi8_emu(uint128m b)
{
	uint128m result = { 0 };
	const uint128m M = { 0x2d361b00,0x415a776c,0xf5eec3d8,0x9982afb4 };
	const uint128m Q = { 0x80808080, 0x80808080, 0x80808080, 0x80808080 };
	const uint128m W = b & Q;


#pragma unroll
	for (int i = 0; i < 8; i++)
	{
		if (!((uint8_t *)&W)[i])
		{
		((uint8_t *)&result)[i] = ((uint8_t *)&M)[((uint8_t *)&b)[i] & 0xf];
		}
	}

	return result;
}



__device__  __forceinline__ uint128m _mm_srli_si128_emu(uint128m input, int imm8)
{
	//we can cheat here as its an 8 byte shift just copy the 64bits
	uint128m temp;
	((uint64_t*)&temp)[0] = ((uint64_t*)&input)[1];
	((uint64_t*)&temp)[1] = 0;


	return temp;
}



__device__  uint128m _mm_mulhrs_epi16_emu(uint128m _a, uint128m _b)
{
	int16_t result[8];

	int32_t po;
	int16_t *a = (int16_t*)&_a, *b = (int16_t*)&_b;
#pragma nounroll
	for (int i = 0; i < 8; i++)
	{
		asm("mad.lo.s32 %0, %1, %2, 16384; ": "=r"(po) : "r"((int32_t)a[i]), "r"((int32_t)b[i]));

		result[i] = po >> 15;
	//	result[i] = (int16_t)((((int32_t)(a[i]) * (int32_t)(b[i])) + 0x4000) >> 15);
	
	}
	return *(uint128m *)result;
}


__device__  __forceinline__ void case_0(uint128m &prand, uint128m &prandex, const  uint128m *pbuf,
	uint64_t selector, uint128m &acc)
{
	const uint128m temp1 = prandex;

	const uint128m temp2 = _mm_load_si128_emu(pbuf - (((selector & 1) << 1) - 1));


	const uint128m add1 = _mm_xor_si128_emu(temp1, temp2);

	const uint128m clprod1 = _mm_clmulepi64_si128_emu(add1, add1, 0x10);
	acc = _mm_xor_si128_emu(clprod1, acc);

	const uint128m tempa1 = _mm_mulhrs_epi16_emu(acc, temp1);
	const uint128m tempa2 = _mm_xor_si128_emu(tempa1, temp1);

	const uint128m temp12 = prand;
	prand = tempa2;


	const uint128m temp22 = _mm_load_si128_emu(pbuf);
	const uint128m add12 = _mm_xor_si128_emu(temp12, temp22);
	const uint128m clprod12 = _mm_clmulepi64_si128_emu(add12, add12, 0x10);
	acc = _mm_xor_si128_emu(clprod12, acc);

	const uint128m tempb1 = _mm_mulhrs_epi16_emu(acc, temp12);
	const uint128m tempb2 = _mm_xor_si128_emu(tempb1, temp12);
	prandex = tempb2;

}

__device__  __forceinline__ void case_4(uint128m &prand, uint128m &prandex, const  uint128m *pbuf,
	uint64_t selector, uint128m &acc)
{
	const uint128m temp1 = prand;
	const uint128m temp2 = _mm_load_si128_emu(pbuf);
	const uint128m add1 = _mm_xor_si128_emu(temp1, temp2);
	const uint128m clprod1 = _mm_clmulepi64_si128_emu(add1, add1, 0x10);
	acc = _mm_xor_si128_emu(clprod1, acc);
	const uint128m clprod2 = _mm_clmulepi64_si128_emu(temp2, temp2, 0x10);
	acc = _mm_xor_si128_emu(clprod2, acc);

	const uint128m tempa1 = _mm_mulhrs_epi16_emu(acc, temp1);
	const uint128m tempa2 = _mm_xor_si128_emu(tempa1, temp1);

	const uint128m temp12 = prandex;
	prandex = tempa2;

	const uint128m temp22 = _mm_load_si128_emu(pbuf - (((selector & 1) << 1) - 1));
	const uint128m add12 = _mm_xor_si128_emu(temp12, temp22);
	acc = _mm_xor_si128_emu(add12, acc);

	const uint128m tempb1 = _mm_mulhrs_epi16_emu(acc, temp12);
	const uint128m tempb2 = _mm_xor_si128_emu(tempb1, temp12);
	prand = tempb2;
}

__device__  void case_8(uint128m &prand, uint128m &prandex, const  uint128m *pbuf,
	uint64_t selector, uint128m &acc)
{
	const uint128m temp1 = prandex;
	const uint128m temp2 = _mm_load_si128_emu(pbuf);
	const uint128m add1 = _mm_xor_si128_emu(temp1, temp2);
	acc = _mm_xor_si128_emu(add1, acc);

	const uint128m tempa1 = _mm_mulhrs_epi16_emu(acc, temp1);
	const uint128m tempa2 = _mm_xor_si128_emu(tempa1, temp1);

	const uint128m temp12 = prand;
	prand = tempa2;

	const uint128m temp22 = _mm_load_si128_emu(pbuf - (((selector & 1) << 1) - 1));
	const uint128m add12 = _mm_xor_si128_emu(temp12, temp22);
	const uint128m clprod12 = _mm_clmulepi64_si128_emu(add12, add12, 0x10);
	acc = _mm_xor_si128_emu(clprod12, acc);
	const uint128m clprod22 = _mm_clmulepi64_si128_emu(temp22, temp22, 0x10);
	acc = _mm_xor_si128_emu(clprod22, acc);

	const uint128m tempb1 = _mm_mulhrs_epi16_emu(acc, temp12);
	const uint128m tempb2 = _mm_xor_si128_emu(tempb1, temp12);
	prandex = tempb2;
}

__device__ void case_0c(uint128m &prand, uint128m &prandex, const  uint128m *pbuf,
	uint64_t selector, uint128m &acc)
{
	const uint128m temp1 = prand;
	const uint128m temp2 = _mm_load_si128_emu(pbuf - (((selector & 1) << 1) - 1));
	const uint128m add1 = _mm_xor_si128_emu(temp1, temp2);

	// cannot be zero here
	const int32_t divisor = ((uint32_t*)&selector)[0];

	acc = _mm_xor_si128_emu(add1, acc);

	int64_t dividend = _mm_cvtsi128_si64_emu(acc);
	int64_t tmpmod = dividend % divisor;
	const uint128m modulo = _mm_cvtsi32_si128_emu(tmpmod);
	acc = _mm_xor_si128_emu(modulo, acc);

	const uint128m tempa1 = _mm_mulhrs_epi16_emu(acc, temp1);
	const uint128m tempa2 = _mm_xor_si128_emu(tempa1, temp1);
	dividend &= 1;
	if (dividend)
	{
		const uint128m temp12 = prandex;
		prandex = tempa2;

		const uint128m temp22 = _mm_load_si128_emu(pbuf);
		const uint128m add12 = _mm_xor_si128_emu(temp12, temp22);
		const uint128m clprod12 = _mm_clmulepi64_si128_emu(add12, add12, 0x10);
		acc = _mm_xor_si128_emu(clprod12, acc);
		const uint128m clprod22 = _mm_clmulepi64_si128_emu(temp22, temp22, 0x10);
		acc = _mm_xor_si128_emu(clprod22, acc);

		const uint128m tempb1 = _mm_mulhrs_epi16_emu(acc, temp12);
		const uint128m tempb2 = _mm_xor_si128_emu(tempb1, temp12);
		prand = tempb2;
	}
	else
	{
		const uint128m tempb3 = prandex;
		prandex = tempa2;
		prand = tempb3;
	}
}
__device__ void case_10(uint128m &prand, uint128m &prandex, const  uint128m *pbuf,
	uint64_t selector, uint128m &acc, uint128m *randomsource, uint32_t prand_idx, uint32_t *sharedMemory1)
{			// a few AES operations
			uint128m rc[12];

			rc[0] = prand; 

	//uint128m *rc = &randomsource[prand_idx];
			rc[1] = randomsource[prand_idx + 1];
		rc[2] = randomsource[prand_idx + 2];
	rc[3] = randomsource[prand_idx + 3];
	rc[4] = randomsource[prand_idx + 4];
	rc[5] = randomsource[prand_idx + 5];
	rc[6] = randomsource[prand_idx + 6];
	rc[7] = randomsource[prand_idx + 7];
	rc[8] = randomsource[prand_idx + 8];
	rc[9] = randomsource[prand_idx + 9];
	rc[10] = randomsource[prand_idx + 10];
	rc[11] = randomsource[prand_idx + 11];
	uint128m tmp;

	uint128m temp1 = _mm_load_si128_emu(pbuf - (((selector & 1) << 1) - 1));
	uint128m temp2 = _mm_load_si128_emu(pbuf);

	AES2_EMU(temp1, temp2, 0);
	MIX2_EMU(temp1, temp2);


	AES2_EMU(temp1, temp2, 4);
	MIX2_EMU(temp1, temp2);

	AES2_EMU(temp1, temp2, 8);
	MIX2_EMU(temp1, temp2);


	acc = _mm_xor_si128_emu(temp1, acc);
	acc = _mm_xor_si128_emu(temp2, acc);

	const uint128m tempa1 = prand;
	const uint128m tempa2 = _mm_mulhrs_epi16_emu(acc, tempa1);
	const uint128m tempa3 = _mm_xor_si128_emu(tempa1, tempa2);

	const uint128m tempa4 = prandex;
	prandex = tempa3;
	prand = tempa4;
}
__device__ void case_14(uint128m &prand, uint128m &prandex, const  uint128m *pbuf,
	uint64_t selector, uint128m &acc, uint128m *randomsource, uint32_t prand_idx, uint32_t *sharedMemory1)
{
	// we'll just call this one the monkins loop, inspired by Chris
	const uint128m *buftmp = pbuf - (((selector & 1) << 1) - 1);
	uint128m tmp; // used by MIX2

	uint64_t rounds = selector >> 61; // loop randomly between 1 and 8 times
	uint128m *rc = &randomsource[prand_idx];


	uint64_t aesround = 0;
	uint128m onekey;
	uint64_t loop_c;

	for (int i = 0; i<8;i++)
	{
		if (rounds <= 8) {
			loop_c = selector & (0x10000000 << rounds);
			if (loop_c)
			{
				onekey = _mm_load_si128_emu(rc++);
				const uint128m temp2 = _mm_load_si128_emu(rounds & 1 ? pbuf : buftmp);
				const uint128m add1 = _mm_xor_si128_emu(onekey, temp2);
				const uint128m clprod1 = _mm_clmulepi64_si128_emu(add1, add1, 0x10);
				acc = _mm_xor_si128_emu(clprod1, acc);
			}
			else
			{
				onekey = _mm_load_si128_emu(rc++);
				uint128m temp2 = _mm_load_si128_emu(rounds & 1 ? buftmp : pbuf);

				const uint64_t roundidx = aesround++ << 2;
				AES2_EMU(onekey, temp2, roundidx);

				MIX2_EMU(onekey, temp2);

				acc = _mm_xor_si128_emu(onekey, acc);
				acc = _mm_xor_si128_emu(temp2, acc);

			}
		}
 (rounds--);
	} 

	const uint128m tempa1 = (prand);
	const uint128m tempa2 = _mm_mulhrs_epi16_emu(acc, tempa1);
	const uint128m tempa3 = _mm_xor_si128_emu(tempa1, tempa2);

	const uint128m tempa4 = (prandex);
	prandex = tempa3;
	prand = tempa4;
}

__device__ void __forceinline__  case_18(uint128m &prand, uint128m &prandex, const  uint128m *pbuf,
	uint64_t selector, uint128m &acc)
{
	const uint128m temp1 = _mm_load_si128_emu(pbuf - (((selector & 1) << 1) - 1));
	const uint128m temp2 = (prand);
	const uint128m add1 = _mm_xor_si128_emu(temp1, temp2);
	const uint128m clprod1 = _mm_clmulepi64_si128_emu(add1, add1, 0x10);
	acc = _mm_xor_si128_emu(clprod1, acc);

	const uint128m tempa1 = _mm_mulhrs_epi16_emu(acc, temp2);
	const uint128m tempa2 = _mm_xor_si128_emu(tempa1, temp2);

	const uint128m tempb3 = (prandex);
	prandex = tempa2;
	prand = tempb3;
}

__device__  __forceinline__ void case_1c(uint128m &prand, uint128m &prandex, const  uint128m *pbuf,
	uint64_t selector, uint128m &acc)
{
	const uint128m temp1 = _mm_load_si128_emu(pbuf);
	const uint128m temp2 = (prandex);
	const uint128m add1 = _mm_xor_si128_emu(temp1, temp2);
	const uint128m clprod1 = _mm_clmulepi64_si128_emu(add1, add1, 0x10);
	acc = _mm_xor_si128_emu(clprod1, acc);


	const uint128m tempa1 = _mm_mulhrs_epi16_emu(acc, temp2);
	const uint128m tempa2 = _mm_xor_si128_emu(tempa1, temp2);
	const uint128m tempa3 = (prand);


	prand = tempa2;

	acc = _mm_xor_si128_emu(tempa3, acc);

	const uint128m tempb1 = _mm_mulhrs_epi16_emu(acc, tempa3);
	const uint128m tempb2 = _mm_xor_si128_emu(tempb1, tempa3);
	prandex = tempb2;
}



__device__ __forceinline__ uint128m __verusclmulwithoutreduction64alignedrepeatgpu(uint128m * __restrict__ randomsource, const  uint128m *  __restrict__  buf ,
	 uint32_t *  __restrict__ sharedMemory1, uint32_t *  __restrict__ d_fix_r, uint32_t *  __restrict__ d_fix_rex)
{
    uint128m const *pbuf;
	//keyMask >>= 4;
	uint128m acc = vkey[513];
	
	
	// divide key mask by 32 from bytes to uint128m
	
	uint16_t prand_idx, prandex_idx;
	uint64_t selector;
	uint128m prand;
	uint128m prandex;
	prand_idx = ((acc.x >> 5) & 511);
	prandex_idx = ((acc.y) & 511);

	prand = vkey[prand_idx];
	prandex = vkey[prandex_idx];
//#pragma unroll
	for (uint8_t i = 0; i < 32; i++)
	{
		
		selector = _mm_cvtsi128_si64_emu(acc);
		if (i > 0) {
		prand_idx = ((acc.x >> 5) & 511);
		prandex_idx = ((acc.y) & 511);

		
		// get two random locations in the key, which will be mutated and swapped
			prand = randomsource[prand_idx];
			prandex = randomsource[prandex_idx];
			
		}

		pbuf = buf + (acc.x & 3);
		uint8_t case_v;
		case_v = selector &  0x1cu;

		
		if(case_v == 0)
		{
			case_0(prand, prandex, pbuf, selector, acc);
		}
		else if (case_v == 4)
		{
			case_4(prand, prandex, pbuf, selector, acc);
		}
		else if (case_v == 8)
		{
			case_8(prand, prandex, pbuf, selector, acc);
			
		}
		else if (case_v == 0xc)
		{
			case_0c(prand, prandex, pbuf, selector, acc);

		}
			
		else if (case_v == 0x10)
		{
			case_10(prand, prandex, pbuf, selector, acc, randomsource, prand_idx, sharedMemory1);


		}
		
		else if (case_v == 0x14)
		{
			case_14(prand, prandex, pbuf, selector, acc, randomsource, prand_idx, sharedMemory1);

		}


		else if(case_v == 0x18)
		{
			case_18(prand, prandex, pbuf, selector, acc);
			
		}
		else 
		{
			case_1c(prand, prandex, pbuf, selector, acc);
		}	
		d_fix_r[i] = prand_idx;
		d_fix_rex[i] = prandex_idx;
		 randomsource[prand_idx] = prand;
		 randomsource[prandex_idx] = prandex;
		 
	}

	return acc;
}


__device__   __forceinline__  uint32_t haraka512_port_keyed2222(uint128m * __restrict__  in, uint128m * __restrict__  rc, uint32_t * __restrict__  sharedMemory1)
{
	uint128m s1,s2,s3,s4, tmp;

	s1 = in[0];
	s2 = in[1];
	s3 = in[2];
	s4 = in[3];

	AES4(s1, s2, s3, s4, 0);
	MIX4(s1, s2, s3, s4);

	AES4(s1, s2, s3, s4, 8);
	MIX4(s1, s2, s3, s4);

	AES4(s1, s2, s3, s4, 16);
	MIX4(s1, s2, s3, s4);

	AES4(s1, s2, s3, s4, 24);
	MIX4_LASTBUT1(s1, s2, s3, s4);


	AES4_LAST(s3, 32);

	return s3.z ^ in[3].y;

}

__device__   __forceinline__ uint64_t precompReduction64(uint128m A) {


	//static const uint128m M = { 0x2d361b00,0x415a776c,0xf5eec3d8,0x9982afb4 };
	// const uint128m tmp = { 27 };
	// A.z = 0;
	//tmp.x = 27u;
	uint128m Q2 = _mm_clmulepi64_si128_emu2(A);
	uint128m Q3 = _mm_shuffle_epi8_emu(_mm_srli_si128_emu(Q2, 8));

	//uint128m Q4 = _mm_xor_si128_emu(Q2, A);
	uint128m final;
	final.x = xor3(A.x, Q2.x, Q3.x);
	final.y = xor3(A.y, Q2.y, Q3.y);

	return _mm_cvtsi128_si64_emu(final);/// WARNING: HIGH 64 BITS SHOULD BE ASSUMED TO CONTAIN GARBAGE
}



__global__ __launch_bounds__(THREADS, 1)
void verus_gpu_hash(const uint32_t threads, const uint32_t startNonce, uint32_t * __restrict__ resNonce,
	uint128m * __restrict__ d_key_input, uint128m * __restrict__ d_mid, uint32_t * __restrict__  d_fix_r, uint32_t *__restrict__  d_fix_rex)
{
	const uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	uint128m mid; // , biddy[VERUS_KEY_SIZE128];
	uint128m s[4];

	const uint32_t nounce = startNonce + thread;

	__shared__ uint32_t sharedMemory1[4][256];
	__shared__ uint32_t sharedrand[32 * THREADS];
	__shared__ uint32_t sharedrandex[32 * THREADS];

	//uint32_t save_rand[32] = { 0 };
	//uint32_t save_randex[32] = { 0 };

	s[0] = blockhash_half[0];
	s[1] = blockhash_half[1];
	s[2] = blockhash_half[2];
	s[3] = blockhash_half[3];


	sharedMemory1[0][threadIdx.x] = saes_table[0][threadIdx.x];// copy sbox to shared mem
	sharedMemory1[0][threadIdx.x + 64] = saes_table[0][threadIdx.x + 64];// copy sbox to shared mem
	sharedMemory1[0][threadIdx.x + 128] = saes_table[0][threadIdx.x + 128];// copy sbox to shared mem
	sharedMemory1[0][threadIdx.x + 192] = saes_table[0][threadIdx.x + 192];// copy sbox to shared mem

	sharedMemory1[1][threadIdx.x] = saes_table[1][threadIdx.x];// copy sbox to shared mem
	sharedMemory1[1][threadIdx.x + 64] = saes_table[1][threadIdx.x + 64];// copy sbox to shared mem
	sharedMemory1[1][threadIdx.x + 128] = saes_table[1][threadIdx.x + 128];// copy sbox to shared mem
	sharedMemory1[1][threadIdx.x + 192] = saes_table[1][threadIdx.x + 192];// copy sbox to shared mem

	sharedMemory1[2][threadIdx.x] = saes_table[2][threadIdx.x];// copy sbox to shared mem
	sharedMemory1[2][threadIdx.x + 64] = saes_table[2][threadIdx.x + 64];// copy sbox to shared mem
	sharedMemory1[2][threadIdx.x + 128] = saes_table[2][threadIdx.x + 128];// copy sbox to shared mem
	sharedMemory1[2][threadIdx.x + 192] = saes_table[2][threadIdx.x + 192];// copy sbox to shared mem

	sharedMemory1[3][threadIdx.x] = saes_table[3][threadIdx.x];// copy sbox to shared mem
	sharedMemory1[3][threadIdx.x + 64] = saes_table[3][threadIdx.x + 64];// copy sbox to shared mem
	sharedMemory1[3][threadIdx.x + 128] = saes_table[3][threadIdx.x + 128];// copy sbox to shared mem
	sharedMemory1[3][threadIdx.x + 192] = saes_table[3][threadIdx.x + 192];// copy sbox to shared mem

	__syncthreads();
	s[2].x = nounce;

	

	mid = __verusclmulwithoutreduction64alignedrepeatgpu(&d_key_input[VERUS_KEY_SIZE128 * thread], s, sharedMemory1[0], sharedrand + (threadIdx.x * 32), sharedrandex + (threadIdx.x * 32));
	mid.x  ^= 0x00010000;
	d_mid[thread] = mid;

#pragma unroll
	for (int i = 0; i < 32; i++)
	{
		d_fix_r[(thread * 32) + i] = sharedrand[(threadIdx.x * 32)+i];
		d_fix_rex[(thread * 32) + i] = sharedrandex[(threadIdx.x * 32) + i];
	}
}
	__global__ __launch_bounds__(256, 1)
		void verus_gpu_final(const uint32_t threads, const uint32_t startNonce, uint32_t * __restrict__ resNonce,
			uint128m * __restrict__  d_key_input, const uint128m * __restrict__ d_mid)
	{
		const uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
		uint64_t acc = precompReduction64(d_mid[thread]);;
		//uint128m wizz = d_mid[thread];

		const uint32_t nounce = startNonce + thread;
		uint32_t hash;

		uint128m s[4];
		__shared__ uint32_t sharedMemory1[4][256];
	//	sharedMemory1[threadIdx.x] = sbox[threadIdx.x];// copy sbox to shared mem
		sharedMemory1[0][threadIdx.x] = saes_table[0][threadIdx.x];// copy sbox to shared mem

		sharedMemory1[1][threadIdx.x] = saes_table[1][threadIdx.x];// copy sbox to shared mem

		sharedMemory1[2][threadIdx.x] = saes_table[2][threadIdx.x];// copy sbox to shared mem


		sharedMemory1[3][threadIdx.x] = saes_table[3][threadIdx.x];// copy sbox to shared mem



		s[0] = blockhash_half[0];
		s[1] = blockhash_half[1];
		s[2] = blockhash_half[2];
		s[3] = blockhash_half[3];
		__syncthreads();

	((uint32_t *)&s)[8] = nounce;
	memcpy(((uint8_t*)&s) + 47, &acc, 8);
	memcpy(((uint8_t*)&s) + 55, &acc, 8);
	memcpy(((uint8_t*)&s) + 63, &acc, 1);
	//uint64_t mask = 8191 >> 4;
	acc &= 511;
	
	//haraka512_port_keyed((unsigned char*)hash, (const unsigned char*)s, (const unsigned char*)(biddy + mask), sharedMemory1, nounce);
	hash = haraka512_port_keyed2222(s, (&d_key_input[VERUS_KEY_SIZE128 * thread] + acc), sharedMemory1[0]);
	if (hash < ptarget[7]) { 
		
		resNonce[0] = nounce;

	}


};

__global__ __launch_bounds__(128, 1)
void verus_extra_gpu_prepare(const uint32_t threads, uint128m * d_key_input)
{

	d_key_input[(blockIdx.x * VERUS_KEY_SIZE128) + threadIdx.x] = vkey[threadIdx.x];
	d_key_input[(blockIdx.x * VERUS_KEY_SIZE128) + threadIdx.x + 128 ] = vkey[threadIdx.x + 128];
	d_key_input[(blockIdx.x * VERUS_KEY_SIZE128) + threadIdx.x + 256 ] = vkey[threadIdx.x + 256];
	d_key_input[(blockIdx.x * VERUS_KEY_SIZE128) + threadIdx.x + 384 ] = vkey[threadIdx.x + 384];
	if (threadIdx.x < 40)
		d_key_input[(blockIdx.x * VERUS_KEY_SIZE128) + threadIdx.x + 512 ] = vkey[threadIdx.x + 512];

}

__global__ __launch_bounds__(32, 1)
void verus_extra_gpu_fix(const uint32_t threads, uint128m * __restrict__ d_key_input, uint32_t *d_fix_r, uint32_t *d_fix_rex)
{
	
	d_key_input[(blockIdx.x * VERUS_KEY_SIZE128) + d_fix_r[(blockIdx.x * 32) + threadIdx.x]] = vkey[d_fix_r[(blockIdx.x * 32) +threadIdx.x]];
	d_key_input[(blockIdx.x * VERUS_KEY_SIZE128) + d_fix_rex[(blockIdx.x * 32) + threadIdx.x]] = vkey[d_fix_rex[(blockIdx.x * 32) + threadIdx.x]];
	
}
