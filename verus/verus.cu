#include "hip/hip_runtime.h"
#include <miner.h>

#include <cuda_helper.h>

typedef uint4 uint128m;
#define GPU_DEBUG
#define VERUS_KEY_SIZE 8832
#define VERUS_KEY_SIZE128 552
#define THREADS 64
#define INNERLOOP 16

#define AES2_EMU(s0, s1, rci) \
  aesenc((unsigned char *)&s0, &rc[rci],sharedMemory1); \
  aesenc((unsigned char *)&s1, &rc[rci + 1],sharedMemory1); \
  aesenc((unsigned char *)&s0, &rc[rci + 2],sharedMemory1); \
  aesenc((unsigned char *)&s1, &rc[rci + 3],sharedMemory1);

#define AES4(s0, s1, s2, s3, rci) \
  aesenc((unsigned char *)&s0, &rc[rci],sharedMemory1); \
  aesenc((unsigned char *)&s1, &rc[rci + 1],sharedMemory1); \
  aesenc((unsigned char *)&s2, &rc[rci + 2],sharedMemory1); \
  aesenc((unsigned char *)&s3, &rc[rci + 3],sharedMemory1); \
  aesenc((unsigned char *)&s0, &rc[rci + 4], sharedMemory1); \
  aesenc((unsigned char *)&s1, &rc[rci + 5], sharedMemory1); \
  aesenc((unsigned char *)&s2, &rc[rci + 6], sharedMemory1); \
  aesenc((unsigned char *)&s3, &rc[rci + 7], sharedMemory1);


#define AES4_LAST(s3, rci) \
  aesenc((unsigned char *)&s3, &rc[rci + 2],sharedMemory1); \
  aesenc((unsigned char *)&s3, &rc[rci + 6], sharedMemory1); \


#define TRUNCSTORE(out, s4) \
  *(uint32_t*)(out + 28) = s4.y;

#define MIX2_EMU(s0, s1) \
  tmp = _mm_unpacklo_epi32_emu(s0, s1); \
  s1 = _mm_unpackhi_epi32_emu(s0, s1); \
  s0 = tmp;

#define MIX4(s0, s1, s2, s3) \
  tmp  = _mm_unpacklo_epi32_emu(s0, s1); \
  s0 = _mm_unpackhi_epi32_emu(s0, s1); \
  s1 = _mm_unpacklo_epi32_emu(s2, s3); \
  s2 = _mm_unpackhi_epi32_emu(s2, s3); \
  s3 = _mm_unpacklo_epi32_emu(s0, s2); \
  s0 = _mm_unpackhi_epi32_emu(s0, s2); \
  s2 = _mm_unpackhi_epi32_emu(s1, tmp); \
  s1 = _mm_unpacklo_epi32_emu(s1, tmp);

#define MIX4_LASTBUT1(s0, s1, s2, s3) \
  tmp  = _mm_unpacklo_epi32_emu(s0, s1); \
  s1 = _mm_unpacklo_epi32_emu(s2, s3); \
  s2 = _mm_unpackhi_epi32_emu(s1, tmp); 


__host__ void verus_setBlock(uint8_t *blockf, uint32_t *pTargetIn, uint8_t *lkey, int thr_id);


__device__ const uint32_t sbox[] = {
	0x7b777c63, 0xc56f6bf2, 0x2b670130, 0x76abd7fe, 0x7dc982ca, 0xf04759fa, 0xafa2d4ad, 0xc072a49c, 0x2693fdb7, 0xccf73f36, 0xf1e5a534, 0x1531d871, 0xc323c704, 0x9a059618, 0xe2801207, 0x75b227eb, 0x1a2c8309, 0xa05a6e1b, 0xb3d63b52, 0x842fe329, 0xed00d153, 0x5bb1fc20, 0x39becb6a, 0xcf584c4a, 0xfbaaefd0, 0x85334d43, 0x7f02f945, 0xa89f3c50, 0x8f40a351, 0xf5389d92, 0x21dab6bc, 0xd2f3ff10, 0xec130ccd, 0x1744975f, 0x3d7ea7c4, 0x73195d64, 0xdc4f8160, 0x88902a22, 0x14b8ee46, 0xdb0b5ede, 0x0a3a32e0, 0x5c240649, 0x62acd3c2, 0x79e49591, 0x6d37c8e7, 0xa94ed58d, 0xeaf4566c, 0x08ae7a65, 0x2e2578ba, 0xc6b4a61c, 0x1f74dde8, 0x8a8bbd4b, 0x66b53e70, 0x0ef60348, 0xb9573561, 0x9e1dc186, 0x1198f8e1, 0x948ed969, 0xe9871e9b, 0xdf2855ce, 0x0d89a18c, 0x6842e6bf, 0x0f2d9941, 0x16bb54b0,
	0x7b777c63, 0xc56f6bf2, 0x2b670130, 0x76abd7fe, 0x7dc982ca, 0xf04759fa, 0xafa2d4ad, 0xc072a49c, 0x2693fdb7, 0xccf73f36, 0xf1e5a534, 0x1531d871, 0xc323c704, 0x9a059618, 0xe2801207, 0x75b227eb, 0x1a2c8309, 0xa05a6e1b, 0xb3d63b52, 0x842fe329, 0xed00d153, 0x5bb1fc20, 0x39becb6a, 0xcf584c4a, 0xfbaaefd0, 0x85334d43, 0x7f02f945, 0xa89f3c50, 0x8f40a351, 0xf5389d92, 0x21dab6bc, 0xd2f3ff10, 0xec130ccd, 0x1744975f, 0x3d7ea7c4, 0x73195d64, 0xdc4f8160, 0x88902a22, 0x14b8ee46, 0xdb0b5ede, 0x0a3a32e0, 0x5c240649, 0x62acd3c2, 0x79e49591, 0x6d37c8e7, 0xa94ed58d, 0xeaf4566c, 0x08ae7a65, 0x2e2578ba, 0xc6b4a61c, 0x1f74dde8, 0x8a8bbd4b, 0x66b53e70, 0x0ef60348, 0xb9573561, 0x9e1dc186, 0x1198f8e1, 0x948ed969, 0xe9871e9b, 0xdf2855ce, 0x0d89a18c, 0x6842e6bf, 0x0f2d9941, 0x16bb54b0,
	0x7b777c63, 0xc56f6bf2, 0x2b670130, 0x76abd7fe, 0x7dc982ca, 0xf04759fa, 0xafa2d4ad, 0xc072a49c, 0x2693fdb7, 0xccf73f36, 0xf1e5a534, 0x1531d871, 0xc323c704, 0x9a059618, 0xe2801207, 0x75b227eb, 0x1a2c8309, 0xa05a6e1b, 0xb3d63b52, 0x842fe329, 0xed00d153, 0x5bb1fc20, 0x39becb6a, 0xcf584c4a, 0xfbaaefd0, 0x85334d43, 0x7f02f945, 0xa89f3c50, 0x8f40a351, 0xf5389d92, 0x21dab6bc, 0xd2f3ff10, 0xec130ccd, 0x1744975f, 0x3d7ea7c4, 0x73195d64, 0xdc4f8160, 0x88902a22, 0x14b8ee46, 0xdb0b5ede, 0x0a3a32e0, 0x5c240649, 0x62acd3c2, 0x79e49591, 0x6d37c8e7, 0xa94ed58d, 0xeaf4566c, 0x08ae7a65, 0x2e2578ba, 0xc6b4a61c, 0x1f74dde8, 0x8a8bbd4b, 0x66b53e70, 0x0ef60348, 0xb9573561, 0x9e1dc186, 0x1198f8e1, 0x948ed969, 0xe9871e9b, 0xdf2855ce, 0x0d89a18c, 0x6842e6bf, 0x0f2d9941, 0x16bb54b0,
	0x7b777c63, 0xc56f6bf2, 0x2b670130, 0x76abd7fe, 0x7dc982ca, 0xf04759fa, 0xafa2d4ad, 0xc072a49c, 0x2693fdb7, 0xccf73f36, 0xf1e5a534, 0x1531d871, 0xc323c704, 0x9a059618, 0xe2801207, 0x75b227eb, 0x1a2c8309, 0xa05a6e1b, 0xb3d63b52, 0x842fe329, 0xed00d153, 0x5bb1fc20, 0x39becb6a, 0xcf584c4a, 0xfbaaefd0, 0x85334d43, 0x7f02f945, 0xa89f3c50, 0x8f40a351, 0xf5389d92, 0x21dab6bc, 0xd2f3ff10, 0xec130ccd, 0x1744975f, 0x3d7ea7c4, 0x73195d64, 0xdc4f8160, 0x88902a22, 0x14b8ee46, 0xdb0b5ede, 0x0a3a32e0, 0x5c240649, 0x62acd3c2, 0x79e49591, 0x6d37c8e7, 0xa94ed58d, 0xeaf4566c, 0x08ae7a65, 0x2e2578ba, 0xc6b4a61c, 0x1f74dde8, 0x8a8bbd4b, 0x66b53e70, 0x0ef60348, 0xb9573561, 0x9e1dc186, 0x1198f8e1, 0x948ed969, 0xe9871e9b, 0xdf2855ce, 0x0d89a18c, 0x6842e6bf, 0x0f2d9941, 0x16bb54b0
};

//#define XT(x) (((x) << 1) ^ (((x) >> 7) ? 0x1b : 0))

__global__ void verus_gpu_hash(uint32_t threads, uint32_t startNonce, uint32_t *resNonce, uint128m * d_key_input, uint128m * d_mid, uint32_t *d_fix_r, uint32_t *d_fix_rex);
__global__ void verus_gpu_final(uint32_t threads, uint32_t startNonce, uint32_t *resNonce, uint128m * d_key_input, const  uint128m * d_mid);
__global__ void verus_extra_gpu_prepare(const uint32_t threads, uint128m * d_key_input);
__global__ void verus_extra_gpu_fix(const uint32_t threads, uint128m * d_key_input, uint32_t *d_fix_r, uint32_t *d_fix_rex);


static uint32_t *d_nonces[MAX_GPUS];
static uint32_t *d_fix_rand[MAX_GPUS];
static uint32_t *d_fix_randex[MAX_GPUS];
static uint4 *d_long_keys[MAX_GPUS];

static uint4 *d_mid[MAX_GPUS];

static hipStream_t streams[MAX_GPUS];
static uint8_t run[MAX_GPUS];

__device__ __constant__ uint128m vkey[VERUS_KEY_SIZE128];
__device__ __constant__ uint128m blockhash_half[4];
__device__ __constant__ uint32_t ptarget[8];

__host__
void verus_init(int thr_id, uint32_t throughput)
{
	//hipFuncSetCacheConfig(reinterpret_cast<const void*>(verus_gpu_hash), hipFuncCachePreferL1);
	CUDA_SAFE_CALL(hipMalloc(&d_nonces[thr_id], 1 * sizeof(uint32_t)));

	CUDA_SAFE_CALL(hipMalloc(&d_long_keys[thr_id], throughput * VERUS_KEY_SIZE));
	CUDA_SAFE_CALL(hipMalloc(&d_mid[thr_id], throughput * 16));
	CUDA_SAFE_CALL(hipMalloc(&d_fix_rand[thr_id], throughput * sizeof(uint32_t) * 32));
	CUDA_SAFE_CALL(hipMalloc(&d_fix_randex[thr_id], throughput * sizeof(uint32_t) * 32));

};

__host__
void verus_setBlock(uint8_t *blockf, uint32_t *pTargetIn, uint8_t *lkey, int thr_id, uint32_t throughput)
{
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(ptarget), (void**)pTargetIn, 8 * sizeof(uint32_t), 0, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(blockhash_half), (void**)blockf, 64 * sizeof(uint8_t), 0, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(vkey),(void**)lkey, VERUS_KEY_SIZE * sizeof(uint8_t), 0, hipMemcpyHostToDevice));
	dim3 grid2(throughput);
	verus_extra_gpu_prepare << <grid2, 128 >> > (0, d_long_keys[thr_id]); //setup global mem with lots of keys	

};
__host__
void verus_hash(int thr_id, uint32_t threads, uint32_t startNonce, uint32_t *resNonces)
{
	hipMemset(d_nonces[thr_id], 0xff, 1 * sizeof(uint32_t));
	const uint32_t threadsperblock = THREADS;
	const uint32_t threadsperblock2 = 256;

	dim3 grid((threads + threadsperblock - 1) / threadsperblock);
	dim3 grid3((threads + threadsperblock2 - 1) / threadsperblock2);
	dim3 grid2(threads);
	dim3 block(threadsperblock);


	if (run[thr_id] == 0) {
		hipStreamCreate(&streams[thr_id]);
		run[thr_id] = 1;
	}
//	verus_extra_gpu_prepare << <grid2, 128 >> > (0, d_long_keys[thr_id]); //setup global mem with lots of keys	
	verus_gpu_hash << <grid, block, 0, streams[thr_id] >> >(threads, startNonce, d_nonces[thr_id], d_long_keys[thr_id], d_mid[thr_id], d_fix_rand[thr_id], d_fix_randex[thr_id]);
	verus_gpu_final << <grid3, 256, 0, streams[thr_id] >> >(threads, startNonce, d_nonces[thr_id], d_long_keys[thr_id], d_mid[thr_id]);
	verus_extra_gpu_fix << <grid2, 32, 0, streams[thr_id] >> > (0, d_long_keys[thr_id], d_fix_rand[thr_id], d_fix_randex[thr_id]); //setup global mem with lots of keys	
	CUDA_SAFE_CALL(hipMemcpy(resNonces, d_nonces[thr_id], 1 * sizeof(uint32_t), hipMemcpyDeviceToHost));

};
__device__ __forceinline__
uint32_t xor3x(uint32_t a, uint32_t b, uint32_t c) {
	uint32_t result;
#if __CUDA_ARCH__ >= 500 && CUDA_VERSION >= 7050
	asm("lop3.b32 %0, %1, %2, %3, 0x96;" : "=r"(result) : "r"(a), "r"(b), "r"(c)); //0x96 = 0xF0 ^ 0xCC ^ 0xAA
#else
	result = a^b^c;
#endif
	return result;
}

__device__  __forceinline__  uint128m _mm_xor_si128_emu(uint128m a, uint128m b)
{
	uint128m result;
	asm("xor.b32 %0, %1, %2; // xor1" : "=r"(result.x) : "r"(a.x), "r"(b.x));
	asm("xor.b32 %0, %1, %2; // xor1" : "=r"(result.y) : "r"(a.y), "r"(b.y));
	asm("xor.b32 %0, %1, %2; // xor1" : "=r"(result.z) : "r"(a.z), "r"(b.z));
	asm("xor.b32 %0, %1, %2; // xor1" : "=r"(result.w) : "r"(a.w), "r"(b.w));
	return result;


}
//#define XT4(x) ((((x) << 1) & 0xfefefefe) ^ ((((x) >> 7) & 0x1010101) * 0x1b))

__device__  __forceinline__  uint32_t XT4(uint32_t b)
{
	uint32_t tmp1,tmp2,tmp3;
	
	tmp1 = (b << 1) & 0xfefefefe;
	tmp2 = (b >> 7) & 0x1010101;
	asm("mul.lo.u32 %0, %1, 27; ": "=r"(tmp3) : "r"(tmp2));
	asm("xor.b32 %0, %1, %2; // xor1" : "=r"(tmp2) : "r"(tmp1), "r"(tmp3));
	
	return tmp2;
}

__device__ __forceinline__  uint128m _mm_clmulepi64_si128_emu(uint128m ai, uint128m bi, int imm)
{
	uint64_t a = ((uint64_t*)&ai)[0]; // (0xffffffffull & ai.x) | ((0x00000000ffffffffull & ai.y) << 32);//+ (imm & 1));

	uint64_t b = ((uint64_t*)&bi)[1]; // (0xffffffffull & bi.z) | ((0x00000000ffffffffull & bi.w) << 32);
	
	uint8_t  i; //window size s = 4,
				//uint64_t two_s = 16; //2^s
				//uint64_t smask = 15; //s 15
	uint64_t u[8];
	uint128m r;
	uint64_t tmp;
	//Precomputation

	//#pragma unroll
	u[0] = 0;  //000 x b
	u[1] = b;  //001 x b
	u[2] = u[1] << 1; //010 x b
	u[3] = u[2] ^ b;  //011 x b
	u[4] = u[2] << 1; //100 x b
	u[5] = u[4] ^ b;  //101 x b
	u[6] = u[3] << 1; //110 x b
	u[7] = u[6] ^ b;  //111 x b
					  //Multiply
	((uint64_t*)&r)[0] = u[a & 7]; //first window only affects lower word

	r.z = r.w = 0;
	//#pragma unroll
	for (i = 3; i < 64; i += 3) {
		tmp = u[a >> i & 7];
		((uint64_t*)&r)[0] ^= tmp << i;

		((uint64_t*)&r)[1] ^= tmp >> (64 - i);
	}

	if ((bi.w ) & 0x80000000)
	{
		uint32_t t0 = LIMMY_R(ai.x, ai.y, 1);
		uint32_t t1 = ai.y >> 1;
		r.z ^= (t0 & 0xDB6DB6DB); //0, 21x 110
		r.w ^= (t1 & 0x36DB6DB6); //0x6DB6DB6DB6DB6DB6 -> 0x36DB6DB6DB6DB6DB after >>1
	}
	if ((bi.w ) &  0x40000000)
	{
		uint32_t t0 = LIMMY_R(ai.x, ai.y, 2);
		uint32_t t1 = ai.y >> 2;
		r.z ^= (t0 & 0x49249249); //0, 21x 100
		r.w ^= (t1 & 0x12492492); //0x4924924924924924 -> 0x1249249249249249 after >>2
	}

	return r;
}

__device__ uint128m _mm_clmulepi64_si128_emu2(uint128m ai)
{
	uint64_t a = ((uint64_t*)&ai)[1];

	//uint64_t b = 27 ;
	uint8_t  i; //window size s = 4,
				//uint64_t two_s = 16; //2^s
				//uint64_t smask = 15; //s 15
	uint8_t u[8];
	uint128m r;
	uint64_t tmp;
	//Precomputation

	//#pragma unroll
	u[0] = 0;  //000 x b
	u[1] = 27;  //001 x b
	u[2] = 54; // u[1] << 1; //010 x b
	u[3] = 45;  //011 x b
	u[4] = 108; //100 x b
	u[5] = 119;  //101 x b
	u[6] = 90; //110 x b
	u[7] = 65;  //111 x b
					  //Multiply
	((uint64_t*)&r)[0] = u[a & 7]; //first window only affects lower word

	r.z = r.w = 0;
	//#pragma unroll
	for (i = 3; i < 64; i += 3) {
		tmp = u[a >> i & 7];
		((uint64_t*)&r)[0] ^= tmp << i;

		((uint64_t*)&r)[1] ^= tmp >> (64 - i);
	}

	return r;
}

#define _mm_load_si128_emu(p) (*(uint128m*)(p));

#define _mm_cvtsi128_si64_emu(p) (((int64_t *)&p)[0]);

#define _mm_cvtsi128_si32_emu(p) (((int32_t *)&a)[0]);


__device__  void _mm_unpackboth_epi32_emu(uint128m &a, uint128m &b)
{
	uint64_t value;

	asm("mov.b64 %0, {%1, %2}; ": "=l"(value) : "r"(a.z), "r"(a.y));
	asm("mov.b64 {%0, %1}, %2; ": "=r"(a.y), "=r"(a.z) : "l"(value));

	asm("mov.b64 %0, {%1, %2}; ": "=l"(value) : "r"(b.x), "r"(a.y));
	asm("mov.b64 {%0, %1}, %2; ": "=r"(a.y), "=r"(b.x) : "l"(value));

	asm("mov.b64 %0, {%1, %2}; ": "=l"(value) : "r"(b.z), "r"(a.w));
	asm("mov.b64 {%0, %1}, %2; ": "=r"(a.w), "=r"(b.z) : "l"(value));
	
	asm("mov.b64 %0, {%1, %2}; ": "=l"(value) : "r"(b.y), "r"(a.w));
	asm("mov.b64 {%0, %1}, %2; ": "=r"(a.w), "=r"(b.y) : "l"(value));
}


__device__  __forceinline__ uint128m _mm_unpacklo_epi32_emu(uint128m a, uint128m b)
{

	//uint4 t;

//	t.x = a.x;
	a.z = a.y;
	a.y = b.x;
	a.w = b.y;
	return a;
}

__device__  __forceinline__ uint128m _mm_unpackhi_epi32_emu(uint128m a, uint128m b)
{

	//uint4 t;
	b.x = a.z;
	b.y = b.z;
	b.z = a.w;
	//t.w = b.w;

	return b;
}
__device__   __forceinline__ void aesenc(unsigned char * __restrict__ s, const uint128m * __restrict__ rk, uint32_t * __restrict__ sharedMemory1)
{
//#define XT(x) (((x) << 1) ^ (((x) >> 7) ? 0x1b : 0))

//#define XT4(x) ((((x) << 1) & 0xfefefefe) ^ ((((x) >> 31) & 1) ? 0x1b000000 : 0)^ ((((x) >> 23)&1) ? 0x001b0000 : 0)^ ((((x) >> 15)&1) ? 0x00001b00 : 0)^ ((((x) >> 7)&1) ? 0x0000001b : 0))


	//const uint32_t  t, u, w;
	register uint32_t  v[4];
	//const uint128m rk2 = ((uint128m*)&rk[0])[0];

	((uint8_t*)&v[0])[0] = ((uint8_t*)&sharedMemory1[0])[s[0]];
	((uint8_t*)&v[0])[7] = ((uint8_t*)&sharedMemory1[0])[s[1]];
	((uint8_t*)&v[0])[10] = ((uint8_t*)&sharedMemory1[0])[s[2]];
	((uint8_t*)&v[0])[13] = ((uint8_t*)&sharedMemory1[0])[s[3]];
	((uint8_t*)&v[0])[1] = ((uint8_t*)&sharedMemory1[0])[s[4]];
	((uint8_t*)&v[0])[4] = ((uint8_t*)&sharedMemory1[0])[s[5]];
	((uint8_t*)&v[0])[11] = ((uint8_t*)&sharedMemory1[0])[s[6]];
	((uint8_t*)&v[0])[14] = ((uint8_t*)&sharedMemory1[0])[s[7]];
	((uint8_t*)&v[0])[2] = ((uint8_t*)&sharedMemory1[0])[s[8]];
	((uint8_t*)&v[0])[5] = ((uint8_t*)&sharedMemory1[0])[s[9]];
	((uint8_t*)&v[0])[8] = ((uint8_t*)&sharedMemory1[0])[s[10]];
	((uint8_t*)&v[0])[15] = ((uint8_t*)&sharedMemory1[0])[s[11]];
	((uint8_t*)&v[0])[3] = ((uint8_t*)&sharedMemory1[0])[s[12]];
	((uint8_t*)&v[0])[6] = ((uint8_t*)&sharedMemory1[0])[s[13]];
	((uint8_t*)&v[0])[9] = ((uint8_t*)&sharedMemory1[0])[s[14]];
	((uint8_t*)&v[0])[12] = ((uint8_t*)&sharedMemory1[0])[s[15]];

	uint32_t t = v[0];
	uint32_t w = v[0] ^ v[1];
	uint32_t u; // = w ^ v[2] ^ v[3];
	u = xor3x(w, v[2], v[3]);
	v[0] = xor3x(v[0], u, XT4(w));
	v[1] = xor3x(v[1], u, XT4(v[1] ^ v[2]));
	v[2] = xor3x(v[2], u, XT4(v[2] ^ v[3]));
	v[3] = xor3x(v[3], u, XT4(v[3] ^ t));


	s[0] = ((uint8_t*)&v[0])[0];
	s[1] = ((uint8_t*)&v[0])[4] ;
	s[2] = ((uint8_t*)&v[0])[8] ;
	s[3] = ((uint8_t*)&v[0])[12] ;
	s[4] = ((uint8_t*)&v[0])[1] ;
	s[5] = ((uint8_t*)&v[0])[5] ;

	s[6] = ((uint8_t*)&v[0])[9] ;
	s[7] = ((uint8_t*)&v[0])[13] ;
	s[8] = ((uint8_t*)&v[0])[2] ;
	s[9] = ((uint8_t*)&v[0])[6] ;
	s[10] = ((uint8_t*)&v[0])[10] ;
	s[11] = ((uint8_t*)&v[0])[14] ;
	s[12] = ((uint8_t*)&v[0])[3] ;
	s[13] = ((uint8_t*)&v[0])[7];
	s[14] = ((uint8_t*)&v[0])[11];
	s[15] = ((uint8_t*)&v[0])[15];

	((uint128m*)&s[0])[0] = make_uint4(((uint32_t*)&s[0])[0] ^ rk[0].x, ((uint32_t*)&s[0])[1] ^ rk[0].y, ((uint32_t*)&s[0])[2] ^ rk[0].z, ((uint32_t*)&s[0])[3] ^ rk[0].w);


}
#define AES2_EMU2(s0, s1, rci) \
  aesenc4((unsigned char *)&s0, (unsigned char *)&s1, &rc[rci],sharedMemory1); 

__device__   __forceinline__ void aesenc4(unsigned char * __restrict__ s1, unsigned char * __restrict__ s2, uint128m * __restrict__ rk, uint32_t * __restrict__ sharedMemory1)
{

//#define XT4(x) ((((x) << 1) & 0xfefefefe) ^ ((((x) >> 31) & 1) ? 0x1b000000 : 0)^ ((((x) >> 23)&1) ? 0x001b0000 : 0)^ ((((x) >> 15)&1) ? 0x00001b00 : 0)^ ((((x) >> 7)&1) ? 0x0000001b : 0))

	//const uint32_t  t, u, w;
	uint32_t v[4];
	uint32_t t, w, u;

	//const uint128m rk2 = ((uint128m*)&rk[0])[0];

	((uint8_t*)&v[0])[0] = ((uint8_t*)&sharedMemory1[0])[s1[0]];
	((uint8_t*)&v[0])[7] = ((uint8_t*)&sharedMemory1[0])[s1[1]];
	((uint8_t*)&v[0])[10] = ((uint8_t*)&sharedMemory1[0])[s1[2]];
	((uint8_t*)&v[0])[13] = ((uint8_t*)&sharedMemory1[0])[s1[3]];
	((uint8_t*)&v[0])[1] = ((uint8_t*)&sharedMemory1[0])[s1[4]];
	((uint8_t*)&v[0])[4] = ((uint8_t*)&sharedMemory1[0])[s1[5]];
	((uint8_t*)&v[0])[11] = ((uint8_t*)&sharedMemory1[0])[s1[6]];
	((uint8_t*)&v[0])[14] = ((uint8_t*)&sharedMemory1[0])[s1[7]];
	((uint8_t*)&v[0])[2] = ((uint8_t*)&sharedMemory1[0])[s1[8]];
	((uint8_t*)&v[0])[5] = ((uint8_t*)&sharedMemory1[0])[s1[9]];
	((uint8_t*)&v[0])[8] = ((uint8_t*)&sharedMemory1[0])[s1[10]];
	((uint8_t*)&v[0])[15] = ((uint8_t*)&sharedMemory1[0])[s1[11]];
	((uint8_t*)&v[0])[3] = ((uint8_t*)&sharedMemory1[0])[s1[12]];
	((uint8_t*)&v[0])[6] = ((uint8_t*)&sharedMemory1[0])[s1[13]];
	((uint8_t*)&v[0])[9] = ((uint8_t*)&sharedMemory1[0])[s1[14]];
	((uint8_t*)&v[0])[12] = ((uint8_t*)&sharedMemory1[0])[s1[15]];

	t = v[0];
	 w = v[0] ^ v[1];
	
	 u = xor3x(w, v[2], v[3]);
	 v[0] = xor3x(v[0], u, XT4(w));
	 v[1] = xor3x(v[1], u, XT4(v[1] ^ v[2]));
	 v[2] = xor3x(v[2], u, XT4(v[2] ^ v[3]));
	 v[3] = xor3x(v[3], u, XT4(v[3] ^ t));


	s1[0] = ((uint8_t*)&v[0])[0];
	s1[1] = ((uint8_t*)&v[0])[4];
	s1[2] = ((uint8_t*)&v[0])[8];
	s1[3] = ((uint8_t*)&v[0])[12];
	s1[4] = ((uint8_t*)&v[0])[1];
	s1[5] = ((uint8_t*)&v[0])[5];

	s1[6] = ((uint8_t*)&v[0])[9];
	s1[7] = ((uint8_t*)&v[0])[13];
	s1[8] = ((uint8_t*)&v[0])[2];
	s1[9] = ((uint8_t*)&v[0])[6];
	s1[10] = ((uint8_t*)&v[0])[10];
	s1[11] = ((uint8_t*)&v[0])[14];
	s1[12] = ((uint8_t*)&v[0])[3];
	s1[13] = ((uint8_t*)&v[0])[7];
	s1[14] = ((uint8_t*)&v[0])[11];
	s1[15] = ((uint8_t*)&v[0])[15];

	((uint128m*)&s1[0])[0] = make_uint4(((uint32_t*)&s1[0])[0] ^ rk[0].x, ((uint32_t*)&s1[0])[1] ^ rk[0].y, ((uint32_t*)&s1[0])[2] ^ rk[0].z, ((uint32_t*)&s1[0])[3] ^ rk[0].w);

	((uint8_t*)&v[0])[0] = ((uint8_t*)&sharedMemory1[0])[s2[0]];
	((uint8_t*)&v[0])[7] = ((uint8_t*)&sharedMemory1[0])[s2[1]];
	((uint8_t*)&v[0])[10] = ((uint8_t*)&sharedMemory1[0])[s2[2]];
	((uint8_t*)&v[0])[13] = ((uint8_t*)&sharedMemory1[0])[s2[3]];
	((uint8_t*)&v[0])[1] = ((uint8_t*)&sharedMemory1[0])[s2[4]];
	((uint8_t*)&v[0])[4] = ((uint8_t*)&sharedMemory1[0])[s2[5]];
	((uint8_t*)&v[0])[11] = ((uint8_t*)&sharedMemory1[0])[s2[6]];
	((uint8_t*)&v[0])[14] = ((uint8_t*)&sharedMemory1[0])[s2[7]];
	((uint8_t*)&v[0])[2] = ((uint8_t*)&sharedMemory1[0])[s2[8]];
	((uint8_t*)&v[0])[5] = ((uint8_t*)&sharedMemory1[0])[s2[9]];
	((uint8_t*)&v[0])[8] = ((uint8_t*)&sharedMemory1[0])[s2[10]];
	((uint8_t*)&v[0])[15] = ((uint8_t*)&sharedMemory1[0])[s2[11]];
	((uint8_t*)&v[0])[3] = ((uint8_t*)&sharedMemory1[0])[s2[12]];
	((uint8_t*)&v[0])[6] = ((uint8_t*)&sharedMemory1[0])[s2[13]];
	((uint8_t*)&v[0])[9] = ((uint8_t*)&sharedMemory1[0])[s2[14]];
	((uint8_t*)&v[0])[12] = ((uint8_t*)&sharedMemory1[0])[s2[15]];

	t = v[0];
	w = v[0] ^ v[1];

	u = xor3x(w, v[2], v[3]);
	v[0] = xor3x(v[0], u, XT4(w));
	v[1] = xor3x(v[1], u, XT4(v[1] ^ v[2]));
	v[2] = xor3x(v[2], u, XT4(v[2] ^ v[3]));
	v[3] = xor3x(v[3], u, XT4(v[3] ^ t));


	s2[0] = ((uint8_t*)&v[0])[0];
	s2[1] = ((uint8_t*)&v[0])[4];
	s2[2] = ((uint8_t*)&v[0])[8];
	s2[3] = ((uint8_t*)&v[0])[12];
	s2[4] = ((uint8_t*)&v[0])[1];
	s2[5] = ((uint8_t*)&v[0])[5];

	s2[6] = ((uint8_t*)&v[0])[9];
	s2[7] = ((uint8_t*)&v[0])[13];
	s2[8] = ((uint8_t*)&v[0])[2];
	s2[9] = ((uint8_t*)&v[0])[6];
	s2[10] = ((uint8_t*)&v[0])[10];
	s2[11] = ((uint8_t*)&v[0])[14];
	s2[12] = ((uint8_t*)&v[0])[3];
	s2[13] = ((uint8_t*)&v[0])[7];
	s2[14] = ((uint8_t*)&v[0])[11];
	s2[15] = ((uint8_t*)&v[0])[15];

	((uint128m*)&s2[0])[0] = make_uint4(((uint32_t*)&s2[0])[0] ^ rk[1].x, ((uint32_t*)&s2[0])[1] ^ rk[1].y, ((uint32_t*)&s2[0])[2] ^ rk[1].z, ((uint32_t*)&s2[0])[3] ^ rk[1].w);


	((uint8_t*)&v[0])[0] = ((uint8_t*)&sharedMemory1[0])[s1[0]];
	((uint8_t*)&v[0])[7] = ((uint8_t*)&sharedMemory1[0])[s1[1]];
	((uint8_t*)&v[0])[10] = ((uint8_t*)&sharedMemory1[0])[s1[2]];
	((uint8_t*)&v[0])[13] = ((uint8_t*)&sharedMemory1[0])[s1[3]];
	((uint8_t*)&v[0])[1] = ((uint8_t*)&sharedMemory1[0])[s1[4]];
	((uint8_t*)&v[0])[4] = ((uint8_t*)&sharedMemory1[0])[s1[5]];
	((uint8_t*)&v[0])[11] = ((uint8_t*)&sharedMemory1[0])[s1[6]];
	((uint8_t*)&v[0])[14] = ((uint8_t*)&sharedMemory1[0])[s1[7]];
	((uint8_t*)&v[0])[2] = ((uint8_t*)&sharedMemory1[0])[s1[8]];
	((uint8_t*)&v[0])[5] = ((uint8_t*)&sharedMemory1[0])[s1[9]];
	((uint8_t*)&v[0])[8] = ((uint8_t*)&sharedMemory1[0])[s1[10]];
	((uint8_t*)&v[0])[15] = ((uint8_t*)&sharedMemory1[0])[s1[11]];
	((uint8_t*)&v[0])[3] = ((uint8_t*)&sharedMemory1[0])[s1[12]];
	((uint8_t*)&v[0])[6] = ((uint8_t*)&sharedMemory1[0])[s1[13]];
	((uint8_t*)&v[0])[9] = ((uint8_t*)&sharedMemory1[0])[s1[14]];
	((uint8_t*)&v[0])[12] = ((uint8_t*)&sharedMemory1[0])[s1[15]];

	t = v[0];
	w = v[0] ^ v[1];

	u = xor3x(w, v[2], v[3]);
	v[0] = xor3x(v[0], u, XT4(w));
	v[1] = xor3x(v[1], u, XT4(v[1] ^ v[2]));
	v[2] = xor3x(v[2], u, XT4(v[2] ^ v[3]));
	v[3] = xor3x(v[3], u, XT4(v[3] ^ t));


	s1[0] = ((uint8_t*)&v[0])[0];
	s1[1] = ((uint8_t*)&v[0])[4];
	s1[2] = ((uint8_t*)&v[0])[8];
	s1[3] = ((uint8_t*)&v[0])[12];
	s1[4] = ((uint8_t*)&v[0])[1];
	s1[5] = ((uint8_t*)&v[0])[5];

	s1[6] = ((uint8_t*)&v[0])[9];
	s1[7] = ((uint8_t*)&v[0])[13];
	s1[8] = ((uint8_t*)&v[0])[2];
	s1[9] = ((uint8_t*)&v[0])[6];
	s1[10] = ((uint8_t*)&v[0])[10];
	s1[11] = ((uint8_t*)&v[0])[14];
	s1[12] = ((uint8_t*)&v[0])[3];
	s1[13] = ((uint8_t*)&v[0])[7];
	s1[14] = ((uint8_t*)&v[0])[11];
	s1[15] = ((uint8_t*)&v[0])[15];

	((uint128m*)&s1[0])[0] = make_uint4(((uint32_t*)&s1[0])[0] ^ rk[2].x, ((uint32_t*)&s1[0])[1] ^ rk[2].y, ((uint32_t*)&s1[0])[2] ^ rk[2].z, ((uint32_t*)&s1[0])[3] ^ rk[2].w);

	((uint8_t*)&v[0])[0] = ((uint8_t*)&sharedMemory1[0])[s2[0]];
	((uint8_t*)&v[0])[7] = ((uint8_t*)&sharedMemory1[0])[s2[1]];
	((uint8_t*)&v[0])[10] = ((uint8_t*)&sharedMemory1[0])[s2[2]];
	((uint8_t*)&v[0])[13] = ((uint8_t*)&sharedMemory1[0])[s2[3]];
	((uint8_t*)&v[0])[1] = ((uint8_t*)&sharedMemory1[0])[s2[4]];
	((uint8_t*)&v[0])[4] = ((uint8_t*)&sharedMemory1[0])[s2[5]];
	((uint8_t*)&v[0])[11] = ((uint8_t*)&sharedMemory1[0])[s2[6]];
	((uint8_t*)&v[0])[14] = ((uint8_t*)&sharedMemory1[0])[s2[7]];
	((uint8_t*)&v[0])[2] = ((uint8_t*)&sharedMemory1[0])[s2[8]];
	((uint8_t*)&v[0])[5] = ((uint8_t*)&sharedMemory1[0])[s2[9]];
	((uint8_t*)&v[0])[8] = ((uint8_t*)&sharedMemory1[0])[s2[10]];
	((uint8_t*)&v[0])[15] = ((uint8_t*)&sharedMemory1[0])[s2[11]];
	((uint8_t*)&v[0])[3] = ((uint8_t*)&sharedMemory1[0])[s2[12]];
	((uint8_t*)&v[0])[6] = ((uint8_t*)&sharedMemory1[0])[s2[13]];
	((uint8_t*)&v[0])[9] = ((uint8_t*)&sharedMemory1[0])[s2[14]];
	((uint8_t*)&v[0])[12] = ((uint8_t*)&sharedMemory1[0])[s2[15]];

	t = v[0];
	w = v[0] ^ v[1];

	u = xor3x(w, v[2], v[3]);
	v[0] = xor3x(v[0], u, XT4(w));
	v[1] = xor3x(v[1], u, XT4(v[1] ^ v[2]));
	v[2] = xor3x(v[2], u, XT4(v[2] ^ v[3]));
	v[3] = xor3x(v[3], u, XT4(v[3] ^ t));



	s2[0] = ((uint8_t*)&v[0])[0];
	s2[1] = ((uint8_t*)&v[0])[4];
	s2[2] = ((uint8_t*)&v[0])[8];
	s2[3] = ((uint8_t*)&v[0])[12];
	s2[4] = ((uint8_t*)&v[0])[1];
	s2[5] = ((uint8_t*)&v[0])[5];

	s2[6] = ((uint8_t*)&v[0])[9];
	s2[7] = ((uint8_t*)&v[0])[13];
	s2[8] = ((uint8_t*)&v[0])[2];
	s2[9] = ((uint8_t*)&v[0])[6];
	s2[10] = ((uint8_t*)&v[0])[10];
	s2[11] = ((uint8_t*)&v[0])[14];
	s2[12] = ((uint8_t*)&v[0])[3];
	s2[13] = ((uint8_t*)&v[0])[7];
	s2[14] = ((uint8_t*)&v[0])[11];
	s2[15] = ((uint8_t*)&v[0])[15];

	((uint128m*)&s2[0])[0] = make_uint4(((uint32_t*)&s2[0])[0] ^ rk[3].x, ((uint32_t*)&s2[0])[1] ^ rk[3].y, ((uint32_t*)&s2[0])[2] ^ rk[3].z, ((uint32_t*)&s2[0])[3] ^ rk[3].w);



	_mm_unpackboth_epi32_emu(((uint128m*)&s1[0])[0], ((uint128m*)&s2[0])[0]);



}



__device__  __forceinline__ uint128m _mm_cvtsi32_si128_emu(uint32_t lo)
{
	uint128m result = { 0 };
	result.x= lo;
	//((uint32_t *)&result)[1] = 0;
//	((uint64_t *)&result)[1] = 0;
	return result;
}
__device__  __forceinline__ uint128m _mm_cvtsi64_si128_emu(uint64_t lo)
{
	uint128m result = {0};
	((uint64_t *)&result)[0] = lo;
	//((uint64_t *)&result)[1] = 0;
	return result;
}
__device__  __forceinline__ uint128m _mm_set_epi64x_emu(uint64_t hi, uint64_t lo)
{
	uint128m result;
	((uint64_t *)&result)[0] = lo;
	((uint64_t *)&result)[1] = hi;
	return result;
}
__device__  uint128m _mm_shuffle_epi8_emu(uint128m b)
{
	uint128m result;
	uint128m M = { 0x2d361b00,0x415a776c,0xf5eec3d8,0x9982afb4 };
//#pragma unroll
	for (int i = 0; i < 16; i++)
	{
		if (((uint8_t *)&b)[i] & 0x80)
		{
			((uint8_t *)&result)[i] = 0;
		}
		else
		{
			((uint8_t *)&result)[i] = ((uint8_t *)&M)[((uint8_t *)&b)[i] & 0xf];
		}
	}

	return result;
}



__device__  __forceinline__ uint128m _mm_srli_si128_emu(uint128m input, int imm8)
{
	//we can cheat here as its an 8 byte shift just copy the 64bits
	uint128m temp;
	((uint64_t*)&temp)[0] = ((uint64_t*)&input)[1];
	((uint64_t*)&temp)[1] = 0;


	return temp;
}



__device__  uint128m _mm_mulhrs_epi16_emu(uint128m _a, uint128m _b)
{
	int16_t result[8];

	int32_t po;
	int16_t *a = (int16_t*)&_a, *b = (int16_t*)&_b;
#pragma unroll 8
	for (int i = 0; i < 8; i++)
	{
		asm("mad.lo.s32 %0, %1, %2, 16384; ": "=r"(po) : "r"((int32_t)a[i]), "r"((int32_t)b[i]));

		result[i] = po >> 15;
	//	result[i] = (int16_t)((((int32_t)(a[i]) * (int32_t)(b[i])) + 0x4000) >> 15);
	
	}
	return *(uint128m *)result;
}


__device__  __forceinline__ void case_0(uint128m &prand, uint128m &prandex, const  uint128m *pbuf,
	uint64_t selector, uint128m &acc)
{
	const uint128m temp1 = prandex;

	const uint128m temp2 = _mm_load_si128_emu(pbuf - (((selector & 1) << 1) - 1));


	const uint128m add1 = _mm_xor_si128_emu(temp1, temp2);

	const uint128m clprod1 = _mm_clmulepi64_si128_emu(add1, add1, 0x10);
	acc = _mm_xor_si128_emu(clprod1, acc);

	const uint128m tempa1 = _mm_mulhrs_epi16_emu(acc, temp1);
	const uint128m tempa2 = _mm_xor_si128_emu(tempa1, temp1);

	const uint128m temp12 = prand;
	prand = tempa2;


	const uint128m temp22 = _mm_load_si128_emu(pbuf);
	const uint128m add12 = _mm_xor_si128_emu(temp12, temp22);
	const uint128m clprod12 = _mm_clmulepi64_si128_emu(add12, add12, 0x10);
	acc = _mm_xor_si128_emu(clprod12, acc);

	const uint128m tempb1 = _mm_mulhrs_epi16_emu(acc, temp12);
	const uint128m tempb2 = _mm_xor_si128_emu(tempb1, temp12);
	prandex = tempb2;

}

__device__  __forceinline__ void case_4(uint128m &prand, uint128m &prandex, const  uint128m *pbuf,
	uint64_t selector, uint128m &acc)
{
	const uint128m temp1 = prand;
	const uint128m temp2 = _mm_load_si128_emu(pbuf);
	const uint128m add1 = _mm_xor_si128_emu(temp1, temp2);
	const uint128m clprod1 = _mm_clmulepi64_si128_emu(add1, add1, 0x10);
	acc = _mm_xor_si128_emu(clprod1, acc);
	const uint128m clprod2 = _mm_clmulepi64_si128_emu(temp2, temp2, 0x10);
	acc = _mm_xor_si128_emu(clprod2, acc);

	const uint128m tempa1 = _mm_mulhrs_epi16_emu(acc, temp1);
	const uint128m tempa2 = _mm_xor_si128_emu(tempa1, temp1);

	const uint128m temp12 = prandex;
	prandex = tempa2;

	const uint128m temp22 = _mm_load_si128_emu(pbuf - (((selector & 1) << 1) - 1));
	const uint128m add12 = _mm_xor_si128_emu(temp12, temp22);
	acc = _mm_xor_si128_emu(add12, acc);

	const uint128m tempb1 = _mm_mulhrs_epi16_emu(acc, temp12);
	const uint128m tempb2 = _mm_xor_si128_emu(tempb1, temp12);
	prand = tempb2;
}

__device__ __forceinline__  void case_8(uint128m &prand, uint128m &prandex, const  uint128m *pbuf,
	uint64_t selector, uint128m &acc)
{
	const uint128m temp1 = prandex;
	const uint128m temp2 = _mm_load_si128_emu(pbuf);
	const uint128m add1 = _mm_xor_si128_emu(temp1, temp2);
	acc = _mm_xor_si128_emu(add1, acc);

	const uint128m tempa1 = _mm_mulhrs_epi16_emu(acc, temp1);
	const uint128m tempa2 = _mm_xor_si128_emu(tempa1, temp1);

	const uint128m temp12 = prand;
	prand = tempa2;

	const uint128m temp22 = _mm_load_si128_emu(pbuf - (((selector & 1) << 1) - 1));
	const uint128m add12 = _mm_xor_si128_emu(temp12, temp22);
	const uint128m clprod12 = _mm_clmulepi64_si128_emu(add12, add12, 0x10);
	acc = _mm_xor_si128_emu(clprod12, acc);
	const uint128m clprod22 = _mm_clmulepi64_si128_emu(temp22, temp22, 0x10);
	acc = _mm_xor_si128_emu(clprod22, acc);

	const uint128m tempb1 = _mm_mulhrs_epi16_emu(acc, temp12);
	const uint128m tempb2 = _mm_xor_si128_emu(tempb1, temp12);
	prandex = tempb2;
}

__device__ void case_0c(uint128m &prand, uint128m &prandex, const  uint128m *pbuf,
	uint64_t selector, uint128m &acc)
{
	const uint128m temp1 = prand;
	const uint128m temp2 = _mm_load_si128_emu(pbuf - (((selector & 1) << 1) - 1));
	const uint128m add1 = _mm_xor_si128_emu(temp1, temp2);

	// cannot be zero here
	const int32_t divisor = ((uint32_t*)&selector)[0];

	acc = _mm_xor_si128_emu(add1, acc);

	int64_t dividend = _mm_cvtsi128_si64_emu(acc);
	int64_t tmpmod = dividend % divisor;
	const uint128m modulo = _mm_cvtsi32_si128_emu(tmpmod);
	acc = _mm_xor_si128_emu(modulo, acc);

	const uint128m tempa1 = _mm_mulhrs_epi16_emu(acc, temp1);
	const uint128m tempa2 = _mm_xor_si128_emu(tempa1, temp1);
	dividend &= 1;
	if (dividend)
	{
		const uint128m temp12 = prandex;
		prandex = tempa2;

		const uint128m temp22 = _mm_load_si128_emu(pbuf);
		const uint128m add12 = _mm_xor_si128_emu(temp12, temp22);
		const uint128m clprod12 = _mm_clmulepi64_si128_emu(add12, add12, 0x10);
		acc = _mm_xor_si128_emu(clprod12, acc);
		const uint128m clprod22 = _mm_clmulepi64_si128_emu(temp22, temp22, 0x10);
		acc = _mm_xor_si128_emu(clprod22, acc);

		const uint128m tempb1 = _mm_mulhrs_epi16_emu(acc, temp12);
		const uint128m tempb2 = _mm_xor_si128_emu(tempb1, temp12);
		prand = tempb2;
	}
	else
	{
		const uint128m tempb3 = prandex;
		prandex = tempa2;
		prand = tempb3;
	}
}
__device__ void case_10(uint128m &prand, uint128m &prandex, const  uint128m *pbuf,
	uint64_t selector, uint128m &acc, uint128m *randomsource, uint32_t prand_idx, uint32_t *sharedMemory1)
{			// a few AES operations
			//uint128m rc[12];

			//rc[0] = prand; 

	uint128m *rc = &randomsource[prand_idx];
	/*	rc[2] = randomsource[prand_idx + 2];
	rc[3] = randomsource[prand_idx + 3];
	rc[4] = randomsource[prand_idx + 4];
	rc[5] = randomsource[prand_idx + 5];
	rc[6] = randomsource[prand_idx + 6];
	rc[7] = randomsource[prand_idx + 7];
	rc[8] = randomsource[prand_idx + 8];
	rc[9] = randomsource[prand_idx + 9];
	rc[10] = randomsource[prand_idx + 10];
	rc[11] = randomsource[prand_idx + 11];8*/
//	uint128m tmp;

	uint128m temp1 = _mm_load_si128_emu(pbuf - (((selector & 1) << 1) - 1));
	uint128m temp2 = _mm_load_si128_emu(pbuf);

	AES2_EMU2(temp1, temp2, 0);
//	MIX2_EMU(temp1, temp2);


	AES2_EMU2(temp1, temp2, 4);
//	MIX2_EMU(temp1, temp2);

	AES2_EMU2(temp1, temp2, 8);
//	MIX2_EMU(temp1, temp2);


	acc = _mm_xor_si128_emu(temp1, acc);
	acc = _mm_xor_si128_emu(temp2, acc);

	const uint128m tempa1 = prand;
	const uint128m tempa2 = _mm_mulhrs_epi16_emu(acc, tempa1);
	const uint128m tempa3 = _mm_xor_si128_emu(tempa1, tempa2);

	const uint128m tempa4 = prandex;
	prandex = tempa3;
	prand = tempa4;
}
__device__ void case_14(uint128m &prand, uint128m &prandex, const  uint128m *pbuf,
	uint64_t selector, uint128m &acc, uint128m *randomsource, uint32_t prand_idx, uint32_t *sharedMemory1)
{
	// we'll just call this one the monkins loop, inspired by Chris
	const uint128m *buftmp = pbuf - (((selector & 1) << 1) - 1);
//	uint128m tmp; // used by MIX2

	uint64_t rounds = selector >> 61; // loop randomly between 1 and 8 times
	uint128m *rc = &randomsource[prand_idx];


	uint64_t aesround = 0;
	uint128m onekey;
	uint64_t loop_c;

	for (int i = 0; i<8;i++)
	{
		if (rounds <= 8) {
			loop_c = selector & (0x10000000 << rounds);
			if (loop_c)
			{
				onekey = _mm_load_si128_emu(rc++);
				const uint128m temp2 = _mm_load_si128_emu(rounds & 1 ? pbuf : buftmp);
				const uint128m add1 = _mm_xor_si128_emu(onekey, temp2);
				const uint128m clprod1 = _mm_clmulepi64_si128_emu(add1, add1, 0x10);
				acc = _mm_xor_si128_emu(clprod1, acc);
			}
			else
			{
				onekey = _mm_load_si128_emu(rc++);
				uint128m temp2 = _mm_load_si128_emu(rounds & 1 ? buftmp : pbuf);

				const uint64_t roundidx = aesround++ << 2;
				AES2_EMU2(onekey, temp2, roundidx);

				//	MIX2_EMU(onekey, temp2);

				acc = _mm_xor_si128_emu(onekey, acc);
				acc = _mm_xor_si128_emu(temp2, acc);

			}
		}
 (rounds--);
	} 

	const uint128m tempa1 = (prand);
	const uint128m tempa2 = _mm_mulhrs_epi16_emu(acc, tempa1);
	const uint128m tempa3 = _mm_xor_si128_emu(tempa1, tempa2);

	const uint128m tempa4 = (prandex);
	prandex = tempa3;
	prand = tempa4;
}

__device__ void __forceinline__  case_18(uint128m &prand, uint128m &prandex, const  uint128m *pbuf,
	uint64_t selector, uint128m &acc)
{
	const uint128m temp1 = _mm_load_si128_emu(pbuf - (((selector & 1) << 1) - 1));
	const uint128m temp2 = (prand);
	const uint128m add1 = _mm_xor_si128_emu(temp1, temp2);
	const uint128m clprod1 = _mm_clmulepi64_si128_emu(add1, add1, 0x10);
	acc = _mm_xor_si128_emu(clprod1, acc);

	const uint128m tempa1 = _mm_mulhrs_epi16_emu(acc, temp2);
	const uint128m tempa2 = _mm_xor_si128_emu(tempa1, temp2);

	const uint128m tempb3 = (prandex);
	prandex = tempa2;
	prand = tempb3;
}

__device__  __forceinline__ void case_1c(uint128m &prand, uint128m &prandex, const  uint128m *pbuf,
	uint64_t selector, uint128m &acc)
{
	const uint128m temp1 = _mm_load_si128_emu(pbuf);
	const uint128m temp2 = (prandex);
	const uint128m add1 = _mm_xor_si128_emu(temp1, temp2);
	const uint128m clprod1 = _mm_clmulepi64_si128_emu(add1, add1, 0x10);
	acc = _mm_xor_si128_emu(clprod1, acc);


	const uint128m tempa1 = _mm_mulhrs_epi16_emu(acc, temp2);
	const uint128m tempa2 = _mm_xor_si128_emu(tempa1, temp2);
	const uint128m tempa3 = (prand);


	prand = tempa2;

	acc = _mm_xor_si128_emu(tempa3, acc);

	const uint128m tempb1 = _mm_mulhrs_epi16_emu(acc, tempa3);
	const uint128m tempb2 = _mm_xor_si128_emu(tempb1, tempa3);
	prandex = tempb2;
}



__device__ uint128m __verusclmulwithoutreduction64alignedrepeatgpu(uint128m * __restrict__ randomsource, const  uint128m *  __restrict__  buf ,
	 uint32_t *  __restrict__ sharedMemory1, uint16_t *  __restrict__ d_fix_r, uint16_t *  __restrict__ d_fix_rex)
{
    uint128m const *pbuf;
	//keyMask >>= 4;
	uint128m acc = randomsource[513];
	
#ifdef GPU_DEBUGGY
	if (nounce == 0)
	{
		printf("[GPU]BUF ito verusclmulithout        : ");
		for (int i = 0; i < 64; i++)
			printf("%02x", ((uint8_t*)&buf[0])[i]);
		printf("\n");
		printf("[GPU]KEy ito verusclmulithout        : ");
		for (int e = 0; e < 64; e++)
		printf("%02x", ((uint8_t*)&randomsource[0])[e]);
	printf("\n");
	    printf("[GPU]ACC ito verusclmulithout        : ");
	for (int i = 0; i < 16; i++)
		printf("%02x", ((uint8_t*)&acc)[i]);
	printf("\n");
	}
#endif	
	// divide key mask by 32 from bytes to uint128m
	
	uint16_t prand_idx, prandex_idx;
	uint64_t selector;
	uint128m prand;
	uint128m prandex;

	for (uint8_t i = 0; i < 32; i++)
	{
		
		selector = _mm_cvtsi128_si64_emu(acc);

		
		prand_idx = ((selector >> 5) & 511);
		prandex_idx = ((selector >> 32) & 511);
		// get two random locations in the key, which will be mutated and swapped
		
		prand = randomsource[prand_idx];
		prandex = randomsource[prandex_idx];

	//	save_rand[i] = ((selector >> 5) & keyMask);
	//	save_randex[i] = ((selector >> 32) & keyMask);

		// select random start and order of pbuf processing
		pbuf = buf + (selector & 3);
		uint8_t case_v;
		case_v = selector &  0x1cu;
#ifdef GPU_DEBUGu
		uint64_t egg, nog, salad;
		if (nounce == 0)
		{
			printf("[GPU]*****LOOP[%d]**********\n",i);
			egg = selector & 0x03u;
			nog = ((selector >> 32) & keyMask);
			salad = ((selector >> 5) & keyMask);
			printf("[GPU]selector: %llx\n case: %llx selector &3: ", selector, case_v);
			printf("%llx \n", egg);
			printf("[GPU]((selector >> 32) & keyMask) %d",nog);
			printf("[GPU]((selector >> 5) & keyMask) %d", salad);
			printf("\nacc     : ");
			printf("%016llx%016llx", ((uint64_t*)&acc)[0], ((uint64_t*)&acc)[1]);
			printf("\n");

			printf("[GPU]prand   : ");
			//for (int e = 0; e < 4; e++)
			printf("%016llx%016llx", ((uint64_t*)&prand)[0], ((uint64_t*)&prand)[1]);
			printf("\n");
			printf("[GPU]prandex : ");
			//for (int e = 0; e < 16; e++)
			printf("%016llx%016llx", ((uint64_t*)&prandex)[0], ((uint64_t*)&prandex)[1]);
			printf("\n");


		}

#endif
		
		if(case_v == 0)
		{
			case_0(prand, prandex, pbuf, selector, acc);
		}
		if (case_v == 4)
		{
			case_4(prand, prandex, pbuf, selector, acc);
		}
		if (case_v == 8)
		{
			case_8(prand, prandex, pbuf, selector, acc);
			
		}
		if (case_v == 0xc)
		{
			case_0c(prand, prandex, pbuf, selector, acc);

		}
		if (case_v == 0x10)
		{
			case_10(prand, prandex, pbuf, selector, acc,randomsource, prand_idx, sharedMemory1);


		}
		if(case_v == 0x14)
		{
			case_14(prand, prandex, pbuf, selector, acc, randomsource, prand_idx, sharedMemory1);

		}
		if(case_v == 0x18)
		{
			case_18(prand, prandex, pbuf, selector, acc);
			
		}
		if(case_v == 0x1c)
		{
			case_1c(prand, prandex, pbuf, selector, acc);
		}	
		d_fix_r[i] = prand_idx;
		d_fix_rex[i] = prandex_idx;
		 randomsource[prand_idx] = prand;
		 randomsource[prandex_idx] = prandex;

	}

	return acc;
}


__device__   __forceinline__  uint32_t haraka512_port_keyed2222(const unsigned char * __restrict__  in, uint128m * __restrict__  rc, uint32_t * __restrict__  sharedMemory1, uint32_t nonce)
{
	uint128m s1,s2,s3,s4, tmp;

	s1 = ((uint128m*)&in[0])[0];
	s2 = ((uint128m*)&in[0])[1];
	s3 = ((uint128m*)&in[0])[2];
	s4 = ((uint128m*)&in[0])[3];

	AES4(s1, s2, s3, s4, 0);
	MIX4(s1, s2, s3, s4);

	AES4(s1, s2, s3, s4, 8);
	MIX4(s1, s2, s3, s4);

	AES4(s1, s2, s3, s4, 16);
	MIX4(s1, s2, s3, s4);

	AES4(s1, s2, s3, s4, 24);
	MIX4_LASTBUT1(s1, s2, s3, s4);

	AES4_LAST(s3, 32);

	return s3.z ^ ((uint128m*)&in[0])[3].y;

}

__device__   __forceinline__ uint64_t precompReduction64(uint128m A) {


	//static const uint128m M = { 0x2d361b00,0x415a776c,0xf5eec3d8,0x9982afb4 };
	// const uint128m tmp = { 27 };
	// A.z = 0;
	//tmp.x = 27u;
	uint128m Q2 = _mm_clmulepi64_si128_emu2(A);
	uint128m Q3 = _mm_shuffle_epi8_emu(_mm_srli_si128_emu(Q2, 8));

	//uint128m Q4 = _mm_xor_si128_emu(Q2, A);
	uint128m final;
	final.x = xor3(A.x, Q2.x, Q3.x);
	final.y = xor3(A.y, Q2.y, Q3.y);

	return _mm_cvtsi128_si64_emu(final);/// WARNING: HIGH 64 BITS SHOULD BE ASSUMED TO CONTAIN GARBAGE
}



__global__ __launch_bounds__(THREADS, 1)
void verus_gpu_hash(const uint32_t threads, const uint32_t startNonce, uint32_t * __restrict__ resNonce,
	uint128m * __restrict__ d_key_input, uint128m * __restrict__ d_mid, uint32_t * __restrict__  d_fix_r, uint32_t *__restrict__  d_fix_rex)
{
	const uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	//uint128m mid; // , biddy[VERUS_KEY_SIZE128];
	uint128m s[4];
	const uint32_t nounce = startNonce + thread;

	__shared__ uint32_t sharedMemory1[THREADS];
	__shared__ uint16_t sharedrand[32 * THREADS];
	__shared__ uint16_t sharedrandex[32 * THREADS];

	//uint32_t save_rand[32] = { 0 };
	//uint32_t save_randex[32] = { 0 };

	s[0] = blockhash_half[0];
	s[1] = blockhash_half[1];
	s[2] = blockhash_half[2];
	s[3] = blockhash_half[3];


	sharedMemory1[threadIdx.x] = sbox[threadIdx.x];// copy sbox to shared mem

	((uint32_t *)&s)[8] = nounce;

	static const uint128m lazy = { 0x00010000, 0x00000000, 0x00000000, 0x00000000 };

	__syncthreads();
	s[0] = __verusclmulwithoutreduction64alignedrepeatgpu(&d_key_input[VERUS_KEY_SIZE128 * thread], s, sharedMemory1, sharedrand + (threadIdx.x * 32), sharedrandex + (threadIdx.x * 32));

	d_mid[thread] = _mm_xor_si128_emu(s[0], lazy);

#pragma unroll
	for (int i = 0; i < 32; i++)
	{
		d_fix_r[(thread * 32) + i] = sharedrand[(threadIdx.x * 32)+i];
		d_fix_rex[(thread * 32) + i] = sharedrandex[(threadIdx.x * 32) + i];
	}
}
	__global__ __launch_bounds__(256, 1)
		void verus_gpu_final(const uint32_t threads, const uint32_t startNonce, uint32_t * __restrict__ resNonce,
			uint128m * __restrict__  d_key_input, const uint128m * __restrict__ d_mid)
	{
		const uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
		uint64_t acc = precompReduction64(d_mid[thread]);;
		//uint128m wizz = d_mid[thread];

		const uint32_t nounce = startNonce + thread;
		uint32_t hash;

		uint128m s[4];
		__shared__ uint32_t sharedMemory1[256];
		sharedMemory1[threadIdx.x] = sbox[threadIdx.x];// copy sbox to shared mem
		s[0] = blockhash_half[0];
		s[1] = blockhash_half[1];
		s[2] = blockhash_half[2];
		s[3] = blockhash_half[3];
		__syncthreads();
//	acc = precompReduction64(wizz);
	((uint32_t *)&s)[8] = nounce;
	memcpy(((uint8_t*)&s) + 47, &acc, 8);
	memcpy(((uint8_t*)&s) + 55, &acc, 8);
	memcpy(((uint8_t*)&s) + 63, &acc, 1);
	//uint64_t mask = 8191 >> 4;
	acc &= 511;
	
	//haraka512_port_keyed((unsigned char*)hash, (const unsigned char*)s, (const unsigned char*)(biddy + mask), sharedMemory1, nounce);
	hash = haraka512_port_keyed2222((const unsigned char*)s, (&d_key_input[VERUS_KEY_SIZE128 * thread] + acc), sharedMemory1,nounce);

	if (hash < ptarget[7]) { 
		
		resNonce[0] = nounce;

	}


};

__global__ __launch_bounds__(128, 1)
void verus_extra_gpu_prepare(const uint32_t threads, uint128m * d_key_input)
{

	d_key_input[(blockIdx.x * VERUS_KEY_SIZE128) + threadIdx.x] = vkey[threadIdx.x];
	d_key_input[(blockIdx.x * VERUS_KEY_SIZE128) + threadIdx.x + 128 ] = vkey[threadIdx.x + 128];
	d_key_input[(blockIdx.x * VERUS_KEY_SIZE128) + threadIdx.x + 256 ] = vkey[threadIdx.x + 256];
	d_key_input[(blockIdx.x * VERUS_KEY_SIZE128) + threadIdx.x + 384 ] = vkey[threadIdx.x + 384];
	if (threadIdx.x < 40)
		d_key_input[(blockIdx.x * VERUS_KEY_SIZE128) + threadIdx.x + 512 ] = vkey[threadIdx.x + 512];

}

__global__ __launch_bounds__(32, 1)
void verus_extra_gpu_fix(const uint32_t threads, uint128m * __restrict__ d_key_input, uint32_t *d_fix_r, uint32_t *d_fix_rex)
{
	
	d_key_input[(blockIdx.x * VERUS_KEY_SIZE128) + d_fix_r[(blockIdx.x * 32) + threadIdx.x]] = vkey[d_fix_r[(blockIdx.x * 32) +threadIdx.x]];
	d_key_input[(blockIdx.x * VERUS_KEY_SIZE128) + d_fix_rex[(blockIdx.x * 32) + threadIdx.x]] = vkey[d_fix_rex[(blockIdx.x * 32) + threadIdx.x]];
	
}
