#include "hip/hip_runtime.h"
#include <miner.h>

#include <cuda_helper.h>

typedef uint4 uint128m;
#define GPU_DEBUG
#define VERUS_KEY_SIZE 8832
#define VERUS_KEY_SIZE128 552
#define THREADS 128
#define INNERLOOP 16

#define AES2_EMU(s0, s1, rci) \
  aesenc((unsigned char *)&s0, (unsigned char *)&rc[rci],sharedMemory1); \
  aesenc((unsigned char *)&s1, (unsigned char *)&rc[rci + 1],sharedMemory1); \
  aesenc((unsigned char *)&s0, (unsigned char *)&rc[rci + 2],sharedMemory1); \
  aesenc((unsigned char *)&s1, (unsigned char *)&rc[rci + 3],sharedMemory1);


#define MIX2_EMU(s0, s1) \
  tmp = _mm_unpacklo_epi32_emu(s0, s1); \
  s1 = _mm_unpackhi_epi32_emu(s0, s1); \
  s0 = tmp;

#define AES4(s0, s1, s2, s3, rci) \
  aesenc((unsigned char *)&s0, (unsigned char *)&rc[rci],sharedMemory1); \
  aesenc((unsigned char *)&s1, (unsigned char *)&rc[rci + 1],sharedMemory1); \
  aesenc((unsigned char *)&s2, (unsigned char *)&rc[rci + 2],sharedMemory1); \
  aesenc((unsigned char *)&s3, (unsigned char *)&rc[rci + 3],sharedMemory1); \
  aesenc((unsigned char *)&s0, (unsigned char *)&rc[rci + 4], sharedMemory1); \
  aesenc((unsigned char *)&s1, (unsigned char *)&rc[rci + 5], sharedMemory1); \
  aesenc((unsigned char *)&s2, (unsigned char *)&rc[rci + 6], sharedMemory1); \
  aesenc((unsigned char *)&s3, (unsigned char *)&rc[rci + 7], sharedMemory1);

#define TRUNCSTORE(out, s3) \
  *(uint64_t*)(out + 24) = *(((uint64_t*)&s3 + 0));

#define MIX4(s0, s1, s2, s3) \
  tmp  = _mm_unpacklo_epi32_emu(s0, s1); \
  s0 = _mm_unpackhi_epi32_emu(s0, s1); \
  s1 = _mm_unpacklo_epi32_emu(s2, s3); \
  s2 = _mm_unpackhi_epi32_emu(s2, s3); \
  s3 = _mm_unpacklo_epi32_emu(s0, s2); \
  s0 = _mm_unpackhi_epi32_emu(s0, s2); \
  s2 = _mm_unpackhi_epi32_emu(s1, tmp); \
  s1 = _mm_unpacklo_epi32_emu(s1, tmp);

__host__ void verus_setBlock(uint8_t *blockf, uint32_t *pTargetIn, uint8_t *lkey, int thr_id);


__device__ const uint32_t sbox[] = {
	0x7b777c63, 0xc56f6bf2, 0x2b670130, 0x76abd7fe, 0x7dc982ca, 0xf04759fa, 0xafa2d4ad, 0xc072a49c, 0x2693fdb7, 0xccf73f36, 0xf1e5a534, 0x1531d871, 0xc323c704, 0x9a059618, 0xe2801207, 0x75b227eb, 0x1a2c8309, 0xa05a6e1b, 0xb3d63b52, 0x842fe329, 0xed00d153, 0x5bb1fc20, 0x39becb6a, 0xcf584c4a, 0xfbaaefd0, 0x85334d43, 0x7f02f945, 0xa89f3c50, 0x8f40a351, 0xf5389d92, 0x21dab6bc, 0xd2f3ff10, 0xec130ccd, 0x1744975f, 0x3d7ea7c4, 0x73195d64, 0xdc4f8160, 0x88902a22, 0x14b8ee46, 0xdb0b5ede, 0x0a3a32e0, 0x5c240649, 0x62acd3c2, 0x79e49591, 0x6d37c8e7, 0xa94ed58d, 0xeaf4566c, 0x08ae7a65, 0x2e2578ba, 0xc6b4a61c, 0x1f74dde8, 0x8a8bbd4b, 0x66b53e70, 0x0ef60348, 0xb9573561, 0x9e1dc186, 0x1198f8e1, 0x948ed969, 0xe9871e9b, 0xdf2855ce, 0x0d89a18c, 0x6842e6bf, 0x0f2d9941, 0x16bb54b0,
	0x7b777c63, 0xc56f6bf2, 0x2b670130, 0x76abd7fe, 0x7dc982ca, 0xf04759fa, 0xafa2d4ad, 0xc072a49c, 0x2693fdb7, 0xccf73f36, 0xf1e5a534, 0x1531d871, 0xc323c704, 0x9a059618, 0xe2801207, 0x75b227eb, 0x1a2c8309, 0xa05a6e1b, 0xb3d63b52, 0x842fe329, 0xed00d153, 0x5bb1fc20, 0x39becb6a, 0xcf584c4a, 0xfbaaefd0, 0x85334d43, 0x7f02f945, 0xa89f3c50, 0x8f40a351, 0xf5389d92, 0x21dab6bc, 0xd2f3ff10, 0xec130ccd, 0x1744975f, 0x3d7ea7c4, 0x73195d64, 0xdc4f8160, 0x88902a22, 0x14b8ee46, 0xdb0b5ede, 0x0a3a32e0, 0x5c240649, 0x62acd3c2, 0x79e49591, 0x6d37c8e7, 0xa94ed58d, 0xeaf4566c, 0x08ae7a65, 0x2e2578ba, 0xc6b4a61c, 0x1f74dde8, 0x8a8bbd4b, 0x66b53e70, 0x0ef60348, 0xb9573561, 0x9e1dc186, 0x1198f8e1, 0x948ed969, 0xe9871e9b, 0xdf2855ce, 0x0d89a18c, 0x6842e6bf, 0x0f2d9941, 0x16bb54b0,
	0x7b777c63, 0xc56f6bf2, 0x2b670130, 0x76abd7fe, 0x7dc982ca, 0xf04759fa, 0xafa2d4ad, 0xc072a49c, 0x2693fdb7, 0xccf73f36, 0xf1e5a534, 0x1531d871, 0xc323c704, 0x9a059618, 0xe2801207, 0x75b227eb, 0x1a2c8309, 0xa05a6e1b, 0xb3d63b52, 0x842fe329, 0xed00d153, 0x5bb1fc20, 0x39becb6a, 0xcf584c4a, 0xfbaaefd0, 0x85334d43, 0x7f02f945, 0xa89f3c50, 0x8f40a351, 0xf5389d92, 0x21dab6bc, 0xd2f3ff10, 0xec130ccd, 0x1744975f, 0x3d7ea7c4, 0x73195d64, 0xdc4f8160, 0x88902a22, 0x14b8ee46, 0xdb0b5ede, 0x0a3a32e0, 0x5c240649, 0x62acd3c2, 0x79e49591, 0x6d37c8e7, 0xa94ed58d, 0xeaf4566c, 0x08ae7a65, 0x2e2578ba, 0xc6b4a61c, 0x1f74dde8, 0x8a8bbd4b, 0x66b53e70, 0x0ef60348, 0xb9573561, 0x9e1dc186, 0x1198f8e1, 0x948ed969, 0xe9871e9b, 0xdf2855ce, 0x0d89a18c, 0x6842e6bf, 0x0f2d9941, 0x16bb54b0,
	0x7b777c63, 0xc56f6bf2, 0x2b670130, 0x76abd7fe, 0x7dc982ca, 0xf04759fa, 0xafa2d4ad, 0xc072a49c, 0x2693fdb7, 0xccf73f36, 0xf1e5a534, 0x1531d871, 0xc323c704, 0x9a059618, 0xe2801207, 0x75b227eb, 0x1a2c8309, 0xa05a6e1b, 0xb3d63b52, 0x842fe329, 0xed00d153, 0x5bb1fc20, 0x39becb6a, 0xcf584c4a, 0xfbaaefd0, 0x85334d43, 0x7f02f945, 0xa89f3c50, 0x8f40a351, 0xf5389d92, 0x21dab6bc, 0xd2f3ff10, 0xec130ccd, 0x1744975f, 0x3d7ea7c4, 0x73195d64, 0xdc4f8160, 0x88902a22, 0x14b8ee46, 0xdb0b5ede, 0x0a3a32e0, 0x5c240649, 0x62acd3c2, 0x79e49591, 0x6d37c8e7, 0xa94ed58d, 0xeaf4566c, 0x08ae7a65, 0x2e2578ba, 0xc6b4a61c, 0x1f74dde8, 0x8a8bbd4b, 0x66b53e70, 0x0ef60348, 0xb9573561, 0x9e1dc186, 0x1198f8e1, 0x948ed969, 0xe9871e9b, 0xdf2855ce, 0x0d89a18c, 0x6842e6bf, 0x0f2d9941, 0x16bb54b0
};

#define XT(x) (((x) << 1) ^ (((x) >> 7) ? 0x1b : 0))
__global__ void verus_gpu_hash(uint32_t threads, uint32_t startNonce, uint32_t *resNonce);

//__device__ __device__ uint128m local_key[THREADS][VERUS_KEY_SIZE128];
static uint32_t *d_nonces[MAX_GPUS];

__device__ __constant__ uint128m vkey[VERUS_KEY_SIZE128];
__device__ __constant__ uint8_t blockhash_half[64];
__device__ __constant__ uint32_t ptarget[8];

__host__
void verus_init(int thr_id)
{
	CUDA_SAFE_CALL(hipMalloc(&d_nonces[thr_id], 1 * sizeof(uint32_t)));
	//CUDA_SAFE_CALL(hipMalloc(&vkey[thr_id], VERUS_KEY_SIZE * sizeof(uint8_t)));
	//CUDA_SAFE_CALL(hipMalloc(&local_key[thr_id], THREADS * VERUS_KEY_SIZE * sizeof(uint8_t)));
};


__host__
void verus_setBlock(uint8_t *blockf, uint32_t *pTargetIn, uint8_t *lkey, int thr_id)
{
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(ptarget), (void**)pTargetIn, 8 * sizeof(uint32_t), 0, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(blockhash_half), (void**)blockf, 64 * sizeof(uint8_t), 0, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(vkey),(void**)lkey, VERUS_KEY_SIZE * sizeof(uint8_t), 0, hipMemcpyHostToDevice));
	

};
__host__
void verus_hash(int thr_id, uint32_t threads, uint32_t startNonce, uint32_t *resNonces)
{
	hipMemset(d_nonces[thr_id], 0xff, 1 * sizeof(uint32_t));
	//CUDA_SAFE_CALL(hipMemset(local_key[thr_id], 0x00, THREADS * VERUS_KEY_SIZE * sizeof(uint8_t)));
	const uint32_t threadsperblock = THREADS;

	dim3 grid((threads + threadsperblock - 1) / threadsperblock);
	dim3 block(threadsperblock);

	verus_gpu_hash << <grid, block >> >(threads, startNonce, d_nonces[thr_id]);
	CUDA_SAFE_CALL(hipDeviceSynchronize());
	CUDA_SAFE_CALL(hipMemcpy(resNonces, d_nonces[thr_id], 1 * sizeof(uint32_t), hipMemcpyDeviceToHost));
	//memcpy(resNonces, h_nonces[thr_id], NBN * sizeof(uint32_t));

};

__device__  __forceinline__ uint128m _mm_clmulepi64_si128_emu(uint128m ai, uint128m bi, int imm)
{
	uint64_t a = *((uint64_t*)&ai + (imm & 1));

	uint64_t b = *((uint64_t*)&bi + ((imm & 0x10) >> 4));
	
	uint8_t  i; //window size s = 4,
	//uint64_t two_s = 16; //2^s
	//uint64_t smask = 15; //s 15
	uint64_t u[16];
	uint64_t r[2];
	uint64_t tmp;
	uint64_t ifmask;
	//Precomputation
	u[0] = 0;
	u[1] = b;
#pragma unroll
	for (i = 2; i < 16; i += 2) {
		u[i] = u[i >> 1] << 1; //even indices: left shift
		u[i + 1] = u[i] ^ b; //odd indices: xor b
	}
	//Multiply
	r[0] = u[a & 15]; //first window only affects lower word
	r[1] = 0;
#pragma unroll
	for (i = 4; i < 64; i += 4) {
		tmp = u[a >> i & 15];
		r[0] ^= tmp << i;
		r[1] ^= tmp >> (64 - i);
	}
	//Repair
	uint64_t m = 0xEEEEEEEEEEEEEEEE; //s=4 => 16 times 1110
#pragma unroll
	for (i = 1; i < 4; i++) {
		tmp = ((a & m) >> i);
		m &= m << 1; //shift mask to exclude all bit j': j' mod s = i
		ifmask = -((b >> (64 - i)) & 1); //if the (64-i)th bit of b is 1
		r[1] ^= (tmp & ifmask);
	}
	uint128m out;
	((uint64_t*)&out)[0] = r[0];
	((uint64_t*)&out)[1] = r[1];
	return out;
}

__device__   __forceinline__ void aesenc(unsigned char *s, const unsigned char *rk, uint32_t *sharedMemory1)
{
	//uint32_t  t, u, w;
	//uint32_t v[4][4];

#define XT4(x) ((((x) << 1) & 0xfefefefe) ^ ((((x) >> 31) & 1) ? 0x1b000000 : 0)^ ((((x) >> 23)&1) ? 0x001b0000 : 0)^ ((((x) >> 15)&1) ? 0x00001b00 : 0)^ ((((x) >> 7)&1) ? 0x0000001b : 0))

	uint32_t  t, u;
	uint32_t v[4];

	((uint8_t*)&v[0])[0] = ((uint8_t*)&sharedMemory1[0])[s[0]];
	((uint8_t*)&v[0])[7] = ((uint8_t*)&sharedMemory1[0])[s[1]];
	((uint8_t*)&v[0])[10] = ((uint8_t*)&sharedMemory1[0])[s[2]];
	((uint8_t*)&v[0])[13] = ((uint8_t*)&sharedMemory1[0])[s[3]];
	((uint8_t*)&v[0])[1] = ((uint8_t*)&sharedMemory1[0])[s[4]];
	((uint8_t*)&v[0])[4] = ((uint8_t*)&sharedMemory1[0])[s[5]];
	((uint8_t*)&v[0])[11] = ((uint8_t*)&sharedMemory1[0])[s[6]];
	((uint8_t*)&v[0])[14] = ((uint8_t*)&sharedMemory1[0])[s[7]];
	((uint8_t*)&v[0])[2] = ((uint8_t*)&sharedMemory1[0])[s[8]];
	((uint8_t*)&v[0])[5] = ((uint8_t*)&sharedMemory1[0])[s[9]];
	((uint8_t*)&v[0])[8] = ((uint8_t*)&sharedMemory1[0])[s[10]];
	((uint8_t*)&v[0])[15] = ((uint8_t*)&sharedMemory1[0])[s[11]];
	((uint8_t*)&v[0])[3] = ((uint8_t*)&sharedMemory1[0])[s[12]];
	((uint8_t*)&v[0])[6] = ((uint8_t*)&sharedMemory1[0])[s[13]];
	((uint8_t*)&v[0])[9] = ((uint8_t*)&sharedMemory1[0])[s[14]];
	((uint8_t*)&v[0])[12] = ((uint8_t*)&sharedMemory1[0])[s[15]];

	t = v[0];
	u = v[0] ^ v[1] ^ v[2] ^ v[3];
	v[0] = v[0] ^ u ^ XT4(v[0] ^ v[1]);
	v[1] = v[1] ^ u ^ XT4(v[1] ^ v[2]);
	v[2] = v[2] ^ u ^ XT4(v[2] ^ v[3]);
	v[3] = v[3] ^ u ^ XT4(v[3] ^ t);

	s[0] = ((uint8_t*)&v[0])[0] ^ rk[0];
	s[1] = ((uint8_t*)&v[0])[4] ^ rk[1];
	s[2] = ((uint8_t*)&v[0])[8] ^ rk[2];
	s[3] = ((uint8_t*)&v[0])[12] ^ rk[3];
	s[4] = ((uint8_t*)&v[0])[1] ^ rk[4];
	s[5] = ((uint8_t*)&v[0])[5] ^ rk[5];
	
	s[6] = ((uint8_t*)&v[0])[9] ^ rk[6];
	s[7] = ((uint8_t*)&v[0])[13] ^ rk[7];
	s[8] = ((uint8_t*)&v[0])[2] ^ rk[8];
	s[9] = ((uint8_t*)&v[0])[6] ^ rk[9];
	s[10] = ((uint8_t*)&v[0])[10] ^ rk[10];
	s[11] = ((uint8_t*)&v[0])[14] ^ rk[11];
	s[12] = ((uint8_t*)&v[0])[3] ^ rk[12];
	s[13] = ((uint8_t*)&v[0])[7] ^ rk[13];
	s[14] = ((uint8_t*)&v[0])[11] ^ rk[14];
	s[15] = ((uint8_t*)&v[0])[15] ^ rk[15];

}

#define _mm_xor_si128_emu(a,b) (operator^(a,b))

#define _mm_load_si128_emu(p) (*(uint128m*)(p));

#define _mm_cvtsi128_si64_emu(p) (((int64_t *)&p)[0]);

#define _mm_cvtsi128_si32_emu(p) (((int32_t *)&a)[0]);

__device__  __forceinline__ uint128m _mm_cvtsi32_si128_emu(uint32_t lo)
{
	uint128m result;
	((uint32_t *)&result)[0] = lo;
	((uint32_t *)&result)[1] = 0;
	((uint64_t *)&result)[1] = 0;
	return result;
}
__device__  __forceinline__ uint128m _mm_cvtsi64_si128_emu(uint64_t lo)
{
	uint128m result;
	((uint64_t *)&result)[0] = lo;
	((uint64_t *)&result)[1] = 0;
	return result;
}
__device__  __forceinline__ uint128m _mm_set_epi64x_emu(uint64_t hi, uint64_t lo)
{
	uint128m result;
	((uint64_t *)&result)[0] = lo;
	((uint64_t *)&result)[1] = hi;
	return result;
}
__device__  __forceinline__ uint128m _mm_shuffle_epi8_emu(uint128m a, uint128m b)
{
	uint128m result;
	for (int i = 0; i < 16; i++)
	{
		if (((uint8_t *)&b)[i] & 0x80)
		{
			((uint8_t *)&result)[i] = 0;
		}
		else
		{
			((uint8_t *)&result)[i] = ((uint8_t *)&a)[((uint8_t *)&b)[i] & 0xf];
		}
	}

	return result;
}

__device__  __forceinline__ uint128m _mm_setr_epi8_emu(u_char c0, u_char c1, u_char c2, u_char c3, u_char c4, u_char c5, u_char c6, u_char c7, u_char c8, u_char c9, u_char c10, u_char c11, u_char c12, u_char c13, u_char c14, u_char c15)
{

		uint128m result;

	
		((uint32_t *)&result)[0] = 0x2d361b00;
		((uint32_t *)&result)[1] = 0x415a776c;
		((uint32_t *)&result)[2] = 0xf5eec3d8;
		((uint32_t *)&result)[3] = 0x9982afb4;

	return result;
}


__device__  __forceinline__ uint128m _mm_srli_si128_emu(uint128m input, int imm8)
{
	//we can cheat here as its an 8 byte shift just copy the 64bits
	uint128m temp;
	((uint64_t*)&temp)[0] = ((uint64_t*)&input)[1];
	((uint64_t*)&temp)[1] = 0;


	return temp;
}
__device__  __forceinline__ uint128m _mm_unpacklo_epi32_emu(uint128m a, uint128m b)
{
	uint32_t result[4];
	uint32_t *tmp1 = (uint32_t *)&a, *tmp2 = (uint32_t *)&b;
	result[0] = tmp1[0];
	result[1] = tmp2[0];
	result[2] = tmp1[1];
	result[3] = tmp2[1];
	return ((uint128m *)&result[0])[0];
}

__device__  __forceinline__ uint128m _mm_unpackhi_epi32_emu(uint128m a, uint128m b)
{
	uint32_t result[4];
	uint32_t *tmp1 = (uint32_t *)&a, *tmp2 = (uint32_t *)&b;
	result[0] = tmp1[2];
	result[1] = tmp2[2];
	result[2] = tmp1[3];
	result[3] = tmp2[3];
	return ((uint128m *)&result[0])[0];
}


__device__ __forceinline__ uint128m _mm_mulhrs_epi16_emu(uint128m _a, uint128m _b)
{
	int16_t result[8];
	int16_t *a = (int16_t*)&_a, *b = (int16_t*)&_b;
#pragma unroll 8
	for (int i = 0; i < 8; i++)
	{
		result[i] = (int16_t)((((int32_t)(a[i]) * (int32_t)(b[i])) + 0x4000) >> 15);
	}
	return *(uint128m *)result;
}


__device__ uint128m __verusclmulwithoutreduction64alignedrepeatgpu(uint128m *randomsource, const  uint128m buf [4], uint64_t keyMask, uint32_t *sharedMemory1)
{
    uint128m const *pbuf;
	keyMask >>= 4;
	uint128m acc = randomsource[keyMask + 2];
	
#ifdef GPU_DEBUGGY
	if (nounce == 0)
	{
		printf("[GPU]BUF ito verusclmulithout        : ");
		for (int i = 0; i < 64; i++)
			printf("%02x", ((uint8_t*)&buf[0])[i]);
		printf("\n");
		printf("[GPU]KEy ito verusclmulithout        : ");
		for (int e = 0; e < 64; e++)
		printf("%02x", ((uint8_t*)&randomsource[0])[e]);
	printf("\n");
	    printf("[GPU]ACC ito verusclmulithout        : ");
	for (int i = 0; i < 16; i++)
		printf("%02x", ((uint8_t*)&acc)[i]);
	printf("\n");
	}
#endif	
	// divide key mask by 32 from bytes to uint128m
	
	uint32_t prand_idx, prandex_idx;
	uint64_t selector;
	uint128m prand;
	uint128m prandex;

	for (int64_t i = 0; i < 32; i++)
	{
		
		selector = _mm_cvtsi128_si64_emu(acc);

		
		prand_idx = ((selector >> 5) & keyMask);
		prandex_idx = ((selector >> 32) & keyMask);
		// get two random locations in the key, which will be mutated and swapped
		
		prand = randomsource[prand_idx];
		prandex = randomsource[prandex_idx];

	//	save_rand[i] = ((selector >> 5) & keyMask);
	//	save_randex[i] = ((selector >> 32) & keyMask);

		// select random start and order of pbuf processing
		pbuf = buf + (selector & 3);
		uint64_t case_v;
		case_v = selector &  0x1cu;
#ifdef GPU_DEBUGu
		uint64_t egg, nog, salad;
		if (nounce == 0)
		{
			printf("[GPU]*****LOOP[%d]**********\n",i);
			egg = selector & 0x03u;
			nog = ((selector >> 32) & keyMask);
			salad = ((selector >> 5) & keyMask);
			printf("[GPU]selector: %llx\n case: %llx selector &3: ", selector, case_v);
			printf("%llx \n", egg);
			printf("[GPU]((selector >> 32) & keyMask) %d",nog);
			printf("[GPU]((selector >> 5) & keyMask) %d", salad);
			printf("\nacc     : ");
			printf("%016llx%016llx", ((uint64_t*)&acc)[0], ((uint64_t*)&acc)[1]);
			printf("\n");

			printf("[GPU]prand   : ");
			//for (int e = 0; e < 4; e++)
			printf("%016llx%016llx", ((uint64_t*)&prand)[0], ((uint64_t*)&prand)[1]);
			printf("\n");
			printf("[GPU]prandex : ");
			//for (int e = 0; e < 16; e++)
			printf("%016llx%016llx", ((uint64_t*)&prandex)[0], ((uint64_t*)&prandex)[1]);
			printf("\n");


		}

#endif
		
		if((case_v) == 0)
		{
		const uint128m temp1 = prandex;
	
			const uint128m temp2 = _mm_load_si128_emu(pbuf - (((selector & 1) << 1) - 1));
			

			const uint128m add1 = _mm_xor_si128_emu(temp1, temp2);

			const uint128m clprod1 = _mm_clmulepi64_si128_emu(add1, add1, 0x10);
			acc = _mm_xor_si128_emu(clprod1, acc);

			const uint128m tempa1 = _mm_mulhrs_epi16_emu(acc, temp1);
			const uint128m tempa2 = _mm_xor_si128_emu(tempa1, temp1);

			const uint128m temp12 = prand;
			prand = tempa2;


			const uint128m temp22 = _mm_load_si128_emu(pbuf);
			const uint128m add12 = _mm_xor_si128_emu(temp12, temp22);
			const uint128m clprod12 = _mm_clmulepi64_si128_emu(add12, add12, 0x10);
			acc = _mm_xor_si128_emu(clprod12, acc);

			const uint128m tempb1 = _mm_mulhrs_epi16_emu(acc, temp12);
			const uint128m tempb2 = _mm_xor_si128_emu(tempb1, temp12);
			prandex = tempb2;

		
			
		}
		if (case_v == 4)
		{
			const uint128m temp1 = prand;
			const uint128m temp2 = _mm_load_si128_emu(pbuf);
			const uint128m add1 = _mm_xor_si128_emu(temp1, temp2);
			const uint128m clprod1 = _mm_clmulepi64_si128_emu(add1, add1, 0x10);
			acc = _mm_xor_si128_emu(clprod1, acc);
			const uint128m clprod2 = _mm_clmulepi64_si128_emu(temp2, temp2, 0x10);
			acc = _mm_xor_si128_emu(clprod2, acc);

			const uint128m tempa1 = _mm_mulhrs_epi16_emu(acc, temp1);
			const uint128m tempa2 = _mm_xor_si128_emu(tempa1, temp1);

			const uint128m temp12 = prandex;
			prandex= tempa2;

			const uint128m temp22 = _mm_load_si128_emu(pbuf - (((selector & 1) << 1) - 1));
			const uint128m add12 = _mm_xor_si128_emu(temp12, temp22);
			acc = _mm_xor_si128_emu(add12, acc);

			const uint128m tempb1 = _mm_mulhrs_epi16_emu(acc, temp12);
			const uint128m tempb2 = _mm_xor_si128_emu(tempb1, temp12);
			prand= tempb2;
	
		}
		if (case_v == 8)
		{
			const uint128m temp1 = prandex;
			const uint128m temp2 = _mm_load_si128_emu(pbuf);
			const uint128m add1 = _mm_xor_si128_emu(temp1, temp2);
			acc = _mm_xor_si128_emu(add1, acc);

			const uint128m tempa1 = _mm_mulhrs_epi16_emu(acc, temp1);
			const uint128m tempa2 = _mm_xor_si128_emu(tempa1, temp1);

			const uint128m temp12 = prand;
			prand= tempa2;

			const uint128m temp22 = _mm_load_si128_emu(pbuf - (((selector & 1) << 1) - 1));
			const uint128m add12 = _mm_xor_si128_emu(temp12, temp22);
			const uint128m clprod12 = _mm_clmulepi64_si128_emu(add12, add12, 0x10);
			acc = _mm_xor_si128_emu(clprod12, acc);
			const uint128m clprod22 = _mm_clmulepi64_si128_emu(temp22, temp22, 0x10);
			acc = _mm_xor_si128_emu(clprod22, acc);

			const uint128m tempb1 = _mm_mulhrs_epi16_emu(acc, temp12);
			const uint128m tempb2 = _mm_xor_si128_emu(tempb1, temp12);
			prandex=tempb2;
			
		}
		if (case_v == 0xc)
		{
			const uint128m temp1 = prand;
			const uint128m temp2 = _mm_load_si128_emu(pbuf - (((selector & 1) << 1) - 1));
			const uint128m add1 = _mm_xor_si128_emu(temp1, temp2);

			// cannot be zero here
			const int32_t divisor = ((uint32_t*)&selector)[0];

			acc = _mm_xor_si128_emu(add1, acc);

			int64_t dividend = _mm_cvtsi128_si64_emu(acc);
			int64_t tmpmod = dividend % divisor;
			const uint128m modulo = _mm_cvtsi32_si128_emu(tmpmod);
			acc = _mm_xor_si128_emu(modulo, acc);

			const uint128m tempa1 = _mm_mulhrs_epi16_emu(acc, temp1);
			const uint128m tempa2 = _mm_xor_si128_emu(tempa1, temp1);
			dividend &= 1;
			if (dividend)
			{
				const uint128m temp12 = prandex;
				prandex = tempa2;

				const uint128m temp22 = _mm_load_si128_emu(pbuf);
				const uint128m add12 = _mm_xor_si128_emu(temp12, temp22);
				const uint128m clprod12 = _mm_clmulepi64_si128_emu(add12, add12, 0x10);
				acc = _mm_xor_si128_emu(clprod12, acc);
				const uint128m clprod22 = _mm_clmulepi64_si128_emu(temp22, temp22, 0x10);
				acc = _mm_xor_si128_emu(clprod22, acc);

				const uint128m tempb1 = _mm_mulhrs_epi16_emu(acc, temp12);
				const uint128m tempb2 = _mm_xor_si128_emu(tempb1, temp12);
				prand = tempb2;
			}
			else
			{
				const uint128m tempb3 = prandex;
				prandex = tempa2;
				prand = tempb3;
			}

		}
		if (case_v == 0x10)
		{
			// a few AES operations
			uint128m rc[12];
			
			rc[0] = prand; 

			rc[1] = randomsource[prand_idx + 1];
			rc[2] = randomsource[prand_idx + 2];
			rc[3] = randomsource[prand_idx + 3];
			rc[4] = randomsource[prand_idx + 4];
			rc[5] = randomsource[prand_idx + 5];
			rc[6] = randomsource[prand_idx + 6];
			rc[7] = randomsource[prand_idx + 7];
			rc[8] = randomsource[prand_idx + 8];
			rc[9] = randomsource[prand_idx + 9];
			rc[10] = randomsource[prand_idx + 10];
			rc[11] = randomsource[prand_idx + 11];
			uint128m tmp;
			const uint64_t rr = 0;
			uint128m temp1 = _mm_load_si128_emu(pbuf - (((selector & 1) << 1) - 1));
			uint128m temp2 = _mm_load_si128_emu(pbuf);
			
			AES2_EMU(temp1, temp2, 0);
			MIX2_EMU(temp1, temp2);


			AES2_EMU(temp1, temp2, 4);
			MIX2_EMU(temp1, temp2);

			AES2_EMU(temp1, temp2, 8);
			MIX2_EMU(temp1, temp2);


		    acc = _mm_xor_si128_emu(temp1, acc);
			acc = _mm_xor_si128_emu(temp2, acc);

			const uint128m tempa1 = prand;
			const uint128m tempa2 = _mm_mulhrs_epi16_emu(acc, tempa1);
			const uint128m tempa3 = _mm_xor_si128_emu(tempa1, tempa2);

			const uint128m tempa4 = prandex;
			prandex = tempa3;
			prand = tempa4;


		}
		if(case_v == 0x14)
		{
			// we'll just call this one the monkins loop, inspired by Chris
			const uint128m *buftmp = pbuf - (((selector & 1) << 1) - 1);
			uint128m tmp; // used by MIX2

			uint64_t rounds = selector >> 61; // loop randomly between 1 and 8 times
			uint128m *rc = &randomsource[prand_idx];


			uint64_t aesround = 0;
			uint128m onekey;
			uint64_t loop_c; 
		
			do
			{
				loop_c = selector & (0x10000000 << rounds);
				if (loop_c)
				{
					onekey = _mm_load_si128_emu(rc++);
					const uint128m temp2 = _mm_load_si128_emu(rounds & 1 ? pbuf : buftmp);
					const uint128m add1 = _mm_xor_si128_emu(onekey, temp2);
					const uint128m clprod1 = _mm_clmulepi64_si128_emu(add1, add1, 0x10);
					acc = _mm_xor_si128_emu(clprod1, acc);
				}
				else
				{
					onekey = _mm_load_si128_emu(rc++);
					uint128m temp2 = _mm_load_si128_emu(rounds & 1 ? buftmp : pbuf);
				
					const uint64_t roundidx = aesround++ << 2;
					AES2_EMU(onekey, temp2, roundidx);
				
					MIX2_EMU(onekey, temp2);
				
					acc = _mm_xor_si128_emu(onekey, acc);
					acc = _mm_xor_si128_emu(temp2, acc);

				}

			} while (rounds--);

			const uint128m tempa1 = (prand);
			const uint128m tempa2 = _mm_mulhrs_epi16_emu(acc, tempa1);
			const uint128m tempa3 = _mm_xor_si128_emu(tempa1, tempa2);

			const uint128m tempa4 = (prandex);
			prandex = tempa3;
			prand = tempa4;

		}
		if(case_v == 0x18)
		{
			const uint128m temp1 = _mm_load_si128_emu(pbuf - (((selector & 1) << 1) - 1));
			const uint128m temp2 = (prand);
			const uint128m add1 = _mm_xor_si128_emu(temp1, temp2);
			const uint128m clprod1 = _mm_clmulepi64_si128_emu(add1, add1, 0x10);
			acc = _mm_xor_si128_emu(clprod1, acc);

			const uint128m tempa1 = _mm_mulhrs_epi16_emu(acc, temp2);
			const uint128m tempa2 = _mm_xor_si128_emu(tempa1, temp2);

			const uint128m tempb3 = (prandex);
			prandex = tempa2;
			prand = tempb3;
			
		}
		if(case_v == 0x1c)
		{
			const uint128m temp1 = _mm_load_si128_emu(pbuf);
			const uint128m temp2 = (prandex);
			const uint128m add1 = _mm_xor_si128_emu(temp1, temp2);
			const uint128m clprod1 = _mm_clmulepi64_si128_emu(add1, add1, 0x10);
			acc = _mm_xor_si128_emu(clprod1, acc);


			const uint128m tempa1 = _mm_mulhrs_epi16_emu(acc, temp2);
			const uint128m tempa2 = _mm_xor_si128_emu(tempa1, temp2);
			const uint128m tempa3 = (prand);

			
			prand = tempa2;

			acc = _mm_xor_si128_emu(tempa3, acc);

			const uint128m tempb1 = _mm_mulhrs_epi16_emu(acc, tempa3);
			const uint128m tempb2 = _mm_xor_si128_emu(tempb1, tempa3);
			prandex = tempb2;

		}	

		 randomsource[prand_idx] = prand;
		 randomsource[prandex_idx] = prandex;

	}

	return acc;
}


__device__   __forceinline__ void haraka512_port_keyed2222(unsigned char *out, const unsigned char *in, uint128m *rc, uint32_t *sharedMemory1, uint32_t nonce)
{
	uint128m s[4], tmp;

	s[0] = ((uint128m*)&in[0])[0];
	s[1] = ((uint128m*)&in[0])[1];
	s[2] = ((uint128m*)&in[0])[2];
	s[3] = ((uint128m*)&in[0])[3];

	AES4(s[0], s[1], s[2], s[3], 0);
	MIX4(s[0], s[1], s[2], s[3]);

	AES4(s[0], s[1], s[2], s[3], 8);
	MIX4(s[0], s[1], s[2], s[3]);

	AES4(s[0], s[1], s[2], s[3], 16);
	MIX4(s[0], s[1], s[2], s[3]);

	AES4(s[0], s[1], s[2], s[3], 24);
	MIX4(s[0], s[1], s[2], s[3]);

	AES4(s[0], s[1], s[2], s[3], 32);
	MIX4(s[0], s[1], s[2], s[3]);

	//s[0] = _mm_xor_si128_emu(s[0], ((uint128m*)&in[0])[0]);
	//s[1] = _mm_xor_si128_emu(s[1], ((uint128m*)&in[0])[1]);
	//s[2] = _mm_xor_si128_emu(s[2], ((uint128m*)&in[0])[2]);
	s[3] = _mm_xor_si128_emu(s[3], ((uint128m*)&in[0])[3]);

	TRUNCSTORE(out, s[3]);

	//((uint32_t*)&out[0])[7] = ((uint32_t*)&s[52])[0] ^ ((uint32_t*)&in[52])[0];

}

__device__   __forceinline__ uint128m precompReduction64_si128(uint128m A) {

	//const uint128m C = _mm_set_epi64x(1U,(1U<<4)+(1U<<3)+(1U<<1)+(1U<<0)); // C is the irreducible poly. (64,4,3,1,0)
	//const uint128m C = _mm_cvtsi64_si128_emu(27U);
	uint128m M;
	M.x = 0x2d361b00;
	M.y = 0x415a776c;
	M.z = 0xf5eec3d8;
	M.w = 0x9982afb4;


	uint128m Q2 = _mm_clmulepi64_si128_emu(A, _mm_cvtsi64_si128_emu(27U), 0x01);
	uint128m Q3 = _mm_shuffle_epi8_emu(M,_mm_srli_si128_emu(Q2, 8));

	uint128m Q4 = _mm_xor_si128_emu(Q2, A);
	const uint128m final = _mm_xor_si128_emu(Q3, Q4);
	return final;/// WARNING: HIGH 64 BITS SHOULD BE ASSUMED TO CONTAIN GARBAGE
}

__device__  __forceinline__ uint64_t precompReduction64(uint128m A) {
	uint128m tmp = precompReduction64_si128(A);
	return _mm_cvtsi128_si64_emu(tmp);
}

__global__ __launch_bounds__(THREADS, 2)
void verus_gpu_hash(uint32_t threads, uint32_t startNonce, uint32_t *resNonce)
{
	uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	uint128m mid, biddy[VERUS_KEY_SIZE128];
	int i;
	uint8_t s[64];
	uint32_t nounce = startNonce + thread, hash[32] = { 0 };
	uint64_t acc;
	__shared__ uint32_t sharedMemory1[THREADS];

	
	//uint32_t save_rand[32] = { 0 };
	//uint32_t save_randex[32] = { 0 };
	
	memcpy(s, blockhash_half, 64);
	memcpy(s + 47, blockhash_half, 16);
	memcpy(s + 63, blockhash_half, 1);
//	if (blockIdx.x < 10)
	memcpy(biddy, vkey, VERUS_KEY_SIZE); // 2% speed increase


	sharedMemory1[threadIdx.x] = sbox[threadIdx.x];// copy sbox to shared mem
	

	((uint32_t *)&s)[8] = nounce;
	
	uint128m lazy;
	((uint64_t *)&lazy)[0] = 0x0000000000010000ull;
	((uint64_t *)&lazy)[1] = 0x0000000000000000ull;
	__syncthreads();
	mid = __verusclmulwithoutreduction64alignedrepeatgpu(biddy, (uint128m*)s, 8191, sharedMemory1);

	mid = _mm_xor_si128_emu(mid, lazy);

	
	acc = precompReduction64(mid);

	memcpy(s + 47, &acc, 8);
	memcpy(s + 55, &acc, 8);
	memcpy(s + 63, &acc, 1);
	uint64_t mask = 8191 >> 4;
	mask &= acc;
	
	//haraka512_port_keyed((unsigned char*)hash, (const unsigned char*)s, (const unsigned char*)(biddy + mask), sharedMemory1, nounce);

	haraka512_port_keyed2222((unsigned char*)hash, (const unsigned char*)s, (biddy + mask), sharedMemory1,nounce);

	if (hash[7] < ptarget[7]) { 
		
		resNonce[0] = nounce;

	//	printf("[GPU]Final hash    : ");//	for (int i = 0; i < 32; i++)//		printf("%02x", ((uint8_t*)&hash[0])[i]);
//printf("\n");
	}

	//__syncthreads();
};
