#include "hip/hip_runtime.h"
#include <miner.h>

#include <cuda_helper.h>

typedef uint4 uint128m;
#define GPU_DEBUG
#define VERUS_KEY_SIZE 8832
#define VERUS_KEY_SIZE128 552
#define THREADS 64
#define INNERLOOP 16

#define AES2_EMU(s0, s1, rci) \
  aesenc((unsigned char *)&s0, &rc[rci],sharedMemory1); \
  aesenc((unsigned char *)&s1, &rc[rci + 1],sharedMemory1); \
  aesenc((unsigned char *)&s0, &rc[rci + 2],sharedMemory1); \
  aesenc((unsigned char *)&s1, &rc[rci + 3],sharedMemory1);




#define AES4(s0, s1, s2, s3, rci) \
  aesenc((unsigned char *)&s0, &rc[rci],sharedMemory1); \
  aesenc((unsigned char *)&s1, &rc[rci + 1],sharedMemory1); \
  aesenc((unsigned char *)&s2, &rc[rci + 2],sharedMemory1); \
  aesenc((unsigned char *)&s3, &rc[rci + 3],sharedMemory1); \
  aesenc((unsigned char *)&s0, &rc[rci + 4], sharedMemory1); \
  aesenc((unsigned char *)&s1, &rc[rci + 5], sharedMemory1); \
  aesenc((unsigned char *)&s2, &rc[rci + 6], sharedMemory1); \
  aesenc((unsigned char *)&s3, &rc[rci + 7], sharedMemory1);

#define TRUNCSTORE(out, s3) \
  *(uint64_t*)(out + 24) = *(((uint64_t*)&s3 + 0));

#define MIX2_EMU(s0, s1) \
  tmp = _mm_unpacklo_epi32_emu(s0, s1); \
  s1 = _mm_unpackhi_epi32_emu(s0, s1); \
  s0 = tmp;

#define MIX4(s0, s1, s2, s3) \
  tmp  = _mm_unpacklo_epi32_emu(s0, s1); \
  s0 = _mm_unpackhi_epi32_emu(s0, s1); \
  s1 = _mm_unpacklo_epi32_emu(s2, s3); \
  s2 = _mm_unpackhi_epi32_emu(s2, s3); \
  s3 = _mm_unpacklo_epi32_emu(s0, s2); \
  s0 = _mm_unpackhi_epi32_emu(s0, s2); \
  s2 = _mm_unpackhi_epi32_emu(s1, tmp); \
  s1 = _mm_unpacklo_epi32_emu(s1, tmp);



__host__ void verus_setBlock(uint8_t *blockf, uint32_t *pTargetIn, uint8_t *lkey, int thr_id);


__device__ const uint32_t sbox[] = {
	0x7b777c63, 0xc56f6bf2, 0x2b670130, 0x76abd7fe, 0x7dc982ca, 0xf04759fa, 0xafa2d4ad, 0xc072a49c, 0x2693fdb7, 0xccf73f36, 0xf1e5a534, 0x1531d871, 0xc323c704, 0x9a059618, 0xe2801207, 0x75b227eb, 0x1a2c8309, 0xa05a6e1b, 0xb3d63b52, 0x842fe329, 0xed00d153, 0x5bb1fc20, 0x39becb6a, 0xcf584c4a, 0xfbaaefd0, 0x85334d43, 0x7f02f945, 0xa89f3c50, 0x8f40a351, 0xf5389d92, 0x21dab6bc, 0xd2f3ff10, 0xec130ccd, 0x1744975f, 0x3d7ea7c4, 0x73195d64, 0xdc4f8160, 0x88902a22, 0x14b8ee46, 0xdb0b5ede, 0x0a3a32e0, 0x5c240649, 0x62acd3c2, 0x79e49591, 0x6d37c8e7, 0xa94ed58d, 0xeaf4566c, 0x08ae7a65, 0x2e2578ba, 0xc6b4a61c, 0x1f74dde8, 0x8a8bbd4b, 0x66b53e70, 0x0ef60348, 0xb9573561, 0x9e1dc186, 0x1198f8e1, 0x948ed969, 0xe9871e9b, 0xdf2855ce, 0x0d89a18c, 0x6842e6bf, 0x0f2d9941, 0x16bb54b0,
	0x7b777c63, 0xc56f6bf2, 0x2b670130, 0x76abd7fe, 0x7dc982ca, 0xf04759fa, 0xafa2d4ad, 0xc072a49c, 0x2693fdb7, 0xccf73f36, 0xf1e5a534, 0x1531d871, 0xc323c704, 0x9a059618, 0xe2801207, 0x75b227eb, 0x1a2c8309, 0xa05a6e1b, 0xb3d63b52, 0x842fe329, 0xed00d153, 0x5bb1fc20, 0x39becb6a, 0xcf584c4a, 0xfbaaefd0, 0x85334d43, 0x7f02f945, 0xa89f3c50, 0x8f40a351, 0xf5389d92, 0x21dab6bc, 0xd2f3ff10, 0xec130ccd, 0x1744975f, 0x3d7ea7c4, 0x73195d64, 0xdc4f8160, 0x88902a22, 0x14b8ee46, 0xdb0b5ede, 0x0a3a32e0, 0x5c240649, 0x62acd3c2, 0x79e49591, 0x6d37c8e7, 0xa94ed58d, 0xeaf4566c, 0x08ae7a65, 0x2e2578ba, 0xc6b4a61c, 0x1f74dde8, 0x8a8bbd4b, 0x66b53e70, 0x0ef60348, 0xb9573561, 0x9e1dc186, 0x1198f8e1, 0x948ed969, 0xe9871e9b, 0xdf2855ce, 0x0d89a18c, 0x6842e6bf, 0x0f2d9941, 0x16bb54b0,
	0x7b777c63, 0xc56f6bf2, 0x2b670130, 0x76abd7fe, 0x7dc982ca, 0xf04759fa, 0xafa2d4ad, 0xc072a49c, 0x2693fdb7, 0xccf73f36, 0xf1e5a534, 0x1531d871, 0xc323c704, 0x9a059618, 0xe2801207, 0x75b227eb, 0x1a2c8309, 0xa05a6e1b, 0xb3d63b52, 0x842fe329, 0xed00d153, 0x5bb1fc20, 0x39becb6a, 0xcf584c4a, 0xfbaaefd0, 0x85334d43, 0x7f02f945, 0xa89f3c50, 0x8f40a351, 0xf5389d92, 0x21dab6bc, 0xd2f3ff10, 0xec130ccd, 0x1744975f, 0x3d7ea7c4, 0x73195d64, 0xdc4f8160, 0x88902a22, 0x14b8ee46, 0xdb0b5ede, 0x0a3a32e0, 0x5c240649, 0x62acd3c2, 0x79e49591, 0x6d37c8e7, 0xa94ed58d, 0xeaf4566c, 0x08ae7a65, 0x2e2578ba, 0xc6b4a61c, 0x1f74dde8, 0x8a8bbd4b, 0x66b53e70, 0x0ef60348, 0xb9573561, 0x9e1dc186, 0x1198f8e1, 0x948ed969, 0xe9871e9b, 0xdf2855ce, 0x0d89a18c, 0x6842e6bf, 0x0f2d9941, 0x16bb54b0,
	0x7b777c63, 0xc56f6bf2, 0x2b670130, 0x76abd7fe, 0x7dc982ca, 0xf04759fa, 0xafa2d4ad, 0xc072a49c, 0x2693fdb7, 0xccf73f36, 0xf1e5a534, 0x1531d871, 0xc323c704, 0x9a059618, 0xe2801207, 0x75b227eb, 0x1a2c8309, 0xa05a6e1b, 0xb3d63b52, 0x842fe329, 0xed00d153, 0x5bb1fc20, 0x39becb6a, 0xcf584c4a, 0xfbaaefd0, 0x85334d43, 0x7f02f945, 0xa89f3c50, 0x8f40a351, 0xf5389d92, 0x21dab6bc, 0xd2f3ff10, 0xec130ccd, 0x1744975f, 0x3d7ea7c4, 0x73195d64, 0xdc4f8160, 0x88902a22, 0x14b8ee46, 0xdb0b5ede, 0x0a3a32e0, 0x5c240649, 0x62acd3c2, 0x79e49591, 0x6d37c8e7, 0xa94ed58d, 0xeaf4566c, 0x08ae7a65, 0x2e2578ba, 0xc6b4a61c, 0x1f74dde8, 0x8a8bbd4b, 0x66b53e70, 0x0ef60348, 0xb9573561, 0x9e1dc186, 0x1198f8e1, 0x948ed969, 0xe9871e9b, 0xdf2855ce, 0x0d89a18c, 0x6842e6bf, 0x0f2d9941, 0x16bb54b0
};

//#define XT(x) (((x) << 1) ^ (((x) >> 7) ? 0x1b : 0))

__global__ void verus_gpu_hash(uint32_t threads, uint32_t startNonce, uint32_t *resNonce, uint128m * d_key_input, uint128m * d_mid, uint32_t *d_fix_r, uint32_t *d_fix_rex);
__global__ void verus_gpu_final(uint32_t threads, uint32_t startNonce, uint32_t *resNonce, uint128m * d_key_input, const  uint128m * d_mid);
__global__ void verus_extra_gpu_prepare(const uint32_t threads, uint128m * d_key_input);
__global__ void verus_extra_gpu_fix(const uint32_t threads, uint128m * d_key_input, uint32_t *d_fix_r, uint32_t *d_fix_rex);


static uint32_t *d_nonces[MAX_GPUS];
static uint32_t *d_fix_rand[MAX_GPUS];
static uint32_t *d_fix_randex[MAX_GPUS];
static uint4 *d_long_keys[MAX_GPUS];

static uint4 *d_mid[MAX_GPUS];

static hipStream_t streams[MAX_GPUS];
static uint8_t run[MAX_GPUS];

__device__ __constant__ uint128m vkey[VERUS_KEY_SIZE128];
__device__ __constant__ uint8_t blockhash_half[64];
__device__ __constant__ uint32_t ptarget[8];

__host__
void verus_init(int thr_id, uint32_t throughput)
{
	CUDA_SAFE_CALL(hipMalloc(&d_nonces[thr_id], 1 * sizeof(uint32_t)));

	CUDA_SAFE_CALL(hipMalloc(&d_long_keys[thr_id], throughput * VERUS_KEY_SIZE));
	CUDA_SAFE_CALL(hipMalloc(&d_mid[thr_id], throughput * 16));
	CUDA_SAFE_CALL(hipMalloc(&d_fix_rand[thr_id], throughput * sizeof(uint32_t) * 32));
	CUDA_SAFE_CALL(hipMalloc(&d_fix_randex[thr_id], throughput * sizeof(uint32_t) * 32));

};

__host__
void verus_setBlock(uint8_t *blockf, uint32_t *pTargetIn, uint8_t *lkey, int thr_id, uint32_t throughput)
{
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(ptarget), (void**)pTargetIn, 8 * sizeof(uint32_t), 0, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(blockhash_half), (void**)blockf, 64 * sizeof(uint8_t), 0, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(vkey),(void**)lkey, VERUS_KEY_SIZE * sizeof(uint8_t), 0, hipMemcpyHostToDevice));
	dim3 grid2(throughput);
	verus_extra_gpu_prepare << <grid2, 128 >> > (0, d_long_keys[thr_id]); //setup global mem with lots of keys	

};
__host__
void verus_hash(int thr_id, uint32_t threads, uint32_t startNonce, uint32_t *resNonces)
{
	hipMemset(d_nonces[thr_id], 0xff, 1 * sizeof(uint32_t));
	const uint32_t threadsperblock = THREADS;

	dim3 grid((threads + threadsperblock - 1) / threadsperblock);
	dim3 grid2(threads);
	dim3 block(threadsperblock);


	if (run[thr_id] == 0) {
		hipStreamCreate(&streams[thr_id]);
		run[thr_id] = 1;
	}
//	verus_extra_gpu_prepare << <grid2, 128 >> > (0, d_long_keys[thr_id]); //setup global mem with lots of keys	
	verus_gpu_hash << <grid, block, 0, streams[thr_id] >> >(threads, startNonce, d_nonces[thr_id], d_long_keys[thr_id], d_mid[thr_id], d_fix_rand[thr_id], d_fix_randex[thr_id]);
	verus_gpu_final << <grid, block, 0, streams[thr_id] >> >(threads, startNonce, d_nonces[thr_id], d_long_keys[thr_id], d_mid[thr_id]);
	verus_extra_gpu_fix << <grid2, 32, 0, streams[thr_id] >> > (0, d_long_keys[thr_id], d_fix_rand[thr_id], d_fix_randex[thr_id]); //setup global mem with lots of keys	
	CUDA_SAFE_CALL(hipMemcpy(resNonces, d_nonces[thr_id], 1 * sizeof(uint32_t), hipMemcpyDeviceToHost));

};

//#define XT4(x) ((((x) << 1) & 0xfefefefe) ^ ((((x) >> 7) & 0x1010101) * 0x1b))

__device__  __forceinline__  uint32_t XT4(uint32_t b)
{
	uint32_t tmp1,tmp2,tmp3;
	
	tmp1 = (b << 1) & 0xfefefefe;
	tmp2 = (b >> 7) & 0x1010101;
	asm("mul24.lo.u32 %0, %1, %2; ": "=r"(tmp3) : "r"(tmp2), "r"(0x1b));
	asm("xor.b32 %0, %1, %2; // xor1" : "=r"(tmp2) : "r"(tmp1), "r"(tmp3));
	
	return tmp2;
}

__device__ uint128m _mm_clmulepi64_si128_emu(uint128m ai, uint128m bi, int imm)
{
	uint64_t a = *((uint64_t*)&ai + (imm & 1));

	uint64_t b = *((uint64_t*)&bi + ((imm & 0x10) >> 4));
	
	uint8_t  i; //window size s = 4,
	//uint64_t two_s = 16; //2^s
	//uint64_t smask = 15; //s 15
	uint64_t u[16];
	uint64_t r[2];
	uint64_t tmp;
	uint64_t ifmask;
	//Precomputation
	u[0] = 0;
	u[1] = b;
#pragma unroll
	for (i = 2; i < 16; i += 2) {
		u[i] = u[i >> 1] << 1; //even indices: left shift
		u[i + 1] = u[i] ^ b; //odd indices: xor b
	}
	//Multiply
	r[0] = u[a & 15]; //first window only affects lower word
	r[1] = 0;
#pragma unroll
	for (i = 4; i < 64; i += 4) {
		tmp = u[a >> i & 15];
		r[0] ^= tmp << i;
		r[1] ^= tmp >> (64 - i);
	}
	//Repair
	uint64_t m = 0xEEEEEEEEEEEEEEEE; //s=4 => 16 times 1110
#pragma unroll
	for (i = 1; i < 4; i++) {
		tmp = ((a & m) >> i);
		m &= m << 1; //shift mask to exclude all bit j': j' mod s = i
		ifmask = -((b >> (64 - i)) & 1); //if the (64-i)th bit of b is 1
		r[1] ^= (tmp & ifmask);
	}
	uint128m out;
	((uint64_t*)&out)[0] = r[0];
	((uint64_t*)&out)[1] = r[1];
	return out;
}



__device__  __forceinline__  uint128m _mm_xor_si128_emu(uint128m a, uint128m b)
{
	uint128m result;
	asm("xor.b32 %0, %1, %2; // xor1" : "=r"(result.x) : "r"(a.x), "r"(b.x));
	asm("xor.b32 %0, %1, %2; // xor1" : "=r"(result.y) : "r"(a.y), "r"(b.y));
	asm("xor.b32 %0, %1, %2; // xor1" : "=r"(result.z) : "r"(a.z), "r"(b.z));
	asm("xor.b32 %0, %1, %2; // xor1" : "=r"(result.w) : "r"(a.w), "r"(b.w));
	return result;


}
#define _mm_load_si128_emu(p) (*(uint128m*)(p));

#define _mm_cvtsi128_si64_emu(p) (((int64_t *)&p)[0]);

#define _mm_cvtsi128_si32_emu(p) (((int32_t *)&a)[0]);


__device__  void _mm_unpackboth_epi32_emu(uint128m &a, uint128m &b)
{
	uint64_t value;

	asm("mov.b64 %0, {%1, %2}; ": "=l"(value) : "r"(a.z), "r"(a.y));
	asm("mov.b64 {%0, %1}, %2; ": "=r"(a.y), "=r"(a.z) : "l"(value));

	asm("mov.b64 %0, {%1, %2}; ": "=l"(value) : "r"(b.x), "r"(a.y));
	asm("mov.b64 {%0, %1}, %2; ": "=r"(a.y), "=r"(b.x) : "l"(value));

	asm("mov.b64 %0, {%1, %2}; ": "=l"(value) : "r"(b.z), "r"(a.w));
	asm("mov.b64 {%0, %1}, %2; ": "=r"(a.w), "=r"(b.z) : "l"(value));
	
	asm("mov.b64 %0, {%1, %2}; ": "=l"(value) : "r"(b.y), "r"(a.w));
	asm("mov.b64 {%0, %1}, %2; ": "=r"(a.w), "=r"(b.y) : "l"(value));
}


__device__  __forceinline__ uint128m _mm_unpacklo_epi32_emu(uint128m a, uint128m b)
{

	uint4 t;
	t.x = a.x;
	t.y = b.x;
	t.z = a.y;
	t.w = b.y;
	return t;
}

__device__  __forceinline__ uint128m _mm_unpackhi_epi32_emu(uint128m a, uint128m b)
{

	uint4 t;
	t.x = a.z;
	t.y = b.z;
	t.z = a.w;
	t.w = b.w;

	return t;
}
__device__   __forceinline__ void aesenc(unsigned char * __restrict__ s, const uint128m * __restrict__ rk, uint32_t * __restrict__ sharedMemory1)
{
//#define XT(x) (((x) << 1) ^ (((x) >> 7) ? 0x1b : 0))

//#define XT4(x) ((((x) << 1) & 0xfefefefe) ^ ((((x) >> 31) & 1) ? 0x1b000000 : 0)^ ((((x) >> 23)&1) ? 0x001b0000 : 0)^ ((((x) >> 15)&1) ? 0x00001b00 : 0)^ ((((x) >> 7)&1) ? 0x0000001b : 0))


	//const uint32_t  t, u, w;
	uint32_t __align__(16) v[4];
	//const uint128m rk2 = ((uint128m*)&rk[0])[0];

	((uint8_t*)&v[0])[0] = ((uint8_t*)&sharedMemory1[0])[s[0]];
	((uint8_t*)&v[0])[7] = ((uint8_t*)&sharedMemory1[0])[s[1]];
	((uint8_t*)&v[0])[10] = ((uint8_t*)&sharedMemory1[0])[s[2]];
	((uint8_t*)&v[0])[13] = ((uint8_t*)&sharedMemory1[0])[s[3]];
	((uint8_t*)&v[0])[1] = ((uint8_t*)&sharedMemory1[0])[s[4]];
	((uint8_t*)&v[0])[4] = ((uint8_t*)&sharedMemory1[0])[s[5]];
	((uint8_t*)&v[0])[11] = ((uint8_t*)&sharedMemory1[0])[s[6]];
	((uint8_t*)&v[0])[14] = ((uint8_t*)&sharedMemory1[0])[s[7]];
	((uint8_t*)&v[0])[2] = ((uint8_t*)&sharedMemory1[0])[s[8]];
	((uint8_t*)&v[0])[5] = ((uint8_t*)&sharedMemory1[0])[s[9]];
	((uint8_t*)&v[0])[8] = ((uint8_t*)&sharedMemory1[0])[s[10]];
	((uint8_t*)&v[0])[15] = ((uint8_t*)&sharedMemory1[0])[s[11]];
	((uint8_t*)&v[0])[3] = ((uint8_t*)&sharedMemory1[0])[s[12]];
	((uint8_t*)&v[0])[6] = ((uint8_t*)&sharedMemory1[0])[s[13]];
	((uint8_t*)&v[0])[9] = ((uint8_t*)&sharedMemory1[0])[s[14]];
	((uint8_t*)&v[0])[12] = ((uint8_t*)&sharedMemory1[0])[s[15]];

	const uint32_t t = v[0];
	const uint32_t w = v[0] ^ v[1];
	const uint32_t u = w ^ v[2] ^ v[3];
	v[0] = v[0] ^ u ^ XT4(w);
	v[1] = v[1] ^ u ^ XT4(v[1] ^ v[2]);
	v[2] = v[2] ^ u ^ XT4(v[2] ^ v[3]);
	v[3] = v[3] ^ u ^ XT4(v[3] ^ t);

/*	s[0] = ((uint8_t*)&v[0])[0] ^ rk[0];
	s[1] = ((uint8_t*)&v[0])[4] ^ rk[1];F
	s[2] = ((uint8_t*)&v[0])[8] ^ rk[2];
	s[3] = ((uint8_t*)&v[0])[12] ^ rk[3];
	s[4] = ((uint8_t*)&v[0])[1] ^ rk[4];
	s[5] = ((uint8_t*)&v[0])[5] ^ rk[5];

	s[6] = ((uint8_t*)&v[0])[9] ^ rk[6];
	s[7] = ((uint8_t*)&v[0])[13] ^ rk[7];
	s[8] = ((uint8_t*)&v[0])[2] ^ rk[8];
	s[9] = ((uint8_t*)&v[0])[6] ^ rk[9];
	s[10] = ((uint8_t*)&v[0])[10] ^ rk[10];
	s[11] = ((uint8_t*)&v[0])[14] ^ rk[11];
	s[12] = ((uint8_t*)&v[0])[3] ^ rk[12];
	s[13] = ((uint8_t*)&v[0])[7] ^ rk[13];
	s[14] = ((uint8_t*)&v[0])[11] ^ rk[14];
	s[15] = ((uint8_t*)&v[0])[15] ^ rk[15];*/

	s[0] = ((uint8_t*)&v[0])[0];
	s[1] = ((uint8_t*)&v[0])[4] ;
	s[2] = ((uint8_t*)&v[0])[8] ;
	s[3] = ((uint8_t*)&v[0])[12] ;
	s[4] = ((uint8_t*)&v[0])[1] ;
	s[5] = ((uint8_t*)&v[0])[5] ;

	s[6] = ((uint8_t*)&v[0])[9] ;
	s[7] = ((uint8_t*)&v[0])[13] ;
	s[8] = ((uint8_t*)&v[0])[2] ;
	s[9] = ((uint8_t*)&v[0])[6] ;
	s[10] = ((uint8_t*)&v[0])[10] ;
	s[11] = ((uint8_t*)&v[0])[14] ;
	s[12] = ((uint8_t*)&v[0])[3] ;
	s[13] = ((uint8_t*)&v[0])[7];
	s[14] = ((uint8_t*)&v[0])[11];
	s[15] = ((uint8_t*)&v[0])[15];

	((uint128m*)&s[0])[0] = make_uint4(((uint32_t*)&s[0])[0] ^ rk[0].x, ((uint32_t*)&s[0])[1] ^ rk[0].y, ((uint32_t*)&s[0])[2] ^ rk[0].z, ((uint32_t*)&s[0])[3] ^ rk[0].w);


}
#define AES2_EMU2(s0, s1, rci) \
  aesenc4((unsigned char *)&s0, (unsigned char *)&s1, &rc[rci],sharedMemory1); 

__device__   __forceinline__ void aesenc4(unsigned char * __restrict__ s1, unsigned char * __restrict__ s2, const uint128m * __restrict__ rk, uint32_t * __restrict__ sharedMemory1)
{

//#define XT4(x) ((((x) << 1) & 0xfefefefe) ^ ((((x) >> 31) & 1) ? 0x1b000000 : 0)^ ((((x) >> 23)&1) ? 0x001b0000 : 0)^ ((((x) >> 15)&1) ? 0x00001b00 : 0)^ ((((x) >> 7)&1) ? 0x0000001b : 0))

	//const uint32_t  t, u, w;
	uint32_t v[4];
	uint32_t t, w, u;

	//const uint128m rk2 = ((uint128m*)&rk[0])[0];

	((uint8_t*)&v[0])[0] = ((uint8_t*)&sharedMemory1[0])[s1[0]];
	((uint8_t*)&v[0])[7] = ((uint8_t*)&sharedMemory1[0])[s1[1]];
	((uint8_t*)&v[0])[10] = ((uint8_t*)&sharedMemory1[0])[s1[2]];
	((uint8_t*)&v[0])[13] = ((uint8_t*)&sharedMemory1[0])[s1[3]];
	((uint8_t*)&v[0])[1] = ((uint8_t*)&sharedMemory1[0])[s1[4]];
	((uint8_t*)&v[0])[4] = ((uint8_t*)&sharedMemory1[0])[s1[5]];
	((uint8_t*)&v[0])[11] = ((uint8_t*)&sharedMemory1[0])[s1[6]];
	((uint8_t*)&v[0])[14] = ((uint8_t*)&sharedMemory1[0])[s1[7]];
	((uint8_t*)&v[0])[2] = ((uint8_t*)&sharedMemory1[0])[s1[8]];
	((uint8_t*)&v[0])[5] = ((uint8_t*)&sharedMemory1[0])[s1[9]];
	((uint8_t*)&v[0])[8] = ((uint8_t*)&sharedMemory1[0])[s1[10]];
	((uint8_t*)&v[0])[15] = ((uint8_t*)&sharedMemory1[0])[s1[11]];
	((uint8_t*)&v[0])[3] = ((uint8_t*)&sharedMemory1[0])[s1[12]];
	((uint8_t*)&v[0])[6] = ((uint8_t*)&sharedMemory1[0])[s1[13]];
	((uint8_t*)&v[0])[9] = ((uint8_t*)&sharedMemory1[0])[s1[14]];
	((uint8_t*)&v[0])[12] = ((uint8_t*)&sharedMemory1[0])[s1[15]];

	t = v[0];
	w = v[0] ^ v[1];
	u = w ^ v[2] ^ v[3];
	v[0] = v[0] ^ u ^ XT4(w);
	v[1] = v[1] ^ u ^ XT4(v[1] ^ v[2]);
	v[2] = v[2] ^ u ^ XT4(v[2] ^ v[3]);
	v[3] = v[3] ^ u ^ XT4(v[3] ^ t);


	s1[0] = ((uint8_t*)&v[0])[0];
	s1[1] = ((uint8_t*)&v[0])[4];
	s1[2] = ((uint8_t*)&v[0])[8];
	s1[3] = ((uint8_t*)&v[0])[12];
	s1[4] = ((uint8_t*)&v[0])[1];
	s1[5] = ((uint8_t*)&v[0])[5];

	s1[6] = ((uint8_t*)&v[0])[9];
	s1[7] = ((uint8_t*)&v[0])[13];
	s1[8] = ((uint8_t*)&v[0])[2];
	s1[9] = ((uint8_t*)&v[0])[6];
	s1[10] = ((uint8_t*)&v[0])[10];
	s1[11] = ((uint8_t*)&v[0])[14];
	s1[12] = ((uint8_t*)&v[0])[3];
	s1[13] = ((uint8_t*)&v[0])[7];
	s1[14] = ((uint8_t*)&v[0])[11];
	s1[15] = ((uint8_t*)&v[0])[15];

	((uint128m*)&s1[0])[0] = make_uint4(((uint32_t*)&s1[0])[0] ^ rk[0].x, ((uint32_t*)&s1[0])[1] ^ rk[0].y, ((uint32_t*)&s1[0])[2] ^ rk[0].z, ((uint32_t*)&s1[0])[3] ^ rk[0].w);

	((uint8_t*)&v[0])[0] = ((uint8_t*)&sharedMemory1[0])[s2[0]];
	((uint8_t*)&v[0])[7] = ((uint8_t*)&sharedMemory1[0])[s2[1]];
	((uint8_t*)&v[0])[10] = ((uint8_t*)&sharedMemory1[0])[s2[2]];
	((uint8_t*)&v[0])[13] = ((uint8_t*)&sharedMemory1[0])[s2[3]];
	((uint8_t*)&v[0])[1] = ((uint8_t*)&sharedMemory1[0])[s2[4]];
	((uint8_t*)&v[0])[4] = ((uint8_t*)&sharedMemory1[0])[s2[5]];
	((uint8_t*)&v[0])[11] = ((uint8_t*)&sharedMemory1[0])[s2[6]];
	((uint8_t*)&v[0])[14] = ((uint8_t*)&sharedMemory1[0])[s2[7]];
	((uint8_t*)&v[0])[2] = ((uint8_t*)&sharedMemory1[0])[s2[8]];
	((uint8_t*)&v[0])[5] = ((uint8_t*)&sharedMemory1[0])[s2[9]];
	((uint8_t*)&v[0])[8] = ((uint8_t*)&sharedMemory1[0])[s2[10]];
	((uint8_t*)&v[0])[15] = ((uint8_t*)&sharedMemory1[0])[s2[11]];
	((uint8_t*)&v[0])[3] = ((uint8_t*)&sharedMemory1[0])[s2[12]];
	((uint8_t*)&v[0])[6] = ((uint8_t*)&sharedMemory1[0])[s2[13]];
	((uint8_t*)&v[0])[9] = ((uint8_t*)&sharedMemory1[0])[s2[14]];
	((uint8_t*)&v[0])[12] = ((uint8_t*)&sharedMemory1[0])[s2[15]];

	t = v[0];
	w = v[0] ^ v[1];
	u = w ^ v[2] ^ v[3];
	v[0] = v[0] ^ u ^ XT4(w);
	v[1] = v[1] ^ u ^ XT4(v[1] ^ v[2]);
	v[2] = v[2] ^ u ^ XT4(v[2] ^ v[3]);
	v[3] = v[3] ^ u ^ XT4(v[3] ^ t);


	s2[0] = ((uint8_t*)&v[0])[0];
	s2[1] = ((uint8_t*)&v[0])[4];
	s2[2] = ((uint8_t*)&v[0])[8];
	s2[3] = ((uint8_t*)&v[0])[12];
	s2[4] = ((uint8_t*)&v[0])[1];
	s2[5] = ((uint8_t*)&v[0])[5];

	s2[6] = ((uint8_t*)&v[0])[9];
	s2[7] = ((uint8_t*)&v[0])[13];
	s2[8] = ((uint8_t*)&v[0])[2];
	s2[9] = ((uint8_t*)&v[0])[6];
	s2[10] = ((uint8_t*)&v[0])[10];
	s2[11] = ((uint8_t*)&v[0])[14];
	s2[12] = ((uint8_t*)&v[0])[3];
	s2[13] = ((uint8_t*)&v[0])[7];
	s2[14] = ((uint8_t*)&v[0])[11];
	s2[15] = ((uint8_t*)&v[0])[15];

	((uint128m*)&s2[0])[0] = make_uint4(((uint32_t*)&s2[0])[0] ^ rk[1].x, ((uint32_t*)&s2[0])[1] ^ rk[1].y, ((uint32_t*)&s2[0])[2] ^ rk[1].z, ((uint32_t*)&s2[0])[3] ^ rk[1].w);


	((uint8_t*)&v[0])[0] = ((uint8_t*)&sharedMemory1[0])[s1[0]];
	((uint8_t*)&v[0])[7] = ((uint8_t*)&sharedMemory1[0])[s1[1]];
	((uint8_t*)&v[0])[10] = ((uint8_t*)&sharedMemory1[0])[s1[2]];
	((uint8_t*)&v[0])[13] = ((uint8_t*)&sharedMemory1[0])[s1[3]];
	((uint8_t*)&v[0])[1] = ((uint8_t*)&sharedMemory1[0])[s1[4]];
	((uint8_t*)&v[0])[4] = ((uint8_t*)&sharedMemory1[0])[s1[5]];
	((uint8_t*)&v[0])[11] = ((uint8_t*)&sharedMemory1[0])[s1[6]];
	((uint8_t*)&v[0])[14] = ((uint8_t*)&sharedMemory1[0])[s1[7]];
	((uint8_t*)&v[0])[2] = ((uint8_t*)&sharedMemory1[0])[s1[8]];
	((uint8_t*)&v[0])[5] = ((uint8_t*)&sharedMemory1[0])[s1[9]];
	((uint8_t*)&v[0])[8] = ((uint8_t*)&sharedMemory1[0])[s1[10]];
	((uint8_t*)&v[0])[15] = ((uint8_t*)&sharedMemory1[0])[s1[11]];
	((uint8_t*)&v[0])[3] = ((uint8_t*)&sharedMemory1[0])[s1[12]];
	((uint8_t*)&v[0])[6] = ((uint8_t*)&sharedMemory1[0])[s1[13]];
	((uint8_t*)&v[0])[9] = ((uint8_t*)&sharedMemory1[0])[s1[14]];
	((uint8_t*)&v[0])[12] = ((uint8_t*)&sharedMemory1[0])[s1[15]];

	t = v[0];
	w = v[0] ^ v[1];
	u = w ^ v[2] ^ v[3];
	v[0] = v[0] ^ u ^ XT4(w);
	v[1] = v[1] ^ u ^ XT4(v[1] ^ v[2]);
	v[2] = v[2] ^ u ^ XT4(v[2] ^ v[3]);
	v[3] = v[3] ^ u ^ XT4(v[3] ^ t);


	s1[0] = ((uint8_t*)&v[0])[0];
	s1[1] = ((uint8_t*)&v[0])[4];
	s1[2] = ((uint8_t*)&v[0])[8];
	s1[3] = ((uint8_t*)&v[0])[12];
	s1[4] = ((uint8_t*)&v[0])[1];
	s1[5] = ((uint8_t*)&v[0])[5];

	s1[6] = ((uint8_t*)&v[0])[9];
	s1[7] = ((uint8_t*)&v[0])[13];
	s1[8] = ((uint8_t*)&v[0])[2];
	s1[9] = ((uint8_t*)&v[0])[6];
	s1[10] = ((uint8_t*)&v[0])[10];
	s1[11] = ((uint8_t*)&v[0])[14];
	s1[12] = ((uint8_t*)&v[0])[3];
	s1[13] = ((uint8_t*)&v[0])[7];
	s1[14] = ((uint8_t*)&v[0])[11];
	s1[15] = ((uint8_t*)&v[0])[15];

	((uint128m*)&s1[0])[0] = make_uint4(((uint32_t*)&s1[0])[0] ^ rk[2].x, ((uint32_t*)&s1[0])[1] ^ rk[2].y, ((uint32_t*)&s1[0])[2] ^ rk[2].z, ((uint32_t*)&s1[0])[3] ^ rk[2].w);

	((uint8_t*)&v[0])[0] = ((uint8_t*)&sharedMemory1[0])[s2[0]];
	((uint8_t*)&v[0])[7] = ((uint8_t*)&sharedMemory1[0])[s2[1]];
	((uint8_t*)&v[0])[10] = ((uint8_t*)&sharedMemory1[0])[s2[2]];
	((uint8_t*)&v[0])[13] = ((uint8_t*)&sharedMemory1[0])[s2[3]];
	((uint8_t*)&v[0])[1] = ((uint8_t*)&sharedMemory1[0])[s2[4]];
	((uint8_t*)&v[0])[4] = ((uint8_t*)&sharedMemory1[0])[s2[5]];
	((uint8_t*)&v[0])[11] = ((uint8_t*)&sharedMemory1[0])[s2[6]];
	((uint8_t*)&v[0])[14] = ((uint8_t*)&sharedMemory1[0])[s2[7]];
	((uint8_t*)&v[0])[2] = ((uint8_t*)&sharedMemory1[0])[s2[8]];
	((uint8_t*)&v[0])[5] = ((uint8_t*)&sharedMemory1[0])[s2[9]];
	((uint8_t*)&v[0])[8] = ((uint8_t*)&sharedMemory1[0])[s2[10]];
	((uint8_t*)&v[0])[15] = ((uint8_t*)&sharedMemory1[0])[s2[11]];
	((uint8_t*)&v[0])[3] = ((uint8_t*)&sharedMemory1[0])[s2[12]];
	((uint8_t*)&v[0])[6] = ((uint8_t*)&sharedMemory1[0])[s2[13]];
	((uint8_t*)&v[0])[9] = ((uint8_t*)&sharedMemory1[0])[s2[14]];
	((uint8_t*)&v[0])[12] = ((uint8_t*)&sharedMemory1[0])[s2[15]];

	t = v[0];
	w = v[0] ^ v[1];
	u = w ^ v[2] ^ v[3];
	v[0] = v[0] ^ u ^ XT4(w);
	v[1] = v[1] ^ u ^ XT4(v[1] ^ v[2]);
	v[2] = v[2] ^ u ^ XT4(v[2] ^ v[3]);
	v[3] = v[3] ^ u ^ XT4(v[3] ^ t);


	s2[0] = ((uint8_t*)&v[0])[0];
	s2[1] = ((uint8_t*)&v[0])[4];
	s2[2] = ((uint8_t*)&v[0])[8];
	s2[3] = ((uint8_t*)&v[0])[12];
	s2[4] = ((uint8_t*)&v[0])[1];
	s2[5] = ((uint8_t*)&v[0])[5];

	s2[6] = ((uint8_t*)&v[0])[9];
	s2[7] = ((uint8_t*)&v[0])[13];
	s2[8] = ((uint8_t*)&v[0])[2];
	s2[9] = ((uint8_t*)&v[0])[6];
	s2[10] = ((uint8_t*)&v[0])[10];
	s2[11] = ((uint8_t*)&v[0])[14];
	s2[12] = ((uint8_t*)&v[0])[3];
	s2[13] = ((uint8_t*)&v[0])[7];
	s2[14] = ((uint8_t*)&v[0])[11];
	s2[15] = ((uint8_t*)&v[0])[15];

	((uint128m*)&s2[0])[0] = make_uint4(((uint32_t*)&s2[0])[0] ^ rk[3].x, ((uint32_t*)&s2[0])[1] ^ rk[3].y, ((uint32_t*)&s2[0])[2] ^ rk[3].z, ((uint32_t*)&s2[0])[3] ^ rk[3].w);



	_mm_unpackboth_epi32_emu(((uint128m*)&s1[0])[0], ((uint128m*)&s2[0])[0]);



}


__device__  __forceinline__ uint128m _mm_cvtsi32_si128_emu(uint32_t lo)
{
	uint128m result = { 0 };
	result.x= lo;
	//((uint32_t *)&result)[1] = 0;
//	((uint64_t *)&result)[1] = 0;
	return result;
}
__device__  __forceinline__ uint128m _mm_cvtsi64_si128_emu(uint64_t lo)
{
	uint128m result = {0};
	((uint64_t *)&result)[0] = lo;
	//((uint64_t *)&result)[1] = 0;
	return result;
}
__device__  __forceinline__ uint128m _mm_set_epi64x_emu(uint64_t hi, uint64_t lo)
{
	uint128m result;
	((uint64_t *)&result)[0] = lo;
	((uint64_t *)&result)[1] = hi;
	return result;
}
__device__  __forceinline__ uint128m _mm_shuffle_epi8_emu(uint128m a, uint128m b)
{
	uint128m result;
	for (int i = 0; i < 16; i++)
	{
		if (((uint8_t *)&b)[i] & 0x80)
		{
			((uint8_t *)&result)[i] = 0;
		}
		else
		{
			((uint8_t *)&result)[i] = ((uint8_t *)&a)[((uint8_t *)&b)[i] & 0xf];
		}
	}

	return result;
}



__device__  __forceinline__ uint128m _mm_srli_si128_emu(uint128m input, int imm8)
{
	//we can cheat here as its an 8 byte shift just copy the 64bits
	uint128m temp;
	((uint64_t*)&temp)[0] = ((uint64_t*)&input)[1];
	((uint64_t*)&temp)[1] = 0;


	return temp;
}



__device__  uint128m _mm_mulhrs_epi16_emu(uint128m _a, uint128m _b)
{
	int16_t result[8];
	const int32_t jim = 0x4000;
	int32_t pip, pop, po;
	int16_t *a = (int16_t*)&_a, *b = (int16_t*)&_b;
#pragma unroll 8
	for (int i = 0; i < 8; i++)
	{
		pip = a[i];
		pop = b[i];
		asm("mad.lo.s32 %0, %1, %2, %3; ": "=r"(po) : "r"(pip), "r"(pop), "r"(jim));

		result[i] = po >> 15;
	//	result[i] = (int16_t)((((int32_t)(a[i]) * (int32_t)(b[i])) + 0x4000) >> 15);
	
	}
	return *(uint128m *)result;
}


__device__  __forceinline__ void case_0(uint128m &prand, uint128m &prandex, const  uint128m *pbuf,
	uint64_t selector, uint128m &acc)
{
	const uint128m temp1 = prandex;

	const uint128m temp2 = _mm_load_si128_emu(pbuf - (((selector & 1) << 1) - 1));


	const uint128m add1 = _mm_xor_si128_emu(temp1, temp2);

	const uint128m clprod1 = _mm_clmulepi64_si128_emu(add1, add1, 0x10);
	acc = _mm_xor_si128_emu(clprod1, acc);

	const uint128m tempa1 = _mm_mulhrs_epi16_emu(acc, temp1);
	const uint128m tempa2 = _mm_xor_si128_emu(tempa1, temp1);

	const uint128m temp12 = prand;
	prand = tempa2;


	const uint128m temp22 = _mm_load_si128_emu(pbuf);
	const uint128m add12 = _mm_xor_si128_emu(temp12, temp22);
	const uint128m clprod12 = _mm_clmulepi64_si128_emu(add12, add12, 0x10);
	acc = _mm_xor_si128_emu(clprod12, acc);

	const uint128m tempb1 = _mm_mulhrs_epi16_emu(acc, temp12);
	const uint128m tempb2 = _mm_xor_si128_emu(tempb1, temp12);
	prandex = tempb2;

}

__device__  __forceinline__ void case_4(uint128m &prand, uint128m &prandex, const  uint128m *pbuf,
	uint64_t selector, uint128m &acc)
{
	const uint128m temp1 = prand;
	const uint128m temp2 = _mm_load_si128_emu(pbuf);
	const uint128m add1 = _mm_xor_si128_emu(temp1, temp2);
	const uint128m clprod1 = _mm_clmulepi64_si128_emu(add1, add1, 0x10);
	acc = _mm_xor_si128_emu(clprod1, acc);
	const uint128m clprod2 = _mm_clmulepi64_si128_emu(temp2, temp2, 0x10);
	acc = _mm_xor_si128_emu(clprod2, acc);

	const uint128m tempa1 = _mm_mulhrs_epi16_emu(acc, temp1);
	const uint128m tempa2 = _mm_xor_si128_emu(tempa1, temp1);

	const uint128m temp12 = prandex;
	prandex = tempa2;

	const uint128m temp22 = _mm_load_si128_emu(pbuf - (((selector & 1) << 1) - 1));
	const uint128m add12 = _mm_xor_si128_emu(temp12, temp22);
	acc = _mm_xor_si128_emu(add12, acc);

	const uint128m tempb1 = _mm_mulhrs_epi16_emu(acc, temp12);
	const uint128m tempb2 = _mm_xor_si128_emu(tempb1, temp12);
	prand = tempb2;
}

__device__ __forceinline__  void case_8(uint128m &prand, uint128m &prandex, const  uint128m *pbuf,
	uint64_t selector, uint128m &acc)
{
	const uint128m temp1 = prandex;
	const uint128m temp2 = _mm_load_si128_emu(pbuf);
	const uint128m add1 = _mm_xor_si128_emu(temp1, temp2);
	acc = _mm_xor_si128_emu(add1, acc);

	const uint128m tempa1 = _mm_mulhrs_epi16_emu(acc, temp1);
	const uint128m tempa2 = _mm_xor_si128_emu(tempa1, temp1);

	const uint128m temp12 = prand;
	prand = tempa2;

	const uint128m temp22 = _mm_load_si128_emu(pbuf - (((selector & 1) << 1) - 1));
	const uint128m add12 = _mm_xor_si128_emu(temp12, temp22);
	const uint128m clprod12 = _mm_clmulepi64_si128_emu(add12, add12, 0x10);
	acc = _mm_xor_si128_emu(clprod12, acc);
	const uint128m clprod22 = _mm_clmulepi64_si128_emu(temp22, temp22, 0x10);
	acc = _mm_xor_si128_emu(clprod22, acc);

	const uint128m tempb1 = _mm_mulhrs_epi16_emu(acc, temp12);
	const uint128m tempb2 = _mm_xor_si128_emu(tempb1, temp12);
	prandex = tempb2;
}

__device__ void case_0c(uint128m &prand, uint128m &prandex, const  uint128m *pbuf,
	uint64_t selector, uint128m &acc)
{
	const uint128m temp1 = prand;
	const uint128m temp2 = _mm_load_si128_emu(pbuf - (((selector & 1) << 1) - 1));
	const uint128m add1 = _mm_xor_si128_emu(temp1, temp2);

	// cannot be zero here
	const int32_t divisor = ((uint32_t*)&selector)[0];

	acc = _mm_xor_si128_emu(add1, acc);

	int64_t dividend = _mm_cvtsi128_si64_emu(acc);
	int64_t tmpmod = dividend % divisor;
	const uint128m modulo = _mm_cvtsi32_si128_emu(tmpmod);
	acc = _mm_xor_si128_emu(modulo, acc);

	const uint128m tempa1 = _mm_mulhrs_epi16_emu(acc, temp1);
	const uint128m tempa2 = _mm_xor_si128_emu(tempa1, temp1);
	dividend &= 1;
	if (dividend)
	{
		const uint128m temp12 = prandex;
		prandex = tempa2;

		const uint128m temp22 = _mm_load_si128_emu(pbuf);
		const uint128m add12 = _mm_xor_si128_emu(temp12, temp22);
		const uint128m clprod12 = _mm_clmulepi64_si128_emu(add12, add12, 0x10);
		acc = _mm_xor_si128_emu(clprod12, acc);
		const uint128m clprod22 = _mm_clmulepi64_si128_emu(temp22, temp22, 0x10);
		acc = _mm_xor_si128_emu(clprod22, acc);

		const uint128m tempb1 = _mm_mulhrs_epi16_emu(acc, temp12);
		const uint128m tempb2 = _mm_xor_si128_emu(tempb1, temp12);
		prand = tempb2;
	}
	else
	{
		const uint128m tempb3 = prandex;
		prandex = tempa2;
		prand = tempb3;
	}
}
__device__ void case_10(uint128m &prand, uint128m &prandex, const  uint128m *pbuf,
	uint64_t selector, uint128m &acc, uint128m *randomsource, uint32_t prand_idx, uint32_t *sharedMemory1)
{			// a few AES operations
			//uint128m rc[12];

			//rc[0] = prand; 

	uint128m *rc = &randomsource[prand_idx];
	/*	rc[2] = randomsource[prand_idx + 2];
	rc[3] = randomsource[prand_idx + 3];
	rc[4] = randomsource[prand_idx + 4];
	rc[5] = randomsource[prand_idx + 5];
	rc[6] = randomsource[prand_idx + 6];
	rc[7] = randomsource[prand_idx + 7];
	rc[8] = randomsource[prand_idx + 8];
	rc[9] = randomsource[prand_idx + 9];
	rc[10] = randomsource[prand_idx + 10];
	rc[11] = randomsource[prand_idx + 11];8*/
//	uint128m tmp;

	uint128m temp1 = _mm_load_si128_emu(pbuf - (((selector & 1) << 1) - 1));
	uint128m temp2 = _mm_load_si128_emu(pbuf);

	AES2_EMU2(temp1, temp2, 0);
//	MIX2_EMU(temp1, temp2);


	AES2_EMU2(temp1, temp2, 4);
//	MIX2_EMU(temp1, temp2);

	AES2_EMU2(temp1, temp2, 8);
//	MIX2_EMU(temp1, temp2);


	acc = _mm_xor_si128_emu(temp1, acc);
	acc = _mm_xor_si128_emu(temp2, acc);

	const uint128m tempa1 = prand;
	const uint128m tempa2 = _mm_mulhrs_epi16_emu(acc, tempa1);
	const uint128m tempa3 = _mm_xor_si128_emu(tempa1, tempa2);

	const uint128m tempa4 = prandex;
	prandex = tempa3;
	prand = tempa4;
}
__device__ void case_14(uint128m &prand, uint128m &prandex, const  uint128m *pbuf,
	uint64_t selector, uint128m &acc, uint128m *randomsource, uint32_t prand_idx, uint32_t *sharedMemory1)
{
	// we'll just call this one the monkins loop, inspired by Chris
	const uint128m *buftmp = pbuf - (((selector & 1) << 1) - 1);
//	uint128m tmp; // used by MIX2

	uint64_t rounds = selector >> 61; // loop randomly between 1 and 8 times
	uint128m *rc = &randomsource[prand_idx];


	uint64_t aesround = 0;
	uint128m onekey;
	uint64_t loop_c;

	do
	{
		loop_c = selector & (0x10000000 << rounds);
		if (loop_c)
		{
			onekey = _mm_load_si128_emu(rc++);
			const uint128m temp2 = _mm_load_si128_emu(rounds & 1 ? pbuf : buftmp);
			const uint128m add1 = _mm_xor_si128_emu(onekey, temp2);
			const uint128m clprod1 = _mm_clmulepi64_si128_emu(add1, add1, 0x10);
			acc = _mm_xor_si128_emu(clprod1, acc);
		}
		else
		{
			onekey = _mm_load_si128_emu(rc++);
			uint128m temp2 = _mm_load_si128_emu(rounds & 1 ? buftmp : pbuf);

			const uint64_t roundidx = aesround++ << 2;
			AES2_EMU2(onekey, temp2, roundidx);

		//	MIX2_EMU(onekey, temp2);

			acc = _mm_xor_si128_emu(onekey, acc);
			acc = _mm_xor_si128_emu(temp2, acc);

		}

	} while (rounds--);

	const uint128m tempa1 = (prand);
	const uint128m tempa2 = _mm_mulhrs_epi16_emu(acc, tempa1);
	const uint128m tempa3 = _mm_xor_si128_emu(tempa1, tempa2);

	const uint128m tempa4 = (prandex);
	prandex = tempa3;
	prand = tempa4;
}

__device__ void __forceinline__  case_18(uint128m &prand, uint128m &prandex, const  uint128m *pbuf,
	uint64_t selector, uint128m &acc)
{
	const uint128m temp1 = _mm_load_si128_emu(pbuf - (((selector & 1) << 1) - 1));
	const uint128m temp2 = (prand);
	const uint128m add1 = _mm_xor_si128_emu(temp1, temp2);
	const uint128m clprod1 = _mm_clmulepi64_si128_emu(add1, add1, 0x10);
	acc = _mm_xor_si128_emu(clprod1, acc);

	const uint128m tempa1 = _mm_mulhrs_epi16_emu(acc, temp2);
	const uint128m tempa2 = _mm_xor_si128_emu(tempa1, temp2);

	const uint128m tempb3 = (prandex);
	prandex = tempa2;
	prand = tempb3;
}

__device__  __forceinline__ void case_1c(uint128m &prand, uint128m &prandex, const  uint128m *pbuf,
	uint64_t selector, uint128m &acc)
{
	const uint128m temp1 = _mm_load_si128_emu(pbuf);
	const uint128m temp2 = (prandex);
	const uint128m add1 = _mm_xor_si128_emu(temp1, temp2);
	const uint128m clprod1 = _mm_clmulepi64_si128_emu(add1, add1, 0x10);
	acc = _mm_xor_si128_emu(clprod1, acc);


	const uint128m tempa1 = _mm_mulhrs_epi16_emu(acc, temp2);
	const uint128m tempa2 = _mm_xor_si128_emu(tempa1, temp2);
	const uint128m tempa3 = (prand);


	prand = tempa2;

	acc = _mm_xor_si128_emu(tempa3, acc);

	const uint128m tempb1 = _mm_mulhrs_epi16_emu(acc, tempa3);
	const uint128m tempb2 = _mm_xor_si128_emu(tempb1, tempa3);
	prandex = tempb2;
}



__device__ uint128m __verusclmulwithoutreduction64alignedrepeatgpu(uint128m * __restrict__ randomsource, const  uint128m *  __restrict__  buf ,
	uint64_t keyMask, uint32_t *  __restrict__ sharedMemory1, uint32_t *  __restrict__ d_fix_r, uint32_t *  __restrict__ d_fix_rex)
{
    uint128m const *pbuf;
	keyMask >>= 4;
	uint128m acc = randomsource[keyMask + 2];
	
#ifdef GPU_DEBUGGY
	if (nounce == 0)
	{
		printf("[GPU]BUF ito verusclmulithout        : ");
		for (int i = 0; i < 64; i++)
			printf("%02x", ((uint8_t*)&buf[0])[i]);
		printf("\n");
		printf("[GPU]KEy ito verusclmulithout        : ");
		for (int e = 0; e < 64; e++)
		printf("%02x", ((uint8_t*)&randomsource[0])[e]);
	printf("\n");
	    printf("[GPU]ACC ito verusclmulithout        : ");
	for (int i = 0; i < 16; i++)
		printf("%02x", ((uint8_t*)&acc)[i]);
	printf("\n");
	}
#endif	
	// divide key mask by 32 from bytes to uint128m
	
	uint32_t prand_idx, prandex_idx;
	uint64_t selector;
	uint128m prand;
	uint128m prandex;

	for (int64_t i = 0; i < 32; i++)
	{
		
		selector = _mm_cvtsi128_si64_emu(acc);

		
		prand_idx = ((selector >> 5) & keyMask);
		prandex_idx = ((selector >> 32) & keyMask);
		// get two random locations in the key, which will be mutated and swapped
		
		prand = randomsource[prand_idx];
		prandex = randomsource[prandex_idx];

	//	save_rand[i] = ((selector >> 5) & keyMask);
	//	save_randex[i] = ((selector >> 32) & keyMask);

		// select random start and order of pbuf processing
		pbuf = buf + (selector & 3);
		uint64_t case_v;
		case_v = selector &  0x1cu;
#ifdef GPU_DEBUGu
		uint64_t egg, nog, salad;
		if (nounce == 0)
		{
			printf("[GPU]*****LOOP[%d]**********\n",i);
			egg = selector & 0x03u;
			nog = ((selector >> 32) & keyMask);
			salad = ((selector >> 5) & keyMask);
			printf("[GPU]selector: %llx\n case: %llx selector &3: ", selector, case_v);
			printf("%llx \n", egg);
			printf("[GPU]((selector >> 32) & keyMask) %d",nog);
			printf("[GPU]((selector >> 5) & keyMask) %d", salad);
			printf("\nacc     : ");
			printf("%016llx%016llx", ((uint64_t*)&acc)[0], ((uint64_t*)&acc)[1]);
			printf("\n");

			printf("[GPU]prand   : ");
			//for (int e = 0; e < 4; e++)
			printf("%016llx%016llx", ((uint64_t*)&prand)[0], ((uint64_t*)&prand)[1]);
			printf("\n");
			printf("[GPU]prandex : ");
			//for (int e = 0; e < 16; e++)
			printf("%016llx%016llx", ((uint64_t*)&prandex)[0], ((uint64_t*)&prandex)[1]);
			printf("\n");


		}

#endif
		
		if(case_v == 0)
		{
			case_0(prand, prandex, pbuf, selector, acc);
		}
		if (case_v == 4)
		{
			case_4(prand, prandex, pbuf, selector, acc);
		}
		if (case_v == 8)
		{
			case_8(prand, prandex, pbuf, selector, acc);
			
		}
		if (case_v == 0xc)
		{
			case_0c(prand, prandex, pbuf, selector, acc);

		}
		if (case_v == 0x10)
		{
			case_10(prand, prandex, pbuf, selector, acc,randomsource, prand_idx, sharedMemory1);


		}
		if(case_v == 0x14)
		{
			case_14(prand, prandex, pbuf, selector, acc, randomsource, prand_idx, sharedMemory1);

		}
		if(case_v == 0x18)
		{
			case_18(prand, prandex, pbuf, selector, acc);
			
		}
		if(case_v == 0x1c)
		{
			case_1c(prand, prandex, pbuf, selector, acc);
		}	
		d_fix_r[i] = prand_idx;
		d_fix_rex[i] = prandex_idx;
		 randomsource[prand_idx] = prand;
		 randomsource[prandex_idx] = prandex;

	}

	return acc;
}


__device__   __forceinline__ void haraka512_port_keyed2222(unsigned char *out, const unsigned char *in, uint128m *rc, uint32_t *sharedMemory1, uint32_t nonce)
{
	uint128m s1,s2,s3,s4, tmp;

	s1 = ((uint128m*)&in[0])[0];
	s2 = ((uint128m*)&in[0])[1];
	s3 = ((uint128m*)&in[0])[2];
	s4 = ((uint128m*)&in[0])[3];

	AES4(s1, s2, s3, s4, 0);
	MIX4(s1, s2, s3, s4);

	AES4(s1, s2, s3, s4, 8);
	MIX4(s1, s2, s3, s4);

	AES4(s1, s2, s3, s4, 16);
	MIX4(s1, s2, s3, s4);

	AES4(s1, s2, s3, s4, 24);
	MIX4(s1, s2, s3, s4);

	AES4(s1, s2, s3, s4, 32);
	MIX4(s1, s2, s3, s4);

	//s[0] = _mm_xor_si128_emu(s[0], ((uint128m*)&in[0])[0]);
	//s[1] = _mm_xor_si128_emu(s[1], ((uint128m*)&in[0])[1]);
	//s[2] = _mm_xor_si128_emu(s[2], ((uint128m*)&in[0])[2]);
	s4 = _mm_xor_si128_emu(s4, ((uint128m*)&in[0])[3]);

	TRUNCSTORE(out, s4);

	//((uint32_t*)&out[0])[7] = ((uint32_t*)&s[52])[0] ^ ((uint32_t*)&in[52])[0];

}

__device__   __forceinline__ uint128m precompReduction64_si128(uint128m A) {

	//const uint128m C = _mm_set_epi64x(1U,(1U<<4)+(1U<<3)+(1U<<1)+(1U<<0)); // C is the irreducible poly. (64,4,3,1,0)
	//const uint128m C = _mm_cvtsi64_si128_emu(27U);
	uint128m M;
	M.x = 0x2d361b00;
	M.y = 0x415a776c;
	M.z = 0xf5eec3d8;
	M.w = 0x9982afb4;


	uint128m Q2 = _mm_clmulepi64_si128_emu(A, _mm_cvtsi64_si128_emu(27U), 0x01);
	uint128m Q3 = _mm_shuffle_epi8_emu(M,_mm_srli_si128_emu(Q2, 8));

	uint128m Q4 = _mm_xor_si128_emu(Q2, A);
	const uint128m final = _mm_xor_si128_emu(Q3, Q4);
	return final;/// WARNING: HIGH 64 BITS SHOULD BE ASSUMED TO CONTAIN GARBAGE
}

__device__  __forceinline__ uint64_t precompReduction64(uint128m A) {
	uint128m tmp = precompReduction64_si128(A);
	return _mm_cvtsi128_si64_emu(tmp);
}

__global__ __launch_bounds__(THREADS, 1)
void verus_gpu_hash(const uint32_t threads, const uint32_t startNonce, uint32_t * __restrict__ resNonce,
	uint128m * __restrict__ d_key_input, uint128m * __restrict__ d_mid, uint32_t * __restrict__  d_fix_r, uint32_t *__restrict__  d_fix_rex)
{
	const uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	uint128m mid; // , biddy[VERUS_KEY_SIZE128];
	uint8_t s[64];
	uint32_t nounce = startNonce + thread;

	__shared__ uint32_t sharedMemory1[THREADS];
	__shared__ uint32_t sharedrand[32 * THREADS];
	__shared__ uint32_t sharedrandex[32 * THREADS];

	//uint32_t save_rand[32] = { 0 };
	//uint32_t save_randex[32] = { 0 };

	((uint4*)&s[0])[0] = ((uint4*)&blockhash_half[0])[0];
	((uint4*)&s[0])[1] = ((uint4*)&blockhash_half[0])[1];
	((uint4*)&s[0])[2] = ((uint4*)&blockhash_half[0])[2];
	((uint4*)&s[0])[3] = ((uint4*)&blockhash_half[0])[3];


	sharedMemory1[threadIdx.x] = sbox[threadIdx.x];// copy sbox to shared mem

	((uint32_t *)&s)[8] = nounce;

	const uint128m lazy = make_uint4(0x00010000, 0x00000000, 0x00000000, 0x00000000);

	__syncthreads();
	mid = __verusclmulwithoutreduction64alignedrepeatgpu(&d_key_input[VERUS_KEY_SIZE128 * thread], (uint128m*)s, 8191, sharedMemory1, sharedrand + (threadIdx.x * 32), sharedrandex + (threadIdx.x * 32));

	d_mid[thread] = _mm_xor_si128_emu(mid, lazy);

#pragma unroll
	for (int i = 0; i < 32; i++)
	{
		d_fix_r[(thread * 32) + i] = sharedrand[(threadIdx.x * 32)+i];
		d_fix_rex[(thread * 32) + i] = sharedrandex[(threadIdx.x * 32) + i];
	}
}
	__global__ __launch_bounds__(THREADS, 1)
		void verus_gpu_final(const uint32_t threads, const uint32_t startNonce, uint32_t * __restrict__ resNonce,
			uint128m * __restrict__  d_key_input, const uint128m * __restrict__ d_mid)
	{
		const uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
		uint64_t acc;
		uint128m wizz = d_mid[thread];

		uint32_t nounce = startNonce + thread, hash[32] = { 0 };

		uint8_t s[64];
		__shared__ uint32_t sharedMemory1[THREADS];
		sharedMemory1[threadIdx.x] = sbox[threadIdx.x];// copy sbox to shared mem
		((uint4*)&s[0])[0] = ((uint4*)&blockhash_half[0])[0];
		((uint4*)&s[0])[1] = ((uint4*)&blockhash_half[0])[1];
		((uint4*)&s[0])[2] = ((uint4*)&blockhash_half[0])[2];
		((uint4*)&s[0])[3] = ((uint4*)&blockhash_half[0])[3];
		__syncthreads();
	acc = precompReduction64(wizz);
	((uint32_t *)&s)[8] = nounce;
	memcpy(s + 47, &acc, 8);
	memcpy(s + 55, &acc, 8);
	memcpy(s + 63, &acc, 1);
	//uint64_t mask = 8191 >> 4;
	acc &= 511;
	
	//haraka512_port_keyed((unsigned char*)hash, (const unsigned char*)s, (const unsigned char*)(biddy + mask), sharedMemory1, nounce);
	haraka512_port_keyed2222((unsigned char*)hash, (const unsigned char*)s, (&d_key_input[VERUS_KEY_SIZE128 * thread] + acc), sharedMemory1,nounce);

	if (hash[7] < ptarget[7]) { 
		
		resNonce[0] = nounce;

	}


};

__global__ __launch_bounds__(128, 1)
void verus_extra_gpu_prepare(const uint32_t threads, uint128m * d_key_input)
{

	d_key_input[(blockIdx.x * VERUS_KEY_SIZE128) + threadIdx.x] = vkey[threadIdx.x];
	d_key_input[(blockIdx.x * VERUS_KEY_SIZE128) + threadIdx.x + 128 ] = vkey[threadIdx.x + 128];
	d_key_input[(blockIdx.x * VERUS_KEY_SIZE128) + threadIdx.x + 256 ] = vkey[threadIdx.x + 256];
	d_key_input[(blockIdx.x * VERUS_KEY_SIZE128) + threadIdx.x + 384 ] = vkey[threadIdx.x + 384];
	if (threadIdx.x < 40)
		d_key_input[(blockIdx.x * VERUS_KEY_SIZE128) + threadIdx.x + 512 ] = vkey[threadIdx.x + 512];

}

__global__ __launch_bounds__(32, 1)
void verus_extra_gpu_fix(const uint32_t threads, uint128m * __restrict__ d_key_input, uint32_t *d_fix_r, uint32_t *d_fix_rex)
{
	
	d_key_input[(blockIdx.x * VERUS_KEY_SIZE128) + d_fix_r[(blockIdx.x * 32) + threadIdx.x]] = vkey[d_fix_r[(blockIdx.x * 32) +threadIdx.x]];
	d_key_input[(blockIdx.x * VERUS_KEY_SIZE128) + d_fix_rex[(blockIdx.x * 32) + threadIdx.x]] = vkey[d_fix_rex[(blockIdx.x * 32) + threadIdx.x]];
	
}